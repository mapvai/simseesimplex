#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "toursimplexgpus.h"

const double CasiCero_Simplex = 1.0E-7;
// const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

void resolver_cpu(TSimplexGPUs &simplex) ;
bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);
int locate_zpos(TSimplexGPUs &smp);
int locate_qpos(TSimplexGPUs &smp, int zpos);
void resolver_ejemplo1();
void resolver_ejemplo2();
void printStatus(TSimplexGPUs &smp);
void printResult(TSimplexGPUs &smp);

extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	/*
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]);
	}
	*/
	resolver_ejemplo1();
	resolver_ejemplo2();
	
}

void resolver_ejemplo1() {
	
	/*
	Problema Propuesto por Cami
	  max z = 7x1 + 4x2
	  s.a.
		2x1 + x2   <= 20
		x1  + x2   <= 18
		x1         <= 8  
	*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 5;
	simplex.NRestricciones = 3;
	
	simplex.top = (int*)malloc((simplex.NVariables)*sizeof(int));
	for (int i = 0; i < simplex.NVariables; i++) simplex.top[i] = i +1;
	
    simplex.left = (int*)malloc((simplex.NRestricciones)*sizeof(int));
	for (int i = 0; i < simplex.NRestricciones; i++) simplex.left[i] = i +1;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		-7, -4, 0, 0, 0, 0, 
		2, 1, 1, 0, 0, 20,
		1, 1, 0, 1, 0, 18,
		1, 0, 0, 0, 1, 8,
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}

void resolver_ejemplo2() {
	
	/*
	Problema Propuesto por Cami, mas que agrego una restriccion de igualdad, con 0 en las slack variables
	  max z = 7x1 + 4x2
	  s.a.
		2x1 + x2   <= 20
		x1  + x2   <= 18
		x1         <= 8  
	*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 5;
	simplex.NRestricciones = 4;
	
	simplex.top = (int*)malloc((simplex.NVariables)*sizeof(int));
	for (int i = 0; i < simplex.NVariables; i++) simplex.top[i] = i + 1;
	
    simplex.left = (int*)malloc((simplex.NRestricciones)*sizeof(int));
	for (int i = 0; i < simplex.NRestricciones; i++) simplex.left[i] = i - 1;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		-7, -4, 0, 0, 0, 0, 
		2, 1, 1, 0, 0, 20,
		1, 1, 0, 1, 0, 18,
		1, 0, 0, 1, 0, 8,
		0, 1, 0, 0, 0, 16
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}


void resolver_cpu(TSimplexGPUs &simplex) {
	int zpos, qpos, it;
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_zpos(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s\n", "Condicion de parada maximo encontrado");
			printResult(simplex);
			return;
		}
		
		qpos = locate_qpos(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s\n", "Posicion de cociente no encontrada");
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		if (it == 4) return;
		
	} while (true);
	
}


int locate_zpos(TSimplexGPUs &smp) {
	int mejorz, z;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 0; z < smp.NVariables; z++) {
		apz = smp.tabloide[z];
		if (apz < 0 && apz < min_apz) {
			mejorz = z;
			min_apz = apz;
		}
	}
	
	return mejorz;
}


int locate_qpos(TSimplexGPUs &smp, int zpos) {
	int mejorq, q;
	double min_apq, apq, qz;

	mejorq = -1;
	min_apq = MaxNReal;
	for (q = 1; q <= smp.NRestricciones; q++) {
		qz = smp.tabloide[q * (smp.NVariables + 1) + zpos]; // [q][zpos]
		if (qz > CasiCero_Simplex) { // > 0
			apq = smp.tabloide[q * (smp.NVariables + 1) + smp.NVariables]/qz;
			if (apq < min_apq) {
				mejorq = q;
				min_apq = apq;
			}
		}
	}
	
	return mejorq;
	
}

bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int i, j, ipos, k;

	piv = smp.tabloide[kfil * (smp.NVariables + 1) + jcol];
	invPiv = 1 / piv;
	
	ipos = kfil * (smp.NVariables + 1) ;
	for (j = 0; j <= smp.NVariables; j++) {
		smp.tabloide[ipos + j] *= invPiv;
	}
	
	for (i = 0; i <= smp.NRestricciones; i++) {
		if (i != kfil) {
			m = smp.tabloide[i * (smp.NVariables + 1) + jcol] ;
			for (j = 0; j <= smp.NVariables; j++) {
				if (j != jcol) {
					smp.tabloide[i * (smp.NVariables + 1) + j] -= m * smp.tabloide[kfil * (smp.NVariables + 1) + j]; 
				} else {
					smp.tabloide[i * (smp.NVariables + 1) + j] = 0;
				}
			}
		}
	}
	
	for (i = 0; i <= smp.NRestricciones; i++) {
		if (i != kfil) {
			smp.tabloide[i * (smp.NVariables + 1) + jcol] /= -invPiv; 
		} else {
			smp.tabloide[i * (smp.NVariables + 1) + jcol]  = invPiv;
		}
	}
	
	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil];
	smp.left[kfil] = k;

	return true;
 }


void printStatus(TSimplexGPUs &smp) {
	printf("%s\n", "Tabloide");
	for(int i = 0; i <= smp.NRestricciones; i++) {
		for(int j = 0; j <= smp.NVariables; j++) {
			printf("%f ", smp.tabloide[i* (smp.NVariables + 1) + j] );
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	int indx;
	string nvar = "xos";
	int nobvari = smp.NVariables - smp.NRestricciones;
	
	printf("left = [%i", smp.left[0]);
	for(int i = 1; i <= smp.NRestricciones; i++) printf(", %i", smp.left[i]);
	printf("]\n");
	
	printf("z = %f\n", smp.tabloide[smp.NVariables]);
	
	for(int i = 1; i <= smp.NRestricciones; i++) {
		if (smp.left[i] > 0) {
			if (smp.left[i] <= nobvari) {
				nvar = "x";
				indx = smp.left[i];
			} else {
				nvar = "s";
				indx = smp.left[i] - nobvari;
			}
		} else {
			nvar = "s";
			indx = -smp.left[i];
		}
		
		printf("%s%i = %f \n", nvar.c_str(), indx,  smp.tabloide[i* (smp.NVariables + 1) + smp.NVariables]);
	}
	
}





