#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "toursimplexmgpus_final.h"

#define BLOCK_SIZE_P_I 64 //  = 2 * WARP SIZE
#define BLOCK_SIZE_GR 160 //  = 5 * WARP SIZE

const double CasiCero_Simplex = 1.0E-7;
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 1.0E+150; //100; //sqrt(MaxNReal);


// 8 * 32 = 256
const int BLOCK_SIZE_E_1X = 32;  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy the arrange of the warp in the block is giving for a two-dimensional block of size (Dx, Dy),the thread ID of a thread of index (x, y) is (x + y Dx)
const int BLOCK_SIZE_E_1Y = 8;

// 8 * 32 = 256
const int BLOCK_SIZE_E_2X = 32;
const int BLOCK_SIZE_E_2Y = 8;

// 8 * 32 = 256
const int BLOCK_SIZE_E_3X = 32;
const int BLOCK_SIZE_E_3Y = 8;

// 8 * 32 = 256
const int BLOCK_SIZE_E_4X = 32;
const int BLOCK_SIZE_E_4Y = 8;

// 8 * 32 = 256
const int BLOCK_SIZE_E_5X = 32;
const int BLOCK_SIZE_E_5Y = 8;

__device__ TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide);
__device__ void moverseASolFactible(TabloideGPUs &tabloide);
__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide);
__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide);

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide);

__device__ int locate_min_dj(TSimplexGPUs &smp);
__device__ int locate_min_ratio(TSimplexGPUs &smp, int zpos);
__device__ bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);

__device__ void printStatus(TSimplexGPUs &smp);
__device__ void printResult(TSimplexGPUs &smp);
__device__ double findVarXbValue(TSimplexGPUs &smp, int indx);
__device__ int findVarIndex(TSimplexGPUs &smp, int indx);

__global__ void kernel_resolver_etapa_moverse_a_sol_factible(TDAOfSimplexGPUs simplex_array) {
	moverseASolFactible(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_restricciones_cota_sup(TDAOfSimplexGPUs simplex_array) {
	agregarRestriccionesCotaSup(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_variables_holgura_artificiales(TDAOfSimplexGPUs simplex_array) {
	agregarVariablesHolguraArtificiales(simplex_array[blockIdx.x]);
}

__global__ void kernel_resolver_simplex_big_m(TDAOfSimplexGPUs simplex_array) {
	resolver_simplex_big_m(simplex_array[blockIdx.x]);
}


extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	int largo, alto, n_vars, n_rest;
	
	int maxVars = 0;
	int maxRest = 0;
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][3];
		n_vars = (int) simplex_array[kTrayectoria][largo];
		n_rest = (int) simplex_array[kTrayectoria][largo + 1];
		alto = n_rest + 6;
		
		if (n_vars > maxVars) maxVars = n_vars;
		if (n_rest > maxRest) maxRest = n_rest;
		
		hipMemcpy(h_simplex_array[kTrayectoria], simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyHostToDevice);
		
	}

	hipMemcpy(d_simplex_array, h_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyHostToDevice);
	
	hipError_t err;
	
	// Ejecuto los kernels
	// const dim3 DimBlock_e1(maxRest, 1);
	const dim3 DimGrid_e1(NTrayectorias, 1);
	const dim3 DimBlock_e1(BLOCK_SIZE_E_1X, BLOCK_SIZE_E_1Y);
	kernel_resolver_etapa_moverse_a_sol_factible<<< DimGrid_e1, DimBlock_e1, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 1 error", hipGetErrorString(err));
	
	// const dim3 DimBlock_e2(maxVars, 1);
	const dim3 DimGrid_e2(NTrayectorias, 1);
	const dim3 DimBlock_e2(1, 1);
	kernel_resolver_etapa_agregar_restricciones_cota_sup<<< DimGrid_e2, DimBlock_e2, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 2 error", hipGetErrorString(err));
	
	// int cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques fijar_variables: ", cantBloques);
	// const dim3 DimGrid_e3(cantBloques, 1);
	// const dim3 DimGrid_e3(1, 1);
	// const dim3 DimBlock_e3(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e3(NTrayectorias, 1);
	const dim3 DimBlock_e3(1, 1);
	kernel_resolver_etapa_agregar_variables_holgura_artificiales<<< DimGrid_e3, DimBlock_e3, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 3 error", hipGetErrorString(err));
	
	// cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques enfilar_variables_libres: ", cantBloques);
	// const dim3 DimGrid_e4(cantBloques, 1);
	// const dim3 DimBlock_e4(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e4(NTrayectorias, 1);
	const dim3 DimBlock_e4(1, 1);
	kernel_resolver_simplex_big_m<<< DimGrid_e4, DimBlock_e4, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));
	
	hipMemcpy(h_simplex_array, d_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyDeviceToHost);

	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][3];
		alto = (int) simplex_array[kTrayectoria][largo + 1] + 6;
		hipMemcpy(simplex_array[kTrayectoria], h_simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyDeviceToHost);
	}

}

/**************************************************************************************************************************************************************************************************/

__device__ TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide) {
	TSimplexGPUs smp;
	
	smp.tabloide = tabloide;
	
	smp.var_x = (int) tabloide[0];
	smp.rest_ini = (int) tabloide[1];
	smp.mat_adv_row = (int) tabloide[2];
	smp.var_all = (int) tabloide[smp.mat_adv_row];
	smp.rest_fin = (int) tabloide[smp.mat_adv_row + 1];
	
	smp.z = &tabloide[4]; // funcion z, cantidad de variables, horizontal
    smp.flg_x = &tabloide[smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x
	
	smp.sup = &tabloide[2*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	smp.inf = &tabloide[3*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	
	smp.var_type = &tabloide[4*smp.mat_adv_row + 4]; // largo filas, horizontal
	
	smp.top = &tabloide[5*smp.mat_adv_row + 4]; // largo filas, horizontal
    smp.left = &tabloide[6*smp.mat_adv_row]; // largo restricciones finales, vertical
	
	smp.flg_y = &tabloide[6*smp.mat_adv_row + 1]; // 0 restriccion >=, 1 <=, 2 =, vertical
	
	
	smp.Cb = &tabloide[6*smp.mat_adv_row + 2]; // cantidad de restricciones, vertical
	smp.Xb = &tabloide[6*smp.mat_adv_row + 3]; // cantidad de restricciones, vertical
	
    smp.matriz = &tabloide[6*smp.mat_adv_row + 4]; 
	
	return smp;
}

__device__ void moverseASolFactible(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	
	for (int i = threadIdx.y; i < smp.rest_ini; i+= BLOCK_SIZE_E_1Y) {
		if (smp.Xb[i*smp.mat_adv_row] < 0) {
			if (threadIdx.y == 0) {
				smp.Xb[i*smp.mat_adv_row] *= -1;
				smp.flg_y[i*smp.mat_adv_row] = (smp.flg_y[i*smp.mat_adv_row] == 0) ? 1 : 2; // Move >= to <=
			}
			for (int threadIdx.x = 0; j < smp.var_x; j+= BLOCK_SIZE_E_1X) {
				smp.matriz[i*smp.mat_adv_row + j] *= -1;
			}
		}
	}
	
	//__syncthreads();
}

__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	
	int qrest = smp.rest_ini;
	for (int i = 0; i < smp.var_x; i++) {
		if (smp.flg_x[i] > 0) {
			smp.flg_y[(smp.rest_ini + i)*smp.mat_adv_row] = 1;
			smp.Xb[(smp.rest_ini + i)*smp.mat_adv_row] = smp.sup[i];
			for (int j = 0; j < smp.var_x; j++) smp.matriz[qrest*smp.mat_adv_row + j] = (qrest == (j + smp.rest_ini))? 1 : 0;
			qrest ++;
		}
	}
	// printf("Rest count %i / %i \n", smp.rest_fin, qrest);
	if (smp.rest_fin != qrest) printf("DISCREPANCIA EN LA CANTIDAD DE RESTRICCIONES FINAL\n");
	
}

__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	
	int var_s, var_a, var_count;
	var_s = 0; var_a = 0; var_count = smp.var_x;
	
	for (int i = 0; i < smp.var_x; i++) {
		smp.var_type[i] = 0;
		smp.top[i] = i + 1;
	}
	
	/* Esto ya deberia venir completo desde la CPU
	// Completo con 0s la matriz
	for (int i = 0; i < smp.rest_fin; i++) {
		for (int j = var_count; j < smp.var_all; j++) {
			smp.matriz[i*smp.mat_adv_row + j] = 0;
		}
	}
	*/
	
	for (int i = 0; i < smp.rest_fin; i++) {
		if (smp.flg_y[i*smp.mat_adv_row] == 0) { // rest >=
			smp.matriz[i*smp.mat_adv_row + var_count] = -1;
			smp.matriz[i*smp.mat_adv_row + var_count +1] = 1;
			
			smp.var_type[var_count] = 1;
			smp.var_type[var_count +1] = 2;
			
			smp.z[var_count] = 0;
			smp.z[var_count +1] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.top[var_count + 1] = var_count + 2;
			smp.left[i*smp.mat_adv_row] = var_count + 2;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_a++; var_count += 2;
		} else if (smp.flg_y[i*smp.mat_adv_row] == 1) { // rest <=
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 1;
			
			smp.z[var_count] = 0;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = 0;
			
			var_a++; var_count ++;
		} else { // 2: rest =
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 2;
			
			smp.z[var_count] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_count ++;
		}
	}
	
	//printf("Var count %i / %i \n", smp.var_all, var_count);
	if (smp.var_all != var_count) printf("DISCREPANCIA EN LA CANTIDAD DE VARIABLES FINAL\n");
	
}

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide) {
	
	TSimplexGPUs simplex = desestructurarTabloide(tabloide);
	
	int zpos, qpos, it;
	
	printf("resolver_simplex_big_m_final INT \n");
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_min_dj(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
			printResult(simplex);
			return;
		}
		
		qpos = locate_min_ratio(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		
		if (it == 128) {
			printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}

__device__ int locate_min_dj(TSimplexGPUs &smp) {
	int mejorz, z, y, top;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 0; z < smp.var_all; z++) {
		top = smp.top[z] - 1;
		if (smp.var_type[top] != 2) { // it is not an artificial variable
			apz = -smp.z[z];
			for (y = 0; y < smp.rest_fin; y++) {
				apz += smp.Cb[y*smp.mat_adv_row] * smp.matriz[y*smp.mat_adv_row + z]; // Cj
			}
			if (apz < 0 && apz < min_apz) {
				mejorz = z;
				min_apz = apz;
				// printf("MIn Zj-Cj: %f\n",  min_apz);
			}
		}
	}
	
	return mejorz;
}


__device__ int locate_min_ratio(TSimplexGPUs &smp, int zpos) {
	int mejory, y;
	double min_apy, qy, denom;

	mejory = -1;
	min_apy = MaxNReal;
	printf("qy:\t");
	for (y = 0; y < smp.rest_fin; y++) {
		denom = smp.matriz[y*smp.mat_adv_row + zpos];
		// printf("%.1f / %.1f ",  smp.Xb[y*smp.mat_adv_row], denom);
		// printf("Denominador: %f\n",  denom);
		if (denom > CasiCero_Simplex) {
			qy = smp.Xb[y*smp.mat_adv_row] / denom;
			printf(" (%.1f)\t",  qy);
			if (qy > -CasiCero_Simplex && qy < min_apy) {
				mejory = y;
				min_apy = qy;
			}
		} else {
			printf(" (NA)\t");
		}
	}
	// printf("Min Q: %f\n",  min_apy);
	return mejory;
	
}

__device__ bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, invPiv;
	int i, j, ipos, k;
	
	invPiv = 1 / smp.matriz[kfil * smp.mat_adv_row + jcol];
	
	ipos = kfil * smp.mat_adv_row;
	smp.Xb[kfil*smp.mat_adv_row] *= invPiv; // Modifico Xb
	for (j = 0; j < smp.var_all; j++) { // Modifico la k fila
		smp.matriz[ipos + j] *= invPiv;
	}
	smp.matriz[kfil * smp.mat_adv_row + jcol] = 1;
	
	for (i = 0; i < smp.rest_fin; i++) {
		if (i != kfil) {
			m = smp.matriz[i *smp.mat_adv_row + jcol];
			
			smp.Xb[i*smp.mat_adv_row] -= m*smp.Xb[kfil*smp.mat_adv_row]; // Modifico Xb
			for (j = 0; j < smp.var_all; j++) { // Modifico la Matriz
				if (j != jcol) {
					smp.matriz[i *smp.mat_adv_row + j] -= m * smp.matriz[kfil*smp.mat_adv_row + j]; 
				} else {
					smp.matriz[i*smp.mat_adv_row + j] = 0;
				}
			}
		}
	}
	
	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil*smp.mat_adv_row];
	smp.left[kfil*smp.mat_adv_row] = k;
	
	smp.Cb[kfil*smp.mat_adv_row] = smp.z[jcol];
	
	return true;
 }
 
__device__ void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.rest_fin + 6, smp.mat_adv_row);
	for(int i = 0; i < smp.rest_fin + 6; i++) {
		for(int j = 0; j < smp.mat_adv_row; j++) {
			printf("%.2f\t", (double) smp.tabloide[i*smp.mat_adv_row + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

__device__ void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	double bi, val;
	double min = 0;
	int varType;
	
	for(int i = 0; i < smp.var_x; i++) {
		bi = findVarXbValue(smp, i);
		val = bi + smp.inf[i];
		if (val != 0 || bi != 0) {
			printf("x%i = %.12f  (Xbi = %.12f)\n", findVarIndex(smp, i),  val, bi);
			min -= val * smp.z[i];
		}
	}
	
	for(int i = smp.var_x; i < smp.var_all; i++) {
		bi = findVarXbValue(smp, i);
		if (bi != 0) {
			varType = smp.var_type[i];
			if (varType == 1) {
				val = bi;
				printf("s%i = %.12f \n", findVarIndex(smp, i), val);
			} else {
				val = bi;
				printf("a - error%i = %.12f\n", findVarIndex(smp, i),  val);
			}
		}
	}
	
	printf("Z min = %.2f \n", -min);
	
}

__device__ double findVarXbValue(TSimplexGPUs &smp, int indx) {
	int lefti;
	for(int i = 0; i < smp.rest_fin; i++) {
		lefti = ((int) smp.left[i*smp.mat_adv_row]);
		if (indx == (lefti - 1)) {
			return smp.Xb[i*smp.mat_adv_row];
		}
	}
	return 0;
}

__device__ int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx  + 1;
	} else {
		for (int i = 0; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}
