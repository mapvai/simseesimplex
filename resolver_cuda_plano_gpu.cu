#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "toursimplexmgpus_final.h"

#define BLOCK_SIZE_P_I 64 //  = 2 * WARP SIZE
#define BLOCK_SIZE_GR 160 //  = 5 * WARP SIZE

const double CasiCero_Simplex = 1.0E-7;
// const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 1.0E+150; //100; //sqrt(MaxNReal);

const int MAX_VARS = 256; // Esto sera usado para pedir shared memory
const int MAX_RES = 256; // Esto sera usado para pedir shared memory


// 8 * 32 = 256
const int BLOCK_SIZE_E_1X = 32;  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy the arrange of the warp in the block is giving for a two-dimensional block of size (Dx, Dy),the thread ID of a thread of index (x, y) is (x + y Dx)
const int BLOCK_SIZE_E_1Y = 8;

const int BLOCK_SIZE_E_2X = 1;
const int BLOCK_SIZE_E_2Y = 1;

const int BLOCK_SIZE_E_3X = 1;
const int BLOCK_SIZE_E_3Y = 1;

// 8 * 32 = 128
const int BLOCK_SIZE_E_4X = 32;
const int BLOCK_SIZE_E_4Y = 4;

const int MAX_SIMPLEX_ITERATIONS = 128;


__device__ TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide);
__device__ void moverseASolFactible(TabloideGPUs &tabloide);
__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide);
__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide);

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide);

__device__ void locate_min_dj(TSimplexGPUs &smp, int &zpos) ;
__device__ void locate_min_ratio(TSimplexGPUs &smp, int zpos, int &qpos);
__device__ void intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);

__device__ void printStatus(TSimplexGPUs &smp);
__device__ void printResult(TSimplexGPUs &smp);
__device__ double findVarXbValue(TSimplexGPUs &smp, int indx);
__device__ int findVarIndex(TSimplexGPUs &smp, int indx);

__global__ void kernel_resolver_etapa_moverse_a_sol_factible(TDAOfSimplexGPUs simplex_array) {
	moverseASolFactible(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_restricciones_cota_sup(TDAOfSimplexGPUs simplex_array) {
	agregarRestriccionesCotaSup(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_variables_holgura_artificiales(TDAOfSimplexGPUs simplex_array) {
	agregarVariablesHolguraArtificiales(simplex_array[blockIdx.x]);
}

__global__ void kernel_resolver_simplex_big_m(TDAOfSimplexGPUs simplex_array) {
	resolver_simplex_big_m(simplex_array[blockIdx.x]);
}


extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	int largo, alto, n_vars, n_rest;
	
	int maxVars = 0;
	int maxRest = 0;
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][3];
		n_vars = (int) simplex_array[kTrayectoria][largo];
		n_rest = (int) simplex_array[kTrayectoria][largo + 1];
		alto = n_rest + 6;
		
		if (n_vars > maxVars) maxVars = n_vars;
		if (n_rest > maxRest) maxRest = n_rest;
		
		hipMemcpy(h_simplex_array[kTrayectoria], simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyHostToDevice);
		
	}

	hipMemcpy(d_simplex_array, h_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyHostToDevice);
	
	hipError_t err;
	
	// Ejecuto los kernels
	// const dim3 DimBlock_e1(maxRest, 1);
	const dim3 DimGrid_e1(NTrayectorias, 1);
	const dim3 DimBlock_e1(BLOCK_SIZE_E_1X, BLOCK_SIZE_E_1Y);
	kernel_resolver_etapa_moverse_a_sol_factible<<< DimGrid_e1, DimBlock_e1, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 1 error", hipGetErrorString(err));
	
	// const dim3 DimBlock_e2(maxVars, 1);
	const dim3 DimGrid_e2(NTrayectorias, 1);
	const dim3 DimBlock_e2(BLOCK_SIZE_E_2X, BLOCK_SIZE_E_2Y);
	kernel_resolver_etapa_agregar_restricciones_cota_sup<<< DimGrid_e2, DimBlock_e2, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 2 error", hipGetErrorString(err));
	
	// int cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques fijar_variables: ", cantBloques);
	// const dim3 DimGrid_e3(cantBloques, 1);
	// const dim3 DimGrid_e3(1, 1);
	// const dim3 DimBlock_e3(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e3(NTrayectorias, 1);
	const dim3 DimBlock_e3(BLOCK_SIZE_E_3X, BLOCK_SIZE_E_3Y);
	kernel_resolver_etapa_agregar_variables_holgura_artificiales<<< DimGrid_e3, DimBlock_e3, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 3 error", hipGetErrorString(err));
	
	// cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques enfilar_variables_libres: ", cantBloques);
	// const dim3 DimGrid_e4(cantBloques, 1);
	// const dim3 DimBlock_e4(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e4(NTrayectorias, 1);
	const dim3 DimBlock_e4(BLOCK_SIZE_E_4X, BLOCK_SIZE_E_4Y);
	kernel_resolver_simplex_big_m<<< DimGrid_e4, DimBlock_e4, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));
	
	hipMemcpy(h_simplex_array, d_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyDeviceToHost);

	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][3];
		alto = (int) simplex_array[kTrayectoria][largo + 1] + 6;
		hipMemcpy(simplex_array[kTrayectoria], h_simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyDeviceToHost);
	}

}

/**************************************************************************************************************************************************************************************************/

__device__ TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide) {
	TSimplexGPUs smp;
	
	smp.tabloide = tabloide;
	
	smp.var_x = (int) tabloide[0];
	smp.rest_ini = (int) tabloide[1];
	smp.mat_adv_row = (int) tabloide[2];
	smp.var_all = (int) tabloide[smp.mat_adv_row];
	smp.rest_fin = (int) tabloide[smp.mat_adv_row + 1];
	
	smp.z = &tabloide[4]; // funcion z, cantidad de variables, horizontal
    smp.flg_x = &tabloide[smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x
	
	smp.sup = &tabloide[2*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	smp.inf = &tabloide[3*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	
	smp.var_type = &tabloide[4*smp.mat_adv_row + 4]; // largo filas, horizontal
	
	smp.top = &tabloide[5*smp.mat_adv_row + 4]; // largo filas, horizontal
    smp.left = &tabloide[6*smp.mat_adv_row]; // largo restricciones finales, vertical
	
	smp.flg_y = &tabloide[6*smp.mat_adv_row + 1]; // 0 restriccion >=, 1 <=, 2 =, vertical
	
	
	smp.Cb = &tabloide[6*smp.mat_adv_row + 2]; // cantidad de restricciones, vertical
	smp.Xb = &tabloide[6*smp.mat_adv_row + 3]; // cantidad de restricciones, vertical
	
    smp.matriz = &tabloide[6*smp.mat_adv_row + 4]; 
	
	return smp;
}

__device__ void moverseASolFactible(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	/*
		Oprimizacion a desestructurarTabloide
		TSimplexGPUs smp;
		smp.var_x = (int) tabloide[0];
		smp.rest_ini = (int) tabloide[1];
		smp.mat_adv_row = (int) tabloide[2];
		smp.Xb = &tabloide[6*smp.mat_adv_row + 3];
		smp.flg_y = &tabloide[6*smp.mat_adv_row + 1];
		smp.matriz = &tabloide[6*smp.mat_adv_row + 4];
	*/
	
	
	for (int i = threadIdx.y; i < smp.rest_ini; i+= blockDim.y) {
		if (smp.Xb[i*smp.mat_adv_row] < 0) {
			if (threadIdx.y == 0) {
				smp.Xb[i*smp.mat_adv_row] *= -1;
				smp.flg_y[i*smp.mat_adv_row] = (smp.flg_y[i*smp.mat_adv_row] == 0) ? 1 : 2; // Move >= to <=
			}
			for (int j = threadIdx.x; j < smp.var_x; j+= blockDim.x) {
				smp.matriz[i*smp.mat_adv_row + j] *= -1;
			}
		}
	}
	
	//__syncthreads();
}

// No es paralelizable a nivel de bloque. La variable qrest es actualizado en cada bucle, y las celdas a las que se acceden dependen de ese valor, por las actualizaciones a la matriz se deben ejecutar en orden.
__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	
	int qrest = smp.rest_ini;
	for (int i = 0; i < smp.var_x; i++) {
		if (smp.flg_x[i] > 0) {
			smp.flg_y[(smp.rest_ini + i)*smp.mat_adv_row] = 1;
			smp.Xb[(smp.rest_ini + i)*smp.mat_adv_row] = smp.sup[i];
			// for (int j = 0; j < smp.var_x; j++) smp.matriz[qrest*smp.mat_adv_row + j] = (qrest == (j + smp.rest_ini))? 1 : 0; los 0's vienen desde la CPU 
			smp.matriz[qrest*smp.mat_adv_row + i] = 1; 
			qrest ++;
		}
	}
	// printf("Rest count %i / %i \n", smp.rest_fin, qrest);
	if (smp.rest_fin != qrest) printf("DISCREPANCIA EN LA CANTIDAD DE RESTRICCIONES FINAL\n");
	
}

// No es paralelizable a nivel de bloque. La variable var_count es actualizado en cada bucle, y las celdas a las que se acceden dependen de ese valor, por las actualizaciones a la matriz se deben ejecutar en orden.
__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloide(tabloide);
	
	int var_s, var_a, var_count;
	var_s = 0; var_a = 0; var_count = smp.var_x;
	
	for (int i = 0; i < smp.var_x; i++) {
		smp.var_type[i] = 0;
		smp.top[i] = i + 1;
	}
	
	/* Esto ya deberia venir completo desde la CPU
	// Completo con 0s la matriz
	for (int i = 0; i < smp.rest_fin; i++) {
		for (int j = var_count; j < smp.var_all; j++) {
			smp.matriz[i*smp.mat_adv_row + j] = 0;
		}
	}
	*/
	
	for (int i = 0; i < smp.rest_fin; i++) {
		if (smp.flg_y[i*smp.mat_adv_row] == 0) { // rest >=
			smp.matriz[i*smp.mat_adv_row + var_count] = -1;
			smp.matriz[i*smp.mat_adv_row + var_count +1] = 1;
			
			smp.var_type[var_count] = 1;
			smp.var_type[var_count +1] = 2;
			
			smp.z[var_count] = 0;
			smp.z[var_count +1] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.top[var_count + 1] = var_count + 2;
			smp.left[i*smp.mat_adv_row] = var_count + 2;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_a++; var_count += 2;
		} else if (smp.flg_y[i*smp.mat_adv_row] == 1) { // rest <=
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 1;
			
			smp.z[var_count] = 0;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = 0;
			
			var_a++; var_count ++;
		} else { // 2: rest =
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 2;
			
			smp.z[var_count] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_count ++;
		}
	}
	
	//printf("Var count %i / %i \n", smp.var_all, var_count);
	if (smp.var_all != var_count) printf("DISCREPANCIA EN LA CANTIDAD DE VARIABLES FINAL\n");
	
}

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide) {
	
	TSimplexGPUs simplex = desestructurarTabloide(tabloide);
	
	int it;
	__shared__ int zpos, qpos;
	
	if (threadIdx.x == 0 && threadIdx.y == 0) printf("resolver_simplex_big_m_final INT \n");
	
	// printStatus(simplex);
	it = 0;
	
	do {
		locate_min_dj(simplex, zpos);

		if (threadIdx.x == 0 && threadIdx.y == 0) printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			if (threadIdx.x == 0 && threadIdx.y == 0) {
				printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
				printResult(simplex);
			}
			return;
		}
		
		locate_min_ratio(simplex, zpos, qpos);
		
		if (threadIdx.x == 0 && threadIdx.y == 0) printf("%s %d \n", "qpos", qpos);
		
		if (qpos < 0) {
			if (threadIdx.x == 0 && threadIdx.y == 0) {
				printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
				printResult(simplex);
			}
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		__syncthreads();
		
		if (threadIdx.x == 0 && threadIdx.y == 0) printStatus(simplex);
		
		it++;
		
		if (it == MAX_SIMPLEX_ITERATIONS) {
			if (threadIdx.x == 0 && threadIdx.y == 0) printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}


// Cambiar el orden de recorrida y guardar el acumulador en la shared memory, luego usar reduccion para obtener el min
__device__ void locate_min_dj(TSimplexGPUs &smp, int &zpos) {
	__shared__  double apz_acc[MAX_VARS];
	__shared__  int apz_indx[MAX_VARS];
	
	int top;
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;

	for (unsigned int z = threadIdx.y; z < smp.var_all; z += blockDim.y) {
		top = smp.top[z] - 1;
		if (smp.var_type[top] != 2) { // it is not an artificial variable
			apz_acc[z] = -smp.z[z];
			apz_indx[z] = z;
			
			// Aca hay que aplicar una reduccion en vez de atomicAdd, e invertir el orden de lectura para que sea coalesced !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
			
			for (unsigned int y = threadIdx.x; y < smp.rest_fin; y+= blockDim.x) {
				atomicAdd(&apz_acc[z], smp.Cb[y*smp.mat_adv_row] * smp.matriz[y*smp.mat_adv_row + z]);
			}
		} else {
			if (threadIdx.x == 0) apz_indx[z] = -1; // Asi lo excluyo en la reduccion(apz_indx[z] < 0...)
		}
	}
	__syncthreads();
	
	
	// Condicion los hilos en el bloque deben ser mayor o igual que var_all sino hay que agregar un bucle mas para que se proesen el resto del los valores en la reduccion
	if (thd_indx == 0 && smp.var_all > (blockDim.x * blockDim.y)) printf("Condicion los hilos en el bloque deben ser mayor o igual que var_all \n");
	
	// Reduccion
	// Interleaved Addressing
	for (unsigned int s = 1; s < smp.var_all; s *= 2) {
		int index = 2 * s * thd_indx;
		if (index < smp.var_all) {
			if (apz_indx[index + s] >= 0 &&  apz_acc[index + s] < 0 && (apz_indx[index] < 0 || apz_acc[index + s] < apz_acc[index])) {			
				apz_indx[index]  = apz_indx[index + s];
				apz_acc[index] = apz_acc[index + s];
			}
		}
		__syncthreads();
	}
	
	/* Sequential Addressing
	for (unsigned int s = smp.var_all; s > 0; s >>= 1) {
		if (thd_indx < s) {
			if (apz_indx[index + s] >= 0 &&  apz_acc[index + s] && (apz_indx[index] < 0 || apz_acc[index + s] < apz_acc[index])) {			
				apz_indx[index]  = apz_indx[index + s];
				apz_acc[index] = apz_acc[index + s];
			}
		}
		__syncthreads();
	}
	*/
	
	// Escribir resultado
	if (thd_indx == 0) {
		if (apz_indx[0] >= 0 && apz_acc[0] < 0) {
			zpos = apz_indx[0];
		} else {
			zpos = -1;
		}
		// printf("MIn Zj-Cj: %f\n",  apz_acc[0]);
	}
	__syncthreads();
	
}


__device__ void locate_min_ratio(TSimplexGPUs &smp, int zpos, int &qpos) {	
	__shared__  double apy_acc[MAX_RES];
	__shared__  int apy_indx[MAX_RES];
	
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;
	
	double denom;
	
	// Condicion los hilos en el bloque deben ser mayor o igual que var_all sino hay que agregar un bucle mas para que se proesen el resto del los valores en la reduccion
	if (thd_indx == 0 && smp.rest_fin > (blockDim.x * blockDim.y)) printf("Condicion los hilos en el bloque deben ser mayor o igual que rest_fin \n");
	
	// Cargo los valores en memoria compartida
	if (thd_indx < smp.rest_fin)  {
		denom = smp.matriz[thd_indx*smp.mat_adv_row + zpos];
		if (denom > CasiCero_Simplex) {
			apy_indx[thd_indx] = thd_indx;
			apy_acc[thd_indx] = smp.Xb[thd_indx*smp.mat_adv_row] / denom;
		} else {	
			apy_indx[thd_indx] = -1; // Asi lo excluyo en la reduccion(apy_indx[z] < 0...)
		}
	}
	
	__syncthreads();
	
	// Reduccion
	// Interleaved Addressing
	for (unsigned int s = 1; s < smp.rest_fin; s *= 2) {
		int index = 2 * s * thd_indx;
		if (index < smp.rest_fin) {			
			if (apy_indx[index + s] >= 0 &&  apy_acc[index + s]  > -CasiCero_Simplex && (apy_indx[index] < 0 || apy_acc[index + s] < apy_acc[index])) {			
				apy_indx[index]  = apy_indx[index + s];
				apy_acc[index] = apy_acc[index + s];
			}
		}
		__syncthreads();
	}
	
	// Escribir resultado
	if (thd_indx == 0) {
		if (apy_indx[0] >= 0 && apy_acc[0]  > -CasiCero_Simplex) {
			qpos = apy_indx[0];
		} else {
			qpos = -1;
		}
		// printf("Min Q: %f\n",  apy_acc[0]);
	}
	__syncthreads();

}

__device__ void intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, invPiv;
	int i, j, ipos, k;
	
	invPiv = 1 / smp.matriz[kfil * smp.mat_adv_row + jcol];
	
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;
	int block_dim = blockDim.x * blockDim.y;
	
	ipos = kfil * smp.mat_adv_row;
	smp.Xb[kfil*smp.mat_adv_row] *= invPiv; // Modifico Xb
	for (j = thd_indx; j < smp.var_all; j+= block_dim) { // Modifico la fila k
		smp.matriz[ipos + j] *= invPiv;
	}
	
	for (i = thd_indx; j < smp.rest_fin; i+= block_dim) { // Modifico la columna j
		smp.matriz[i*smp.mat_adv_row + jcol] = 0; 
	}
	
	if (thd_indx == 0) smp.matriz[kfil * smp.mat_adv_row + jcol] = 1; // Modifico el pivote
	
	for (i = threadIdx.y; i < smp.rest_fin; i += blockDim.y) {
		if (i != kfil) {
			m = smp.matriz[i *smp.mat_adv_row + jcol];
			if (threadIdx.x == 0) {
				smp.Xb[i*smp.mat_adv_row] -= m*smp.Xb[kfil*smp.mat_adv_row]; // Modifico Xb
			}
			
			for (j = threadIdx.x; j < smp.var_all; j += blockDim.x) { // Modifico la Matriz
				if (j != jcol) {
					smp.matriz[i *smp.mat_adv_row + j] -= m * smp.matriz[kfil*smp.mat_adv_row + j]; // Aca esta actualizacion se hace coalesced y como es la mas importante podemos decir que el acceso es coalesced mayoritariamente
				} // else {
					// smp.matriz[i*smp.mat_adv_row + j] = 0; // Esto lo saque para afuera para generar menos divergencia
				// }
			}
		}
	}
	
	if (thd_indx == 0) {
		k = smp.top[jcol];
		smp.top[jcol] = smp.left[kfil*smp.mat_adv_row];
		smp.left[kfil*smp.mat_adv_row] = k;
		
		smp.Cb[kfil*smp.mat_adv_row] = smp.z[jcol];
	}
	
 }
 
__device__ void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.rest_fin + 6, smp.mat_adv_row);
	for(int i = 0; i < smp.rest_fin + 6; i++) {
		for(int j = 0; j < smp.mat_adv_row; j++) {
			printf("%.2f\t", (double) smp.tabloide[i*smp.mat_adv_row + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

__device__ void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	double bi, val;
	double min = 0;
	int varType;
	
	for(int i = 0; i < smp.var_x; i++) {
		bi = findVarXbValue(smp, i);
		val = bi + smp.inf[i];
		if (val != 0 || bi != 0) {
			printf("x%i = %.12f  (Xbi = %.12f)\n", findVarIndex(smp, i),  val, bi);
			min -= val * smp.z[i];
		}
	}
	
	for(int i = smp.var_x; i < smp.var_all; i++) {
		bi = findVarXbValue(smp, i);
		if (bi != 0) {
			varType = smp.var_type[i];
			if (varType == 1) {
				val = bi;
				printf("s%i = %.12f \n", findVarIndex(smp, i), val);
			} else {
				val = bi;
				printf("a - error%i = %.12f\n", findVarIndex(smp, i),  val);
			}
		}
	}
	
	printf("Z min = %.2f \n", -min);
	
}

__device__ double findVarXbValue(TSimplexGPUs &smp, int indx) {
	int lefti;
	for(int i = 0; i < smp.rest_fin; i++) {
		lefti = ((int) smp.left[i*smp.mat_adv_row]);
		if (indx == (lefti - 1)) {
			return smp.Xb[i*smp.mat_adv_row];
		}
	}
	return 0;
}

__device__ int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx  + 1;
	} else {
		for (int i = 0; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}
