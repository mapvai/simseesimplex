#include <stdio.h>
#include <math.h>

#include "toursimplexmgpus_final.h"


extern "C" void ini_mem(
	TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, 
	int NTrayectorias, int NEnteras, int NVariables, int NRestricciones, int cnt_varfijas, int cnt_RestriccionesRedundantes
) {
	
	int largo, alto;
	
	h_simplex_array = (TabloideGPUs*)malloc(NTrayectorias*sizeof(TabloideGPUs));

	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		
		largo = (int) h_simplex_array[kTrayectoria][3];
		alto = (int) h_simplex_array[kTrayectoria][largo + 1] + 6;
		hipMalloc(&h_simplex_array[kTrayectoria], largo*alto*sizeof(double));
	
	}

 	hipMalloc(&d_simplex_array, NTrayectorias*sizeof(TabloideGPUs));
	hipMemcpy(d_simplex_array, h_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyHostToDevice);
 
}
