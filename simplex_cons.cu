#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "toursimplexmgpus_final.h"

#define BLOCK_SIZE_P_I 64 //  = 2 * WARP SIZE
#define BLOCK_SIZE_GR 160 //  = 5 * WARP SIZE

__constant__ double CasiCero_Simplex = 1.0E-7;
// const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

__constant__ double M = 1.0E+150; // 1.0E+150; 100; //sqrt(MaxNReal);
const double eMe = 1.0E+150; // 1.0E+150; 100;

const int MAX_VARS = 256; // Esto sera usado para pedir shared memory
const int MAX_RES = 256; // Esto sera usado para pedir shared memory


// 8 * 32 = 256
const int BLOCK_SIZE_E_1X = 32;  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy the arrange of the warp in the block is giving for a two-dimensional block of size (Dx, Dy),the thread ID of a thread of index (x, y) is (x + y Dx)
const int BLOCK_SIZE_E_1Y = 8;

const int BLOCK_SIZE_E_2X = 1;
const int BLOCK_SIZE_E_2Y = 1;

const int BLOCK_SIZE_E_3X = 1;
const int BLOCK_SIZE_E_3Y = 1;

// 8 * 32 = 128
const int BLOCK_SIZE_E_4X = 32;
const int BLOCK_SIZE_E_4Y = 4;

const int MAX_SIMPLEX_ITERATIONS = 128; // 128;

__device__ TSimplexGPUs desestructurarTabloideDeb(TabloideGPUs &tabloide);
__device__ void moverseASolFactible(TabloideGPUs &tabloide);
__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide);
__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide);

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide);

__device__ void locate_min_dj(TSimplexGPUs &smp, int &zpos) ;
__device__ void locate_min_ratio(TSimplexGPUs &smp, int zpos, int &qpos);
__device__ void intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);

__device__ void printStatusDev(TSimplexGPUs &smp);
__device__ void printResultDev(TSimplexGPUs &smp);
__device__ double findVarXbValueDev(TSimplexGPUs &smp, int indx);
__device__ int findVarIndexDev(TSimplexGPUs &smp, int indx);

__global__ void kernel_resolver_etapa_moverse_a_sol_factible(TDAOfSimplexGPUs simplex_array) {
	moverseASolFactible(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_restricciones_cota_sup(TDAOfSimplexGPUs simplex_array) {
	agregarRestriccionesCotaSup(simplex_array[blockIdx.x]);
} 

__global__ void kernel_resolver_etapa_agregar_variables_holgura_artificiales(TDAOfSimplexGPUs simplex_array) {
	agregarVariablesHolguraArtificiales(simplex_array[blockIdx.x]);
}

__global__ void kernel_resolver_simplex_big_m(TDAOfSimplexGPUs simplex_array) {
	resolver_simplex_big_m(simplex_array[blockIdx.x]);
}

__global__ void kernel_test(TDAOfSimplexGPUs d_simplex_array) {
	printf("Hello World from GPU!\n");
	printf("%.2f\n", d_simplex_array[0][2]);
	printf("%.2f\n", d_simplex_array[0][16]);
	printf("End test GPU!\n");
}

void printResult(TSimplexGPUs &smp);
void printStatus(TSimplexGPUs &smp);
int findVarIndex(TSimplexGPUs &smp, int indx);
double findVarXbValue(TSimplexGPUs &smp, int indx);
TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide);

void ini_mem(TDAOfSimplexGPUs simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias);
void free_mem(TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias);

void resolver_ejemplo1();
void resolver_ejemplo2trasnform();


int main() {
	printf("Hello World from CPU!\n");
    
	// resolver_ejemplo1();
	
	resolver_ejemplo2trasnform();
	
    return 0;
}

void resolver_ejemplo1() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9
		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol SIMSEE: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado, z min = -21.7
		
=> cambio variable para las cotas inferiores xc = x + cota inf => x = xc - cota inf => Sol xc: x1 = 0, x2 = 0.7, x3 = 2.1
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
		
=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s5 ≥ 0
	
	RESULTADO OUR SIMPLEX: x1 = 0.7, xc2 = 0 => x2 = 0 - 6 = -6, xc3 = 2.8 => x3 = 2.8 - 5 = -2.2 Verificado, da tambien z min = -21.7
*/	
	TDAOfSimplexGPUs d_simplex_array; 
	TDAOfSimplexGPUs h_simplex_array;
	hipError_t err;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		3, 3, 15, 0, -1, -3, -2, 0, -eMe, -eMe, 0, -eMe, 0, 0, 0, 
		11, 6, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 12, 12, 10, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, -6, -5, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 1, 2, 2, 1, 2, 1, 1, 1, 
		0, 0, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 
		5, 0, -eMe, 0.5, 1, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 
		6, 2, -eMe, 0.7, 1, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 
		8, 0, -eMe, 2.1, -1, 0, 1, 0, 0, 0, -1, 1, 0, 0, 0, 
		9, 1, 0, 12, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 
		10, 1, 0, 12, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 
		11, 1, 0, 10, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 
	};
	
	TabloideGPUs tabloide = (double*)&tabl;
	
	int NTrayectorias = 1;
	TDAOfSimplexGPUs simplex_array = (TabloideGPUs*)malloc(sizeof(TabloideGPUs));
	simplex_array[0] = tabloide;
	
	ini_mem(simplex_array, d_simplex_array, h_simplex_array, 1);
	
	const dim3 DimGrid_e4(NTrayectorias, 1);
	const dim3 DimBlock_e4(BLOCK_SIZE_E_4X, BLOCK_SIZE_E_4Y);
	kernel_resolver_simplex_big_m<<< DimGrid_e4, DimBlock_e4, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));
	
	// ...
	
	free_mem(d_simplex_array, h_simplex_array, NTrayectorias);
}

void resolver_ejemplo2trasnform() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9
		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol SIMSEE: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado, z min = -21.7
		
=> cambio variable para las cotas inferiores xc = x + cota inf => x = xc - cota inf => Sol xc: x1 = 0, x2 = 0.7, x3 = 2.1
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
*/	
	TDAOfSimplexGPUs d_simplex_array; 
	TDAOfSimplexGPUs h_simplex_array;
	hipError_t err;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		3, 3, 15, 0, -1, -3, -2, 0, 0, 0, 0, 0, 0, 0, 0, 
		11, 6, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 12, 12, 10, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, -6, -5, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0.5, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 2, 0, 0.7, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 2.1, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
	};
	
	TabloideGPUs tabloide = (double*)&tabl;
	
	/*
	=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s5 ≥ 0
	
	RESULTADO OUR SIMPLEX: x1 = 0.7, xc2 = 0 => x2 = 0 - 6 = -6, xc3 = 2.8 => x3 = 2.8 - 5 = -2.2 Verificado, da tambien z min = -21.7
	*/
	
	int NTrayectorias = 1;
	TDAOfSimplexGPUs simplex_array = (TabloideGPUs*)malloc(NTrayectorias*sizeof(TabloideGPUs));
	simplex_array[0] = tabloide;
	
	TSimplexGPUs smp = desestructurarTabloide(simplex_array[0]);
	
	printStatus(smp);
	
	ini_mem(simplex_array, d_simplex_array, h_simplex_array, NTrayectorias);
	
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 1 error", hipGetErrorString(err));
	
	// Ejecuto los kernels
	const dim3 DimGrid_e1(NTrayectorias, 1);
	const dim3 DimBlock_e1(BLOCK_SIZE_E_1X, BLOCK_SIZE_E_1Y);
	kernel_resolver_etapa_moverse_a_sol_factible<<< DimGrid_e1, DimBlock_e1, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 1 error", hipGetErrorString(err));
	
	const dim3 DimGrid_e2(NTrayectorias, 1);
	const dim3 DimBlock_e2(BLOCK_SIZE_E_2X, BLOCK_SIZE_E_2Y);
	kernel_resolver_etapa_agregar_restricciones_cota_sup<<< DimGrid_e2, DimBlock_e2, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 2 error", hipGetErrorString(err));
	
	const dim3 DimGrid_e3(NTrayectorias, 1);
	const dim3 DimBlock_e3(BLOCK_SIZE_E_3X, BLOCK_SIZE_E_3Y);
	kernel_resolver_etapa_agregar_variables_holgura_artificiales<<< DimGrid_e3, DimBlock_e3, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 3 error", hipGetErrorString(err));
	
	const dim3 DimGrid_e4(NTrayectorias, 1);
	const dim3 DimBlock_e4(BLOCK_SIZE_E_4X, BLOCK_SIZE_E_4Y);
	kernel_resolver_simplex_big_m<<< DimGrid_e4, DimBlock_e4, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));

	int largo, alto;
	
	// hipMemcpy(h_simplex_array, d_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyDeviceToHost);
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][2];
		alto = (int) simplex_array[kTrayectoria][largo + 1] + 6;
		hipMemcpy(simplex_array[kTrayectoria], h_simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyDeviceToHost);
		err = hipGetLastError(); 
		if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));
	}
	
	smp = desestructurarTabloide(simplex_array[0]);
	
	printStatus(smp);
	
	free_mem(d_simplex_array, h_simplex_array, NTrayectorias);
	free(h_simplex_array);
	free(tabloide);

}

void ini_mem(TDAOfSimplexGPUs simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	
	int largo, alto;
	
	h_simplex_array = (TabloideGPUs*)malloc(NTrayectorias*sizeof(TabloideGPUs));
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][2];
		alto = (int) simplex_array[kTrayectoria][largo + 1] + 6;
		
		hipMalloc(&h_simplex_array[kTrayectoria], largo*alto*sizeof(double));
		hipMemcpy(h_simplex_array[kTrayectoria], simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyHostToDevice);
	}
	
	hipMalloc(&d_simplex_array, NTrayectorias*sizeof(TabloideGPUs));
	hipMemcpy(d_simplex_array, h_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyHostToDevice);
	
	hipError_t err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA ini_mem inside error", hipGetErrorString(err));
	
}

void free_mem(TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		hipFree(h_simplex_array[kTrayectoria]);
	}
	
	hipFree(d_simplex_array);
}


void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	
	int largo, alto;
	
	ini_mem(simplex_array, d_simplex_array, h_simplex_array, NTrayectorias);
	
	hipError_t err;
	
	// Ejecuto los kernels
	// const dim3 DimBlock_e1(maxRest, 1);
	const dim3 DimGrid_e1(NTrayectorias, 1);
	const dim3 DimBlock_e1(BLOCK_SIZE_E_1X, BLOCK_SIZE_E_1Y);
	kernel_resolver_etapa_moverse_a_sol_factible<<< DimGrid_e1, DimBlock_e1, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 1 error", hipGetErrorString(err));
	
	// const dim3 DimBlock_e2(maxVars, 1);
	const dim3 DimGrid_e2(NTrayectorias, 1);
	const dim3 DimBlock_e2(BLOCK_SIZE_E_2X, BLOCK_SIZE_E_2Y);
	kernel_resolver_etapa_agregar_restricciones_cota_sup<<< DimGrid_e2, DimBlock_e2, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 2 error", hipGetErrorString(err));
	
	// int cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques fijar_variables: ", cantBloques);
	// const dim3 DimGrid_e3(cantBloques, 1);
	// const dim3 DimGrid_e3(1, 1);
	// const dim3 DimBlock_e3(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e3(NTrayectorias, 1);
	const dim3 DimBlock_e3(BLOCK_SIZE_E_3X, BLOCK_SIZE_E_3Y);
	kernel_resolver_etapa_agregar_variables_holgura_artificiales<<< DimGrid_e3, DimBlock_e3, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 3 error", hipGetErrorString(err));
	
	// cantBloques = ceil((float)NTrayectorias / (float)BLOCK_SIZE_GR);
	// printf("%s: %d\n", "cantBloques enfilar_variables_libres: ", cantBloques);
	// const dim3 DimGrid_e4(cantBloques, 1);
	// const dim3 DimBlock_e4(BLOCK_SIZE_GR, 1);
	const dim3 DimGrid_e4(NTrayectorias, 1);
	const dim3 DimBlock_e4(BLOCK_SIZE_E_4X, BLOCK_SIZE_E_4Y);
	kernel_resolver_simplex_big_m<<< DimGrid_e4, DimBlock_e4, 0, 0 >>>(d_simplex_array);
	hipDeviceSynchronize();
	err = hipGetLastError(); 
	if (err != hipSuccess) printf("%s: %s\n", "CUDA 4 error", hipGetErrorString(err));
	
	hipMemcpy(h_simplex_array, d_simplex_array, NTrayectorias*sizeof(TabloideGPUs), hipMemcpyDeviceToHost);

	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		largo = (int) simplex_array[kTrayectoria][2];
		alto = (int) simplex_array[kTrayectoria][largo + 1] + 6;
		hipMemcpy(simplex_array[kTrayectoria], h_simplex_array[kTrayectoria], largo*alto*sizeof(double), hipMemcpyDeviceToHost);
	}

}

/**************************************************************************************************************************************************************************************************/

__device__ TSimplexGPUs desestructurarTabloideDeb(TabloideGPUs &tabloide) {
	TSimplexGPUs smp;
	
	smp.tabloide = tabloide;
	
	smp.var_x = (int) tabloide[0];
	smp.rest_ini = (int) tabloide[1];
	smp.mat_adv_row = (int) tabloide[2];
	smp.var_all = (int) tabloide[smp.mat_adv_row];
	smp.rest_fin = (int) tabloide[smp.mat_adv_row + 1];
	
	smp.z = &tabloide[4]; // funcion z, cantidad de variables, horizontal
    smp.flg_x = &tabloide[smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x
	
	smp.sup = &tabloide[2*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	smp.inf = &tabloide[3*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	
	smp.var_type = &tabloide[4*smp.mat_adv_row + 4]; // largo filas, horizontal
	
	smp.top = &tabloide[5*smp.mat_adv_row + 4]; // largo filas, horizontal
    smp.left = &tabloide[6*smp.mat_adv_row]; // largo restricciones finales, vertical
	
	smp.flg_y = &tabloide[6*smp.mat_adv_row + 1]; // 0 restriccion >=, 1 <=, 2 =, vertical
	
	
	smp.Cb = &tabloide[6*smp.mat_adv_row + 2]; // cantidad de restricciones, vertical
	smp.Xb = &tabloide[6*smp.mat_adv_row + 3]; // cantidad de restricciones, vertical
	
    smp.matriz = &tabloide[6*smp.mat_adv_row + 4]; 
	
	return smp;
}

TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide) {
	TSimplexGPUs smp;
	
	smp.tabloide = tabloide;
	
	smp.var_x = (int) tabloide[0];
	smp.rest_ini = (int) tabloide[1];
	smp.mat_adv_row = (int) tabloide[2];
	smp.var_all = (int) tabloide[smp.mat_adv_row];
	smp.rest_fin = (int) tabloide[smp.mat_adv_row + 1];
	
	smp.z = &tabloide[4]; // funcion z, cantidad de variables, horizontal
    smp.flg_x = &tabloide[smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x
	
	smp.sup = &tabloide[2*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	smp.inf = &tabloide[3*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	
	smp.var_type = &tabloide[4*smp.mat_adv_row + 4]; // largo filas, horizontal
	
	smp.top = &tabloide[5*smp.mat_adv_row + 4]; // largo filas, horizontal
    smp.left = &tabloide[6*smp.mat_adv_row]; // largo restricciones finales, vertical
	
	smp.flg_y = &tabloide[6*smp.mat_adv_row + 1]; // 0 restriccion >=, 1 <=, 2 =, vertical
	
	
	smp.Cb = &tabloide[6*smp.mat_adv_row + 2]; // cantidad de restricciones, vertical
	smp.Xb = &tabloide[6*smp.mat_adv_row + 3]; // cantidad de restricciones, vertical
	
    smp.matriz = &tabloide[6*smp.mat_adv_row + 4]; 
	
	return smp;
}

__device__ void moverseASolFactible(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloideDeb(tabloide);
	
	if (threadIdx.x == 0 && threadIdx.y == 0) printStatusDev(smp);
	
	/*
		Oprimizacion a desestructurarTabloideDeb
		TSimplexGPUs smp;
		smp.var_x = (int) tabloide[0];
		smp.rest_ini = (int) tabloide[1];
		smp.mat_adv_row = (int) tabloide[2];
		smp.Xb = &tabloide[6*smp.mat_adv_row + 3];
		smp.flg_y = &tabloide[6*smp.mat_adv_row + 1];
		smp.matriz = &tabloide[6*smp.mat_adv_row + 4];
	*/
	
	for (int i = threadIdx.y; i < smp.rest_ini; i += blockDim.y) {
	// if (threadIdx.x < 4) printf("%.2f kk, x:%i, y:%i\n", smp.Xb[i*smp.mat_adv_row], threadIdx.x, threadIdx.y);	
	if (smp.Xb[i*smp.mat_adv_row] < 0) {
			if (threadIdx.x == 0) {
				smp.Xb[i*smp.mat_adv_row] *= -1;	
				smp.flg_y[i*smp.mat_adv_row] = (smp.flg_y[i*smp.mat_adv_row] == 0) ? 1 : 2; // Move >= to <=
			}
			for (int j = threadIdx.x; j < smp.var_x; j += blockDim.x) {
				smp.matriz[i*smp.mat_adv_row + j] *= -1;
			}
		}
	}
	
	__syncthreads();
	
	// if (threadIdx.x == 0 && threadIdx.y == 0) printStatusDev(smp);
}

// No es paralelizable a nivel de bloque. La variable qrest es actualizado en cada bucle, y las celdas a las que se acceden dependen de ese valor, por las actualizaciones a la matriz se deben ejecutar en orden.
__device__ void agregarRestriccionesCotaSup(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloideDeb(tabloide);
	
	int qrest = smp.rest_ini;
	for (int i = 0; i < smp.var_x; i++) {
		if (smp.flg_x[i] > 0) {
			smp.flg_y[(smp.rest_ini + i)*smp.mat_adv_row] = 1;
			smp.Xb[(smp.rest_ini + i)*smp.mat_adv_row] = smp.sup[i];
			// for (int j = 0; j < smp.var_x; j++) smp.matriz[qrest*smp.mat_adv_row + j] = (qrest == (j + smp.rest_ini))? 1 : 0; los 0's vienen desde la CPU 
			smp.matriz[qrest*smp.mat_adv_row + i] = 1; 
			qrest ++;
		}
	}
	// printf("Rest count %i / %i \n", smp.rest_fin, qrest);
	if (smp.rest_fin != qrest) printf("DISCREPANCIA EN LA CANTIDAD DE RESTRICCIONES FINAL\n");
	
	// printStatusDev(smp);
	
}

// No es paralelizable a nivel de bloque. La variable var_count es actualizado en cada bucle, y las celdas a las que se acceden dependen de ese valor, por las actualizaciones a la matriz se deben ejecutar en orden.
__device__ void agregarVariablesHolguraArtificiales(TabloideGPUs &tabloide) {
	
	TSimplexGPUs smp = desestructurarTabloideDeb(tabloide);
	
	int var_s, var_a, var_count;
	var_s = 0; var_a = 0; var_count = smp.var_x;
	
	for (int i = 0; i < smp.var_x; i++) {
		smp.var_type[i] = 0;
		smp.top[i] = i + 1;
	}
	
	/* Esto ya deberia venir completo desde la CPU
	// Completo con 0s la matriz
	for (int i = 0; i < smp.rest_fin; i++) {
		for (int j = var_count; j < smp.var_all; j++) {
			smp.matriz[i*smp.mat_adv_row + j] = 0;
		}
	}
	*/
	
	for (int i = 0; i < smp.rest_fin; i++) {
		if (smp.flg_y[i*smp.mat_adv_row] == 0) { // rest >=
			smp.matriz[i*smp.mat_adv_row + var_count] = -1;
			smp.matriz[i*smp.mat_adv_row + var_count +1] = 1;
			
			smp.var_type[var_count] = 1;
			smp.var_type[var_count +1] = 2;
			
			smp.z[var_count] = 0;
			smp.z[var_count +1] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.top[var_count + 1] = var_count + 2;
			smp.left[i*smp.mat_adv_row] = var_count + 2;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_a++; var_count += 2;
		} else if (smp.flg_y[i*smp.mat_adv_row] == 1) { // rest <=
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 1;
			
			smp.z[var_count] = 0;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = 0;
			
			var_a++; var_count ++;
		} else { // 2: rest =
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 2;
			
			smp.z[var_count] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_count ++;
		}
	}
	
	//printf("Var count %i / %i \n", smp.var_all, var_count);
	if (smp.var_all != var_count) printf("DISCREPANCIA EN LA CANTIDAD DE VARIABLES FINAL\n");
	
	printStatusDev(smp);
	
}

__device__ void resolver_simplex_big_m(TabloideGPUs &tabloide) {
	
	TSimplexGPUs simplex = desestructurarTabloideDeb(tabloide);
	
	int it;
	__shared__ int zpos, qpos;
	
	if (threadIdx.x == 0 && threadIdx.y == 0) printf("resolver_simplex_big_m_final INT \n");
	
	// printStatusDev(simplex);
	it = 0;
	
	do {
		locate_min_dj(simplex, zpos);

		if (threadIdx.x == 0 && threadIdx.y == 0) printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			if (threadIdx.x == 0 && threadIdx.y == 0) {
				printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
				printResultDev(simplex);
			}
			return;
		}
		
		locate_min_ratio(simplex, zpos, qpos);
		
		if (threadIdx.x == 0 && threadIdx.y == 0) printf("%s %d \n", "qpos", qpos);
		
		if (qpos < 0) {
			if (threadIdx.x == 0 && threadIdx.y == 0) {
				printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
				printResultDev(simplex);
			}
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		if (threadIdx.x == 0 && threadIdx.y == 0) printStatusDev(simplex);
		__syncthreads();
		
		it++;
		
		if (it == MAX_SIMPLEX_ITERATIONS) {
			if (threadIdx.x == 0 && threadIdx.y == 0) printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}


// Cambiar el orden de recorrida y guardar el acumulador en la shared memory, luego usar reduccion para obtener el min
__device__ void locate_min_dj(TSimplexGPUs &smp, int &zpos) {
	__shared__  double apz_acc[MAX_VARS];
	__shared__  double apz_acc_mat[BLOCK_SIZE_E_4Y][BLOCK_SIZE_E_4X];
	__shared__  int apz_indx[MAX_VARS];
	
	int top;
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;
	
	// inicializo Cj - Zj con -Zj
	for (unsigned int x = thd_indx; x < smp.var_all; x += blockDim.y*blockDim.x){
		apz_indx[x] = x;
	}
	for (unsigned int x = thd_indx; x < smp.var_all; x += blockDim.y*blockDim.x){
		top = smp.top[x] - 1;
		apz_acc[x] = -smp.z[x];
		if (smp.var_type[top] == 2) { // it is not an artificial variable
			apz_indx[x] = -1; // Asi lo excluyo en la reduccion(apz_indx[z] < 0...)
		}
	}
	__syncthreads();
	
	// Print vector -Z, remover luego
	if (thd_indx == 0) {
		printf("-Z: ");
		for (unsigned int x = 0; x < smp.var_all; x ++){
			printf("%i %.2f\t",  apz_indx[x], apz_acc[x]);
		}
		printf("\n");
	}
	
	// Calcular Cj - Zj y acumularlo en apz_acc
	unsigned int y = threadIdx.y;
	unsigned int x = threadIdx.x;
	cargar_tilt:
	if (y < smp.rest_fin && x < smp.var_all) apz_acc_mat[threadIdx.y][threadIdx.x] = smp.Cb[y*smp.mat_adv_row] * smp.matriz[y*smp.mat_adv_row + x];
	
	__syncthreads();
	
	/* Imprimir tile antes de la reduccion
	if (thd_indx == 0) {
		for (unsigned int i = 0; i < blockDim.y && i < smp.rest_fin; i ++){
			printf("fila %i:\t", i);
			for (unsigned int j = 0; j < blockDim.x && j < smp.var_all; j ++){	
				printf("%.2f\t",  apz_acc_mat[i][j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	*/
	
	// Reduccion Interleaved Addressing en el bloque en shared memory
	for (unsigned int s = 1; s < blockDim.y; s *= 2) {
		int index = 2 * s * threadIdx.y;
		if (index < blockDim.y && (y + index + s) < smp.rest_fin && x < smp.var_all) {
			apz_acc_mat[index][x] += apz_acc_mat[index + s][threadIdx.x];
		}
		__syncthreads();
	}
	
	if (threadIdx.y == 0 && x < smp.var_all) apz_acc[x] += apz_acc_mat[0][threadIdx.x]; // Accumulo el resultado parcial en el vector final
	
	/* Imprimir tile despues de la reduccion
	if (thd_indx == 0) {
		for (unsigned int y = 0; y < blockDim.y && y < smp.rest_fin; y ++){
			printf("fila %i:\t",  y);
			for (unsigned int x = 0; x < blockDim.x && x < smp.var_all; x ++){	
				printf("%.2f\t", apz_acc_mat[y][x]);
			}
			printf("\n");
		}
		printf("\n");
	}
	*/
	
	__syncthreads();
	
	x += blockDim.x;
	if ((x - threadIdx.x) < smp.var_all) {
		goto cargar_tilt;
	}
	y += blockDim.y;
	if ((y - threadIdx.y) < smp.rest_fin) {
		x = threadIdx.x;
		goto cargar_tilt;
	}
	
	/* // Este codigo intrenta hacer el mismo trabajo que "Calcular Cj - Zj y acumularlo en apz_acc" pero genera un deadlock al agregar un __syncthreads() despues los bucles
	   // Creo que es pq no todos los hilos arrivan al punto de __syncthreads()
	for (unsigned int y = threadIdx.y; y < smp.rest_fin; y += blockDim.y) {
		for (unsigned int x = threadIdx.x; x < smp.var_all; x += blockDim.x) {
			// printf("[%i, %i] mete [%i, %i] saltando %i\n",  threadIdx.y, threadIdx.x, y, x, smp.mat_adv_row);
			apz_acc_mat[threadIdx.y][threadIdx.x] = smp.matriz[y*smp.mat_adv_row + x];
			
			__syncthreads();
			if (thd_indx == 0) {
				for (unsigned int y = 0; y < blockDim.y && y < smp.rest_fin; y ++){
					printf("fila %i:\t",  y);
					for (unsigned int x = 0; x < blockDim.x && x < smp.var_all; x ++){	
						printf("%.2f\t",  apz_acc_mat[y][x]);
					}
					printf("\n");
				}
				printf("\n");
			}
			__syncthreads();
			// Reduccion Interleaved Addressing en el bloque en shared memory
			for (unsigned int s = 1; s < blockDim.y; s *= 2) {
				int index = 2 * s * threadIdx.y;
				if (index < blockDim.y && (y + index + s) < smp.rest_fin && x < smp.var_all) {
					apz_acc_mat[index][x] += apz_acc_mat[index + s][threadIdx.x];
				}
				__syncthreads();
			}
			
			__syncthreads();
			
			if (threadIdx.y == 0 && x < smp.var_all) apz_acc[x] += apz_acc_mat[0][threadIdx.x]; // Accumulo el resultado parcial en el vector final
			
			if (thd_indx == 0) {
				for (unsigned int y = 0; y < blockDim.y && y < smp.rest_fin; y ++){
					printf("fila %i:\t",  y);
					for (unsigned int x = 0; x < blockDim.x && x < smp.var_all; x ++){	
						printf("%.2f\t", apz_acc_mat[y][x]);
					}
					printf("\n");
				}
				printf("\n");
			}
			__syncthreads();
		}
	}
	*/
	
	// Imprimir Vector Cj - Zj final, eliminar luego
	if (thd_indx == 0) {
		printf("Cj - Zj final: ");
		for (unsigned int x = 0; x < smp.var_all; x ++) {
			printf("%.2f\t", apz_acc[x]);
		}
		printf("\n");
	}
	
	// Condicion los hilos en el bloque deben ser mayor o igual que var_all sino hay que agregar un bucle mas para que se procesen el resto del los valores en la reduccion
	if (thd_indx == 0 && smp.var_all > (blockDim.x * blockDim.y)) printf("Condicion los hilos en el bloque deben ser mayor o igual que var_all \n");
	
	__syncthreads();
	
	// Reduccion Interleaved Addressing
	for (unsigned int s = 1; s < smp.var_all; s *= 2) {
		int index = 2 * s * thd_indx;
		
		// printf("[%i <- %i]\t", index, index + s);
		// if (index < smp.var_all && (index + s) < smp.var_all) {
		if ((index + s) < smp.var_all) {	
			if (apz_indx[index + s] >= 0 &&  apz_acc[index + s] < 0 && (apz_indx[index] < 0 || apz_acc[index + s] < apz_acc[index])) {			
				apz_indx[index]  = apz_indx[index + s];
				apz_acc[index] = apz_acc[index + s];
				// printf("inside l %.2f at %i\t", apz_acc[index + s], apz_indx[index + s] );
			}
		}
		__syncthreads();
	}
	
	/* Sequential Addressing, hacer algun test de performace para evaluar la mejora que esto conlleva
	for (unsigned int s = smp.var_all/2; s > 0; s >>= 1) {
		if (thd_indx < s) {
			if (apz_indx[thd_indx + s] >= 0 &&  apz_acc[thd_indx + s] && (apz_indx[thd_indx] < 0 || apz_acc[thd_indx + s] < apz_acc[thd_indx])) {			
				apz_indx[thd_indx]  = apz_indx[thd_indx + s];
				apz_acc[thd_indx] = apz_acc[thd_indx + s];
			}
		}
		__syncthreads();
	}
	*/
	
	__syncthreads();
	
	// Escribir resultado
	if (thd_indx == 0) {
		if (apz_indx[0] >= 0 && apz_acc[0] < 0) {
			zpos = apz_indx[0];
		} else {
			zpos = -1;
		}
		printf("\nMin Cj-Zj: ind %i val %.2f\n",  apz_indx[0], apz_acc[0]); // Imprimir resultado, eliminar luego
	}
	__syncthreads();
	
	return;
	
}


__device__ void locate_min_ratio(TSimplexGPUs &smp, int zpos, int &qpos) {	
	__shared__  double apy_acc[MAX_RES];
	__shared__  int apy_indx[MAX_RES];
	
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;
	
	double denom;
	
	// Condicion los hilos en el bloque deben ser mayor o igual que var_all sino hay que agregar un bucle mas para que se proesen el resto del los valores en la reduccion
	if (thd_indx == 0 && smp.rest_fin * 2 > (blockDim.x * blockDim.y)) printf("Condicion los hilos en el bloque deben ser mayor o igual que 2 x rest_fin \n");
	
	// Cargo los valores en memoria compartida
	if (thd_indx < smp.rest_fin)  {
		denom = smp.matriz[thd_indx*smp.mat_adv_row + zpos];
		if (denom > CasiCero_Simplex) {
			apy_indx[thd_indx] = thd_indx;
			apy_acc[thd_indx] = smp.Xb[thd_indx*smp.mat_adv_row] / denom;
		} else {	
			apy_indx[thd_indx] = -1; // Asi lo excluyo en la reduccion(apy_indx[z] < 0...)
		}
		// printf("%i: %.1f / %.1f (%i)\t", thd_indx, smp.Xb[thd_indx*smp.mat_adv_row], denom, apy_indx[thd_indx]);
	}
	
	/*
	__syncthreads();
	if (thd_indx == 0) {
		printf("\nantes:\n");
		for (int l = 0; l < smp.rest_fin; l++) {
			printf("%i: %.2f at %i\t", l, apy_acc[l], apy_indx[l]);
		}
		printf("\n");
	}
	*/
	__syncthreads();
	
	// Reduccion
	// Interleaved Addressing
	for (unsigned int s = 1; s < smp.rest_fin; s *= 2) {
		int index = 2 * s * thd_indx;
		if ((index + s) < smp.rest_fin) {
			// printf("%i: %i, %.2f, %i, %.2f, %.2f\t", index, apy_indx[index + s], apy_acc[index + s], apy_indx[index], apy_acc[index + s], apy_acc[index]);			
			if (apy_indx[index + s] >= 0 &&  apy_acc[index + s]  > -CasiCero_Simplex && (apy_indx[index] < 0 || apy_acc[index + s] < apy_acc[index])) {			
				apy_indx[index] = apy_indx[index + s];
				apy_acc[index] = apy_acc[index + s];
				// printf("\nnew min %.2f at %i\n", apy_acc[index], index);
			}
		}
		__syncthreads();
	}
	
	/*
	__syncthreads();
	if (thd_indx == 0) {
		printf("\ndespues:\n");
		for (int l = 0; l < smp.rest_fin; l++) {
			printf("%i: %.2f at %i\t", l, apy_acc[l], apy_indx[l]);
		}
		printf("\n");
	}
	*/
	__syncthreads();
	
	// Escribir resultado
	if (thd_indx == 0) {
		if (apy_indx[0] >= 0 && apy_acc[0]  > -CasiCero_Simplex) {
			qpos = apy_indx[0];
		} else {
			qpos = -1;
		}
		printf("Min Q: %f at %i\n",  apy_acc[0], apy_indx[0]);
	}
	__syncthreads();

}

__device__ void intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, invPiv;
	int i, j, ipos, k;
	
	invPiv = 1 / smp.matriz[kfil * smp.mat_adv_row + jcol];
	
	int thd_indx = threadIdx.y*blockDim.x + threadIdx.x;
	int block_dim = blockDim.x * blockDim.y;
	
	ipos = kfil * smp.mat_adv_row;

	for (j = thd_indx; j < smp.var_all; j+= block_dim) { // Modifico la fila k
		smp.matriz[ipos + j] *= invPiv;
	}
	
	// if (thd_indx == 0) printf("INI SWAPPING\n");
	__syncthreads();
	
	for (i = threadIdx.y; i < smp.rest_fin; i += blockDim.y) {
		if (i != kfil) {
			m = smp.matriz[i*smp.mat_adv_row + jcol];
			if (threadIdx.x == 0) {
				// printf("%i: %.2f - (%.2f*%.2f)\t",  i, smp.Xb[i*smp.mat_adv_row], m, smp.Xb[ipos]);
				smp.Xb[i*smp.mat_adv_row] -= m*smp.Xb[ipos]; // Modifico Xb, m*smp.Xb[kfil * smp.mat_adv_row]
			}
			
			for (j = threadIdx.x; j < smp.var_all; j += blockDim.x) { // Modifico la Matriz
				if (j != jcol) {
					smp.matriz[i*smp.mat_adv_row + j] -= m * smp.matriz[ipos + j]; // Aca esta actualizacion se hace coalesced y como es la mas importante podemos decir que el acceso es coalesced mayoritariamente
				}
			}
		}
	}
	
	__syncthreads();
	
	for (i = thd_indx; i < smp.rest_fin; i+= block_dim) { // Modifico la columna j
		smp.matriz[i*smp.mat_adv_row + jcol] = 0;
	}
	
	__syncthreads();
	
	if (thd_indx == 0) {
		smp.matriz[ipos + jcol] = 1; // Modifico el pivote
		smp.Xb[ipos] *= invPiv; // Modifico Xb para la kfila
		
		k = smp.top[jcol];
		smp.top[jcol] = smp.left[ipos];
		smp.left[ipos] = k;
		
		smp.Cb[ipos] = smp.z[jcol];
		// printf("END SWAPPING\n");
	}
	__syncthreads();
	
	
 }
 
void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.rest_fin + 6, smp.mat_adv_row);
	for(int i = 0; i < smp.rest_fin + 6; i++) {
		for(int j = 0; j < smp.mat_adv_row; j++) {
			printf("%.2f\t", (double) smp.tabloide[i*smp.mat_adv_row + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
} 
 
__device__ void printStatusDev(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.rest_fin + 6, smp.mat_adv_row);
	for(int i = 0; i < smp.rest_fin + 6; i++) {
		for(int j = 0; j < smp.mat_adv_row; j++) {
			printf("%.2f\t", (double) smp.tabloide[i*smp.mat_adv_row + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	double bi, val;
	double min = 0;
	int varType;
	
	for(int i = 0; i < smp.var_x; i++) {
		bi = findVarXbValue(smp, i);
		val = bi + smp.inf[i];
		if (val != 0 || bi != 0) {
			printf("x%i = %.12f  (Xbi = %.12f)\n", findVarIndex(smp, i),  val, bi);
			min -= val * smp.z[i];
		}
	}
	
	for(int i = smp.var_x; i < smp.var_all; i++) {
		bi = findVarXbValue(smp, i);
		if (bi != 0) {
			varType = smp.var_type[i];
			if (varType == 1) {
				val = bi;
				printf("s%i = %.12f \n", findVarIndex(smp, i), val);
			} else {
				val = bi;
				printf("a - error%i = %.12f\n", findVarIndex(smp, i),  val);
			}
		}
	}
	
	printf("Z min = %.2f \n", -min);
	
}

__device__ void printResultDev(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	double bi, val;
	double min = 0;
	int varType;
	
	for(int i = 0; i < smp.var_x; i++) {
		bi = findVarXbValueDev(smp, i);
		val = bi + smp.inf[i];
		if (val != 0 || bi != 0) {
			printf("x%i = %.12f  (Xbi = %.12f)\n", findVarIndexDev(smp, i),  val, bi);
			min -= val * smp.z[i];
		}
	}
	
	for(int i = smp.var_x; i < smp.var_all; i++) {
		bi = findVarXbValueDev(smp, i);
		if (bi != 0) {
			varType = smp.var_type[i];
			if (varType == 1) {
				val = bi;
				printf("s%i = %.12f \n", findVarIndexDev(smp, i), val);
			} else {
				val = bi;
				printf("a - error%i = %.12f\n", findVarIndexDev(smp, i),  val);
			}
		}
	}
	
	printf("Z min = %.2f \n", -min);
	
}

double findVarXbValue(TSimplexGPUs &smp, int indx) {
	int lefti;
	for(int i = 0; i < smp.rest_fin; i++) {
		lefti = ((int) smp.left[i*smp.mat_adv_row]);
		if (indx == (lefti - 1)) {
			return smp.Xb[i*smp.mat_adv_row];
		}
	}
	return 0;
}

__device__ double findVarXbValueDev(TSimplexGPUs &smp, int indx) {
	int lefti;
	for(int i = 0; i < smp.rest_fin; i++) {
		lefti = ((int) smp.left[i*smp.mat_adv_row]);
		if (indx == (lefti - 1)) {
			return smp.Xb[i*smp.mat_adv_row];
		}
	}
	return 0;
}

int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx  + 1;
	} else {
		for (int i = 0; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}

__device__ int findVarIndexDev(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx  + 1;
	} else {
		for (int i = 0; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}
