#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "toursimplexmgpus.h"

const double CasiCero_Simplex = 1.0E-7;
// const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 100; //100; //sqrt(MaxNReal);

void resolver_cpu(TSimplexGPUs &simplex) ;
void resolver_simplex_big_m(TSimplexGPUs &simplex) ;
bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);
int locate_min_dj(TSimplexGPUs &smp);
int locate_min_ratio(TSimplexGPUs &smp, int zpos);
void resolver_ejemplo1();
void resolver_ejemplo2();
void resolver_ejemplo3();
void printStatus(TSimplexGPUs &smp);
void printResult(TSimplexGPUs &smp);
int findVarIndex(TSimplexGPUs &smp, int indx);

extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	/*
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]);
	}
	*/
	resolver_ejemplo3();
	
}

void resolver_ejemplo1() {
	
/*
	Problema Propuesto por Marco
		Min 4x1 +x2
		st:
		3x1 +1x2 = 3
		4x1 +3x2 ≥ 6
		1x1 +2x2 ≤ 4

		x1, x2 ≥ 0 .
=>		
		Min 4x1 +1x2 +MA1 +MA2
		Subject to:
		3x1 +x2 +A1 = 3
		4x1 +3x2 −s1 +A2 = 6
		x1 +2x2 +s2 = 4
		
		x1, x2, A1, s1, A2, s2 ≥ 0
Res:
	x1 = 0.400000, x2 = 1.800000, s1 = 1.000000 Verificado correcto
*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 6;
	simplex.NColumnas = simplex.NVariables + 3;
	simplex.NRestricciones = 3;
	simplex.Nfilas = simplex.NRestricciones + 3;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
	// L, 		Cb, 	Xb
		0, 	0, 	0,		-4, 	-1, 	0, 	0, 	-M, 	-M, // z function
		0, 	0, 	0, 	0, 	0, 	1, 	1, 	2, 	2, // 0 var, 1 slack var, 2 artificial var
		0, 	0, 	0, 	3, 	4, 	5, 	6, 	7, 	8, // top vector
		-7,	-M,	3,	 	3, 	1, 	0, 	0, 	1,  	0,
		-8, 	-M, 	6, 	4, 	3,  	-1, 	0, 	0, 	1,
		-6, 	0, 	4, 	1, 	2, 	0, 	1, 	0, 	0
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}

void resolver_ejemplo2() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 8x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= 5.3
		x1  		- x3 		≤ 2.9

		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
=>		
		Max -x1 - 8x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= 5.3
		x1  		  - x3 	≤ 2.9
		x1					≤ 12
				x2			≤ 6
						x3	≤ 5
=> cambio variable para las cotas inferiores	
		Max -x1 - 8x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= 5.3 + 6 		= 11.3
		x1  		  - x3 	≤ 2.9        - 5 	= - 2.1 => invert to get 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		0 ≤ x1 ≤ 12,  0 ≤ x2 ≤ 12, 0 ≤ x3 ≤ 10				
=>
		Max -x1 - 8x2 - 2x3
		st:
		x1 + x2 + x3 	- s1 + a1 = 0.5
		x1 + x2 			+ a2 = 11.3
		-x1  		  + x3 	- s2 + a3 = 2.1
		x1					+ s3 = 12
				x2			+ s4 = 12
						x3	+ s5 = 10
=>
*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 11;
	simplex.NColumnas = simplex.NVariables + 3;
	simplex.NRestricciones = 6;
	simplex.Nfilas = simplex.NRestricciones + 3;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
	// L, 		Cb, 	Xb
		0, 	0, 	0,			-1, 	-8, 	-2, 	0,	  -M, 	-M, 	0,		-M, 	0,		0,		0,	 // z function
		0, 	0, 	0, 		0, 	0, 	0, 	1, 	2, 	2, 	1, 	2, 	1, 	1, 	1, // 0 var, 1 slack var, 2 artificial var
		0, 	0, 	0, 		3, 	4, 	5, 	6, 	7, 	8,  	9, 	10, 	11, 	12, 	13,// top vector
		
		-7,	-M,	0.5,		1,		1,		1,		-1,	1,		0,		0,		0, 	0,		0,		0,
		-8, 	-M, 	11.3, 	1,		1,		0,		0,		0,		1,		0,		0, 	0,		0,		0,
		-10, 	-M, 	2.1, 		1,		0,		-1,	0,		0,		0,		-1,	1,		0,		0, 	0, 
		-11, 	0, 	12, 		1,		0,		0,		0,		0,		0,		0,		0, 	1,		0,		0,
		-12, 	0, 	12, 		0,		1,		0,		0,		0,		0,		0,		0, 	0,		1,		0,
		-13, 	0, 	10, 		0,		0,		1,		0,		0,		0,		0,		0, 	0,		0,		1
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}

void resolver_ejemplo3() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9

		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado
		
=> cambio variable para las cotas inferiores
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
		
=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s6 ≥ 0
	
	RES: x1 = 0.7 xc3 = 2.8 => x3 = 2.8 - 5 = -2.2
*/	
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 11;
	simplex.NColumnas = simplex.NVariables + 3;
	simplex.NRestricciones = 6;
	simplex.Nfilas = simplex.NRestricciones + 3;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		0, 0, 0, -1, -3, -2, 0, -M, -M, 0, -M, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 1, 2, 2, 1, 2, 1, 1, 1, 
		0, 0, 0, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 
		7, -M, 0.5, 1, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 
		8, -M, 0.7, 1, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 
		10, -M, 2.1, -1, 0, 1, 0, 0, 0, -1, 1, 0, 0, 0, 
		11, 0, 12, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 
		12, 0, 12, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 
		13, 0, 10, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}


void resolver_cpu(TSimplexGPUs &simplex) {
	resolver_simplex_big_m(simplex);
}

void resolver_simplex_big_m(TSimplexGPUs &simplex) {
	int zpos, qpos, it;
	
	printf("resolver_simplex_big_m INT \n");
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_min_dj(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
			printResult(simplex);
			return;
		}
		
		qpos = locate_min_ratio(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		
		if (it == 14) {
			printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}

int locate_min_dj(TSimplexGPUs &smp) {
	int mejorz, z, y, top;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 3; z < smp.NColumnas; z++) {
		top = smp.tabloide[2*smp.NColumnas + z] ;
		if (top > 0 && smp.tabloide[smp.NColumnas + top] != 2) { // it is not an artificial variable
			apz = -smp.tabloide[z];
			for (y = 3; y < smp.Nfilas; y++) {
				apz += smp.tabloide[y*smp.NColumnas + 1] * smp.tabloide[y*smp.NColumnas + z]; 
			}
			if (apz < 0 && apz < min_apz) {
				mejorz = z;
				min_apz = apz;
				printf("MIn Zj-Cj: %f\n",  min_apz);
			}
		}
	}
	
	return mejorz;
}


int locate_min_ratio(TSimplexGPUs &smp, int zpos) {
	int mejory, y;
	double min_apy, qy, denom;

	mejory = -1;
	min_apy = MaxNReal;
	for (y = 3; y < smp.Nfilas; y++) {
		denom = smp.tabloide[y*smp.NColumnas + zpos];
		// printf("Cociente: %f\n",  denom);
		if (denom > CasiCero_Simplex) {
			qy = smp.tabloide[y*smp.NColumnas + 2] / denom;
			if (qy > 0 && qy < min_apy) {
				mejory = y;
				min_apy = qy;
				printf("MIn Q: %f\n",  min_apy);
			}
		}
	}
	printf("en locate qpos \n");
	return mejory;
	
}

bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int i, j, ipos, k;

	piv = smp.tabloide[kfil * smp.NColumnas + jcol];
	invPiv = 1 / piv;
	
	ipos = kfil * smp.NColumnas;
	for (j = 2; j < smp.NColumnas; j++) {
		smp.tabloide[ipos + j] *= invPiv;
	}
	smp.tabloide[kfil * smp.NColumnas + jcol] = 1;
	
	for (i = 3; i < smp.Nfilas; i++) {
		if (i != kfil) {
			m = smp.tabloide[i *smp.NColumnas + jcol] ;
			for (j = 2; j < smp.NColumnas; j++) {
				if (j != jcol) {
					smp.tabloide[i *smp.NColumnas + j] -= m * smp.tabloide[kfil*smp.NColumnas + j]; 
				} else {
					smp.tabloide[i*smp.NColumnas + j] = 0;
				}
			}
		}
	}
	
	k = smp.tabloide[2*smp.NColumnas + jcol];
	smp.tabloide[2*smp.NColumnas + jcol] = smp.tabloide[smp.NColumnas*kfil];
	smp.tabloide[smp.NColumnas*kfil] = k;
	
	smp.tabloide[kfil*smp.NColumnas + 1] = smp.tabloide[jcol] ;
	
	return true;
 }


void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.Nfilas, smp.NColumnas);
	for(int i = 0; i < smp.Nfilas; i++) {
		for(int j = 0; j < smp.NColumnas; j++) {
			printf("%.1f\t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	int indx;
	string nvar = "xos";
	
	/* // Print left vector
	printf("left = [%f", smp.tabloide[3*smp.NColumnas]);
	for(int i = 4; i < smp.Nfilas; i++) printf(", %f", smp.tabloide[i*smp.NColumnas]);
	printf("]\n");
	*/
	
	//printf("z = %f\n", smp.tabloide[smp.NVariables]);
	int varType;
	for(int i = 3; i < smp.Nfilas; i++) {
		indx = smp.tabloide[i*smp.NColumnas];
		indx = (indx < 0)? -indx: indx;
		varType = smp.tabloide[smp.NColumnas + indx];
		if (varType == 0) {
				nvar = "x";
		} else if (varType == 1) {
			nvar = "s";
		} else {
			nvar = "a - error ";
		}
		
		printf("%s%i = %.2f \n", nvar.c_str(), findVarIndex(smp, indx),  smp.tabloide[i*smp.NColumnas + 2]);
	}
	
}

int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.tabloide[smp.NColumnas + indx];
	if ( varType == 0) {
		return indx - 2;
	} else {
		for (int i = 3; i < indx; i++) {
			if (smp.tabloide[smp.NColumnas + i] == varType) vind++;
		}
	}
	return vind;
}





