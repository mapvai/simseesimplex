#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "toursimplexmgpus.h"

const double CasiCero_Simplex = 1.0E-7;
// const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 100; //100; //sqrt(MaxNReal);

void resolver_cpu(TSimplexGPUs &simplex) ;
void resolver_simplex_big_m(TSimplexGPUs &simplex) ;
bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);
int locate_zpos(TSimplexGPUs &smp);
int locate_qpos(TSimplexGPUs &smp, int zpos);
void resolver_ejemplo1();
void resolver_ejemplo2();
void printStatus(TSimplexGPUs &smp);
void printResult(TSimplexGPUs &smp);

extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	/*
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]);
	}
	*/
	resolver_ejemplo2();
	
}

void resolver_ejemplo1() {
	
/*
	Problema Propuesto por Marco
		Min 4x1 +x2
		st:
		3x1 +1x2 = 3
		4x1 +3x2 ≥ 6
		1x1 +2x2 ≤ 4

		x1, x2 ≥ 0 .
=>		
		Min 4x1 +1x2 +MA1 +MA2
		Subject to:
		3x1 +x2 +A1 = 3
		4x1 +3x2 −s1 +A2 = 6
		x1 +2x2 +s2 = 4
		
		x1, x2, A1, s1, A2, s2 ≥ 0
=>
*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 6;
	simplex.NColumnas = simplex.NVariables + 3;
	simplex.NRestricciones = 3;
	simplex.Nfilas = simplex.NRestricciones + 3;
	simplex.cantArtVars = 2;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
	// L, 		Cb, 	Xb
		0, 	0, 	0,		-4, 	-1, 	0, 	0, 	-M, 	-M, // z function
		0, 	0, 	0, 	0, 	0, 	1, 	1, 	2, 	2, // 0 var, 1 slack var, 2 artificial var
		0, 	0, 	0, 	3, 	4, 	5, 	6, 	7, 	8, // top vector
		-7,	-M,	3,	 	3, 	1, 	0, 	0, 	1,  	0,
		-8, 	-M, 	6, 	4, 	3,  	-1, 	0, 	0, 	1,
		-6, 	0, 	4, 	1, 	2, 	0, 	1, 	0, 	0
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}

void resolver_ejemplo2() {
	
/*
	Problema Propuesto por SimSEE
		Min 4x1 +x2
		st:
		3x1 +1x2 = 3
		4x1 +3x2 ≥ 6
		1x1 +2x2 ≤ 4

		x1, x2 ≥ 0 .
=>		
		Min 4x1 +1x2 +MA1 +MA2
		Subject to:
		3x1 +x2 +A1 = 3
		4x1 +3x2 −s1 +A2 = 6
		x1 +2x2 +s2 = 4
		
		x1, x2, A1, s1, A2, s2 ≥ 0
=>
*/
	
	TSimplexGPUs simplex; // (TSimplexGPUs*)malloc(sizeof(TSimplexGPUs));
	simplex.NVariables = 6;
	simplex.NColumnas = simplex.NVariables + 3;
	simplex.NRestricciones = 3;
	simplex.Nfilas = simplex.NRestricciones + 3;
	simplex.cantArtVars = 2;
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
	// L, 		Cb, 	Xb
		0, 	0, 	0,		-4, 	-1, 	0, 	0, 	-M, 	-M, // z function
		0, 	0, 	0, 	0, 	0, 	1, 	1, 	2, 	2, // 0 var, 1 slack var, 2 artificial var
		0, 	0, 	0, 	3, 	4, 	5, 	6, 	7, 	8, // top vector
		-7,	-M,	3,	 	3, 	1, 	0, 	0, 	1,  	0,
		-8, 	-M, 	6, 	4, 	3,  	-1, 	0, 	0, 	1,
		-6, 	0, 	4, 	1, 	2, 	0, 	1, 	0, 	0
	};
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}


void resolver_cpu(TSimplexGPUs &simplex) {
	resolver_simplex_big_m(simplex);
}

void resolver_simplex_big_m(TSimplexGPUs &simplex) {
	int zpos, qpos, it;
	
	printf("resolver_simplex_big_m INT \n");
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_zpos(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s\n", "Condicion de parada maximo encontrado");
			printResult(simplex);
			return;
		}
		
		qpos = locate_qpos(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s\n", "Posicion de cociente no encontrada");
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		
		if (it == 4) {
			printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}

int locate_zpos(TSimplexGPUs &smp) {
	int mejorz, z, y, top;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 3; z < smp.NColumnas; z++) {
		top = smp.tabloide[2*smp.NColumnas + z] ;
		if (top > 0 && smp.tabloide[smp.NColumnas + top] != 2) { // it is not an artificial variable
			apz = -smp.tabloide[z];
			for (y = 3; y < smp.Nfilas; y++) {
				apz += smp.tabloide[y*smp.NColumnas + 1] * smp.tabloide[y*smp.NColumnas + z]; 
			}
			if (apz < 0 && apz < min_apz) {
				mejorz = z;
				min_apz = apz;
				printf("MIn Zj-Cj: %f\n",  min_apz);
			}
		}
	}
	
	return mejorz;
}


int locate_qpos(TSimplexGPUs &smp, int zpos) {
	int mejory, y;
	double min_apy, qy;

	mejory = -1;
	min_apy = MaxNReal;
	for (y = 3; y < smp.Nfilas; y++) {
		qy = smp.tabloide[y*smp.NColumnas + 2] / smp.tabloide[y*smp.NColumnas + zpos];
		if (qy > 0 && qy < min_apy) {
			mejory = y;
			min_apy = qy;
			printf("MIn Q: %f\n",  min_apy);
		}
	}
	
	return mejory;
	
}

bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int i, j, ipos, k;

	piv = smp.tabloide[kfil * smp.NColumnas + jcol];
	invPiv = 1 / piv;
	
	ipos = kfil * smp.NColumnas;
	for (j = 2; j < smp.NColumnas; j++) {
		smp.tabloide[ipos + j] *= invPiv;
	}
	smp.tabloide[kfil * smp.NColumnas + jcol] = 1;
	
	for (i = 3; i < smp.Nfilas; i++) {
		if (i != kfil) {
			m = smp.tabloide[i *smp.NColumnas + jcol] ;
			for (j = 2; j < smp.NColumnas; j++) {
				if (j != jcol) {
					smp.tabloide[i *smp.NColumnas + j] -= m * smp.tabloide[kfil*smp.NColumnas + j]; 
				} else {
					smp.tabloide[i*smp.NColumnas + j] = 0;
				}
			}
		}
	}
	
	k = smp.tabloide[2*smp.NColumnas + jcol];
	smp.tabloide[2*smp.NColumnas + jcol] = smp.tabloide[smp.NColumnas*kfil];
	smp.tabloide[smp.NColumnas*kfil] = k;
	
	smp.tabloide[kfil*smp.NColumnas + 1] = smp.tabloide[jcol] ;
	
	return true;
 }


void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.Nfilas, smp.NColumnas);
	for(int i = 0; i < smp.Nfilas; i++) {
		for(int j = 0; j < smp.NColumnas; j++) {
			printf("%f \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	int indx;
	string nvar = "xos";
	
	//printf("left = [%i", smp.left[0]);
	for(int i = 3; i < smp.Nfilas; i++) printf(", %f", smp.tabloide[i*smp.Nfilas]);
	printf("]\n");
	
	//printf("z = %f\n", smp.tabloide[smp.NVariables]);
	
	for(int i = 3; i < smp.Nfilas; i++) {
		indx = smp.tabloide[i*smp.NColumnas];
		indx = (indx < 0)? -indx: indx;
		if (smp.tabloide[smp.NColumnas + indx] == 0) {
				nvar = "x";
		} else {
			nvar = "s";
		}
		
		printf("%s%i = %f \n", nvar.c_str(), indx - 2,  smp.tabloide[i*smp.NColumnas + 2]);
	}
	
}





