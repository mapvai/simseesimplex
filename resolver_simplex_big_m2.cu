#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "toursimplexmgpus2.h"

const double CasiCero_Simplex = 1.0E-7;
// const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 100; //100; //sqrt(MaxNReal);

void resolver_cpu(TSimplexGPUs &simplex) ;
void resolver_simplex_big_m(TSimplexGPUs &simplex) ;
bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);
int locate_min_dj(TSimplexGPUs &smp);
int locate_min_ratio(TSimplexGPUs &smp, int zpos);
void resolver_ejemplo1();
void resolver_ejemplo2();
void resolver_ejemplo3();
void printStatus(TSimplexGPUs &smp);
void printResult(TSimplexGPUs &smp);
int findVarIndex(TSimplexGPUs &smp, int indx);

extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	/*
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]);
	}
	*/
	resolver_ejemplo2();
	
}

void resolver_ejemplo1() {
	
/*
	Problema Propuesto por Marco
		Min 4x1 +x2
		st:
		3x1 +1x2 = 3
		4x1 +3x2 ≥ 6
		1x1 +2x2 ≤ 4

		x1, x2 ≥ 0 .
=>		
		Min 4x1 +1x2 +MA1 +MA2
		Subject to:
		3x1 +x2 +A1 = 3
		4x1 +3x2 −s1 +A2 = 6
		x1 +2x2 +s2 = 4
		
		x1, x2, A1, s1, A2, s2 ≥ 0
Res:
	x1 = 0.400000, x2 = 1.800000, s1 = 1.000000 Verificado correcto
*/
	
	TSimplexGPUs simplex;
	simplex.filas = 4;
	simplex.columnas = 7;
	int var_type[] = {0, 0, 0, 1, 1, 2, 2};
	double sup[] = {0, M, M};
	double inf[] = {0, 0, 0};
	int top[] = {0, 1, 2, 3, 4, 5, 6};
	int left[] = {0, 4, 5, 3};
	double Cb[] = {0, -M, -M, 0};
	
	double tabl[] = {
	// Xb
		0,		-4, 	-1, 	0, 	0, 	-M, 	-M, // z function
		3,	 	3, 	1, 	0, 	0, 	1,  	0,
		6, 	4, 	3,  	-1, 	0, 	0, 	1,
		4, 	1, 	2, 	0, 	1, 	0, 	0
	};
	
	simplex.var_type = (int*)&var_type;
	simplex.sup = (double*)&sup;
	simplex.inf = (double*)&inf;
	simplex.top = (int*)&top;
	simplex.left = (int*)&left;
	simplex.Cb = (double*)&Cb;
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}

void resolver_ejemplo2() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9

		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol SIMSEE: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado, z min = -21.7
		
=> cambio variable para las cotas inferiores xc = x + cota inf => x = xc - cota inf => Sol xc: x1 = 0, x2 = 0.7, x3 = 2.1
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
		
=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s5 ≥ 0
	
	RESULTADO OUR SIMPLEX: x1 = 0.7, xc2 = 0 => x2 = 0 - 6 = -6, xc3 = 2.8 => x3 = 2.8 - 5 = -2.2 Verificado, da tambien z min = -21.7
*/	
	TSimplexGPUs simplex;
	simplex.filas = 7;
	simplex.columnas = 12;
	int var_type[] = {0, 0, 0, 0, 1, 2, 2, 1, 2, 1, 1, 1}; // EL PRIMER VALOR EN ESTOS VECTORES ES DUMMY
	double sup[] = {0, 12, 6, 5};
	double inf[] = {0, 0, -6, -5};
	int top[] = {0, 1, 2, 3, 4, 5, 6, 7 , 8, 9, 10, 11};
	int left[] = {0, 4, 5, 7, 8, 9, 10};
	double Cb[] = {0, -M, -M, -M, 0, 0, 0};
	
	double tabl[] = {
	// Xb
		0, 	-1, 	-3, 	-2, 	0, 	-M, 	-M, 	0, 	-M, 	0, 	0, 	0, // z
		0.5, 	1, 	1, 	1, 	-1, 	1, 	0, 	0, 	0, 	0, 	0, 	0, 
		0.7, 	1, 	1, 	0, 	0, 	0, 	1, 	0, 	0, 	0, 	0, 	0, 
		2.1, -1, 	0, 	1, 	0, 	0, 	0, 	-1, 	1, 	0, 	0, 	0, 
		12, 	1, 	0, 	0, 	0, 	0, 	0, 	0, 	0, 	1, 	0, 	0, 
		12, 	0, 	1, 	0, 	0, 	0, 	0, 	0, 	0, 	0, 	1, 	0, 
		10, 	0, 	0, 	1, 	0, 	0, 	0, 	0, 	0, 	0, 	0, 	1
	};
	
	simplex.var_type = (int*)&var_type;
	simplex.sup = (double*)&sup;
	simplex.inf = (double*)&inf;
	simplex.top = (int*)&top;
	simplex.left = (int*)&left;
	simplex.Cb = (double*)&Cb;
	
	simplex.tabloide = (double*)&tabl;
	resolver_cpu(simplex);
}


void resolver_cpu(TSimplexGPUs &simplex) {
	resolver_simplex_big_m(simplex);
}

void resolver_simplex_big_m(TSimplexGPUs &simplex) {
	int zpos, qpos, it;
	
	printf("resolver_simplex_big_m2 INT \n");
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_min_dj(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
			printResult(simplex);
			return;
		}
		
		qpos = locate_min_ratio(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		
		if (it == 14) {
			printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}

int locate_min_dj(TSimplexGPUs &smp) {
	int mejorz, z, y, top;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 1; z < smp.columnas; z++) {
		top = smp.top[z];
		if (top > 0 && smp.var_type[top] != 2) { // it is not an artificial variable
			apz = -smp.tabloide[z];
			for (y = 1; y < smp.filas; y++) {
				apz += smp.Cb[y] * smp.tabloide[y*smp.columnas + z]; 
			}
			if (apz < 0 && apz < min_apz) {
				mejorz = z;
				min_apz = apz;
				printf("MIn Zj-Cj: %f\n",  min_apz);
			}
		}
	}
	
	return mejorz;
}


int locate_min_ratio(TSimplexGPUs &smp, int zpos) {
	int mejory, y;
	double min_apy, qy, denom;

	mejory = -1;
	min_apy = MaxNReal;
	printf("qy:\t");
	for (y = 1; y < smp.filas; y++) {
		denom = smp.tabloide[y*smp.columnas + zpos];
		printf("%.1f / %.1f ",  smp.tabloide[y*smp.columnas], denom);
		// printf("Denominador: %f\n",  denom);
		if (denom > CasiCero_Simplex) {
			qy = smp.tabloide[y*smp.columnas] / denom;
			printf(" (%.1f)\t",  qy);
			if (qy > 0 && qy < min_apy) {
				mejory = y;
				min_apy = qy;
			}
		} else {
			printf(" (NA)\t");
		}
	}
	printf("Min Q: %f\n",  min_apy);
	return mejory;
	
}

bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int i, j, ipos, k;

	piv = smp.tabloide[kfil * smp.columnas + jcol];
	invPiv = 1 / piv;
	
	ipos = kfil * smp.columnas;
	for (j = 2; j < smp.columnas; j++) {
		smp.tabloide[ipos + j] *= invPiv;
	}
	smp.tabloide[kfil * smp.columnas + jcol] = 1;
	
	for (i = 1; i < smp.filas; i++) {
		if (i != kfil) {
			m = smp.tabloide[i *smp.columnas + jcol] ;
			for (j = 0; j < smp.columnas; j++) {
				if (j != jcol) {
					smp.tabloide[i *smp.columnas + j] -= m * smp.tabloide[kfil*smp.columnas + j]; 
				} else {
					smp.tabloide[i*smp.columnas + j] = 0;
				}
			}
		}
	}
	
	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil];
	smp.left[kfil] = k;
	
	smp.Cb[kfil] = smp.tabloide[jcol];
	
	return true;
 }


void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.filas, smp.columnas);
	for(int i = 0; i < smp.filas; i++) {
		for(int j = 0; j < smp.columnas; j++) {
			printf("%.1f\t", smp.tabloide[i*smp.columnas + j] );
			//printf("%E \t", smp.tabloide[i*smp.columnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.columnas) + j, smp.tabloide[(i*smp.columnas) + j]);
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	int indxLeft;
	string nvar = "xos";
	
	//printf("z = %f\n", smp.tabloide[smp.NVariables]);
	int varType;
	double value;
	bool estaEnBase;
	for(int i = 1; i < smp.columnas; i++) {
		estaEnBase = false;
		for (int b = 1; b < smp.filas; b++) {
			if (i == smp.left[b]) {
				estaEnBase = true;
				indxLeft = b;
				break;
			}
		}
		if (estaEnBase) {
			value = smp.tabloide[indxLeft*smp.columnas];
		} else {
			value = 0;
		}
		varType = smp.var_type[i];
		if (varType == 0) {
			nvar = "x";
			value += smp.inf[i];
		} else if (varType == 1) {
			nvar = "s";
		} else {
			nvar = "a";
		}
		
		printf("%s%i = %.2f \n", nvar.c_str(), findVarIndex(smp, i),  value);
	}
	
}

int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx;
	} else {
		for (int i = 1; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}

/*
void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	int indx;
	string nvar = "xos";
	
	 // Print left vector
	printf("left = [%f", smp.tabloide[3*smp.columnas]);
	for(int i = 4; i < smp.Nfilas; i++) printf(", %f", smp.tabloide[i*smp.columnas]);
	printf("]\n");
	
	//printf("z = %f\n", smp.tabloide[smp.NVariables]);
	int varType;
	double value;
	for(int i = 1; i < smp.filas; i++) {
		value = smp.tabloide[i*smp.columnas];
		indx = smp.left[i];
		indx = (indx < 0)? -indx: indx;
		varType = smp.var_type[indx];
		if (varType == 0) {
			nvar = "x";
			value += smp.inf[indx];
		} else if (varType == 1) {
			nvar = "s";
		} else {
			nvar = "a - error ";
		}
		
		printf("%s%i = %.2f \n", nvar.c_str(), findVarIndex(smp, indx),  smp.tabloide[i*smp.columnas]);
	}
	
}
*/




