#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "toursimplexmgpus_final.h"

const double CasiCero_Simplex = 1.0E-7;
// const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO

const double M = 100; //100; //sqrt(MaxNReal);

void resolver_cpu(TSimplexGPUs &simplex) ;
TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide);
void moverseASolFactible(TSimplexGPUs &smp);
void agregarRestriccionesCotaSup(TSimplexGPUs &smp);
void agregarVariablesHolguraArtificiales(TSimplexGPUs &smp);
void resolver_simplex_big_m(TSimplexGPUs &simplex) ;
bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol);
int locate_min_dj(TSimplexGPUs &smp);
int locate_min_ratio(TSimplexGPUs &smp, int zpos);
void resolver_ejemplo1();
void resolver_ejemplo2trasnform();
void printStatus(TSimplexGPUs &smp);
void printResult(TSimplexGPUs &smp);
double findVarXbValue(TSimplexGPUs &smp, int indx);
int findVarIndex(TSimplexGPUs &smp, int indx);

extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	/*
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]);
	}
	*/
	resolver_ejemplo2trasnform();
	
}

void resolver_ejemplo1() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9

		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol SIMSEE: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado, z min = -21.7
		
=> cambio variable para las cotas inferiores xc = x + cota inf => x = xc - cota inf => Sol xc: x1 = 0, x2 = 0.7, x3 = 2.1
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
		
=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s5 ≥ 0
	
	RESULTADO OUR SIMPLEX: x1 = 0.7, xc2 = 0 => x2 = 0 - 6 = -6, xc3 = 2.8 => x3 = 2.8 - 5 = -2.2 Verificado, da tambien z min = -21.7
*/	
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		3, 3, 15, 0, -1, -3, -2, 0, -M, -M, 0, -M, 0, 0, 0, 
		11, 6, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 12, 12, 10, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, -6, -5, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 1, 2, 2, 1, 2, 1, 1, 1, 
		0, 0, 0, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 
		5, 0, -M, 0.5, 1, 1, 1, -1, 1, 0, 0, 0, 0, 0, 0, 
		6, 2, -M, 0.7, 1, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 
		8, 0, -M, 2.1, -1, 0, 1, 0, 0, 0, -1, 1, 0, 0, 0, 
		9, 1, 0, 12, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 
		10, 1, 0, 12, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 
		11, 1, 0, 10, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 
	};
	
	TabloideGPUs tabloide = (double*)&tabl;
	TSimplexGPUs simplex = desestructurarTabloide(tabloide);
	resolver_cpu(simplex);
}

void resolver_ejemplo2trasnform() {
	
/*
	Problema Propuesto por SimSEE
		Min x1 + 3x2 + 2x3
		st:
		x1 + x2 + x3 	≥ -10.5
		x1 + x2 			= - 5.3
		x1  		- x3 		≤ 2.9

		0 ≤ x1 ≤ 12,  -6 ≤ x2 ≤ 6, -5 ≤ x3 ≤ 5
		
		Sol SIMSEE: x1 = 0, x2 = -5.3, x3 = -2.9 Verificado, z min = -21.7
		
=> cambio variable para las cotas inferiores xc = x + cota inf => x = xc - cota inf => Sol xc: x1 = 0, x2 = 0.7, x3 = 2.1
	Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ -10.5 + 6 + 5 = 0.5
		x1 + x2 			= - 5.3 + 6 		 = 0.7
		x1  		  - x3 	≤ 2.9 - 5 			 = -2.1
		x1					≤ 12
				x2			≤ 6 + 6 			 = 12
						x3	≤ 5 + 5				 = 10
						
		x1, x2, x3 > 0
*/	
	
    //simplex->tabloide = (double*)malloc((simplex->NVariables + 1)*(simplex->NRestricciones + 1)*sizeof(double));
	double tabl[] = {
		3, 3, 15, 0, -1, -3, -2, 0, 0, 0, 0, 0, 0, 0, 0, 
		11, 6, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 12, 12, 10, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, -6, -5, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0.5, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 2, 0, 0.7, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 2.1, -1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
	};
	
	TabloideGPUs tabloide = (double*)&tabl;
	TSimplexGPUs simplex = desestructurarTabloide(tabloide);
	
	/*
	=>	Move to a factible solution (Xb > 0)
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	≥ 0.5
		x1 + x2 			= 0.7
	   -x1  		 + x3 	≥ 2.1
		x1					≤ 12
				x2			≤ 12
						x3	≤ 10
						
		x1, x2, x3 > 0
	
=> Agregamos las variables de holgura y demasia 
		Max -x1 - 3x2 - 2x3
		st:
		x1 + x2 + x3 	 - s1 + a1  = 0.5
		x1 + x2 			+ a2          = 0.7
	 - x1  		 + x3 	 - s2 + a3 = 2.1
		x1					+ s3 		 = 12
				x2			+ s4 		 = 12
						x3	+ s5 		 = 10
	
	x1..s5 ≥ 0
	
	RESULTADO OUR SIMPLEX: x1 = 0.7, xc2 = 0 => x2 = 0 - 6 = -6, xc3 = 2.8 => x3 = 2.8 - 5 = -2.2 Verificado, da tambien z min = -21.7
	*/
	
	moverseASolFactible(simplex);
	
	agregarRestriccionesCotaSup(simplex);
	
	agregarVariablesHolguraArtificiales(simplex);
	
	resolver_cpu(simplex);

}

void resolver_cpu(TSimplexGPUs &simplex) {
	resolver_simplex_big_m(simplex);
}

TSimplexGPUs desestructurarTabloide(TabloideGPUs &tabloide) {
	TSimplexGPUs smp;
	
	smp.tabloide = tabloide;
	
	smp.var_x = (int) tabloide[0];
	smp.rest_ini = (int) tabloide[1];
	smp.mat_adv_row = (int) tabloide[2];
	smp.var_all = (int) tabloide[smp.mat_adv_row];
	smp.rest_fin = (int) tabloide[smp.mat_adv_row + 1];
	
	smp.z = &tabloide[4]; // funcion z, cantidad de variables, horizontal
    smp.flg_x = &tabloide[smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x
	
	smp.sup = &tabloide[2*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	smp.inf = &tabloide[3*smp.mat_adv_row + 4]; // cantidad de variables x = smp.var_x, horizontal
	
	smp.var_type = &tabloide[4*smp.mat_adv_row + 4]; // largo filas, horizontal
	
	smp.top = &tabloide[5*smp.mat_adv_row + 4]; // largo filas, horizontal
    smp.left = &tabloide[6*smp.mat_adv_row]; // largo restricciones finales, vertical
	
	smp.flg_y = &tabloide[6*smp.mat_adv_row + 1]; // 0 restriccion >=, 1 <=, 2 =, vertical
	
	
	smp.Cb = &tabloide[6*smp.mat_adv_row + 2]; // cantidad de restricciones, vertical
	smp.Xb = &tabloide[6*smp.mat_adv_row + 3]; // cantidad de restricciones, vertical
	
    smp.matriz = &tabloide[6*smp.mat_adv_row + 4]; 
	
	return smp;
}

void moverseASolFactible(TSimplexGPUs &smp) {
	for (int i = 0; i < smp.rest_ini; i++) {
		if (smp.Xb[i*smp.mat_adv_row] < 0) {
			smp.Xb[i*smp.mat_adv_row] *= -1;
			for (int j = 0; j < smp.var_x; j++) {
				smp.matriz[i*smp.mat_adv_row + j] *= -1;
			}
			smp.flg_y[i*smp.mat_adv_row] = (smp.flg_y[i*smp.mat_adv_row] == 0) ? 1 : 2; // Move >= to <=
		}
	}
}

void agregarRestriccionesCotaSup(TSimplexGPUs &smp) {
	int qrest = smp.rest_ini;
	for (int i = 0; i < smp.var_x; i++) {
		if (smp.flg_x[i] == 1) {
			smp.flg_y[(smp.rest_ini + i)*smp.mat_adv_row] = 1;
			smp.Xb[(smp.rest_ini + i)*smp.mat_adv_row] = smp.sup[i];
			for (int j = 0; j < smp.var_x; j++) smp.matriz[qrest*smp.mat_adv_row + j] = (qrest == (j + smp.rest_ini))? 1 : 0;
			qrest ++;
		}
	}
	printf("%i / %i \n", smp.rest_fin, smp.rest_ini + qrest);
	if (smp.rest_fin != qrest) printf("DISCREPANCIA EN LA CANTIDAD DE RESTRICCIONES FINAL\n");
}

void agregarVariablesHolguraArtificiales(TSimplexGPUs &smp) {
	int var_s, var_a, var_count;
	var_s = 0; var_a = 0; var_count = smp.var_x;
	for (int i = 0; i < smp.var_x; i++) {
		smp.var_type[i] = 0;
		smp.top[i] = i + 1;
	}
	
	// Completo con 0s la matriz
	for (int i = 0; i < smp.rest_fin; i++) {
		for (int j = var_count; j < smp.var_all; j++) {
			smp.matriz[i*smp.mat_adv_row + j] = 0;
		}
	}
	
	for (int i = 0; i < smp.rest_fin; i++) {
		if (smp.flg_y[i*smp.mat_adv_row] == 0) { // rest >=
			smp.matriz[i*smp.mat_adv_row + var_count] = -1;
			smp.matriz[i*smp.mat_adv_row + var_count +1] = 1;
			
			smp.var_type[var_count] = 1;
			smp.var_type[var_count +1] = 2;
			
			smp.z[var_count] = 0;
			smp.z[var_count +1] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.top[var_count + 1] = var_count + 2;
			smp.left[i*smp.mat_adv_row] = var_count + 2;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_a++; var_count += 2;
		} else if (smp.flg_y[i*smp.mat_adv_row] == 1) { // rest <=
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 1;
			
			smp.z[var_count] = 0;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = 0;
			
			var_a++; var_count ++;
		} else { // 2: rest =
			smp.matriz[i*smp.mat_adv_row + var_count] = 1;
			
			smp.var_type[var_count] = 2;
			
			smp.z[var_count] = -M;
			
			smp.top[var_count] = var_count + 1;
			smp.left[i*smp.mat_adv_row] = var_count + 1;
			
			smp.Cb[i*smp.mat_adv_row] = -M;
			
			var_s++; var_count ++;
		}
	}
	
	if (smp.var_all != var_count) printf("DISCREPANCIA EN LA CANTIDAD DE VARIABLES FINAL\n");
	
}

void resolver_simplex_big_m(TSimplexGPUs &simplex) {
	int zpos, qpos, it;
	
	printf("resolver_simplex_big_m_final INT \n");
	
	printStatus(simplex);
	it = 0;
	
	do {
		zpos = locate_min_dj(simplex);
		printf("%s %d \n", "zpos", zpos);
		
		if (zpos < 0) {
			printf("%s %i\n", "Condicion de parada maximo encontrado en iteracion", it);
			printResult(simplex);
			return;
		}
		
		qpos = locate_min_ratio(simplex, zpos);
		printf("%s %d \n", "qpos", qpos);
		if (qpos < 0) {
			printf("%s %i\n", "Posicion de cociente no encontrada en iteracion", it);
			return;
		}
		
		intercambiarvars(simplex, qpos, zpos);
		
		printStatus(simplex);
		
		it++;
		
		if (it == 14) {
			printf("Max %i iterations achieved\n", it);
			return;
		}
	} while (true);
	
}

int locate_min_dj(TSimplexGPUs &smp) {
	int mejorz, z, y, top;
	double min_apz, apz;

	mejorz = -1;
	min_apz = 0;
	for (z = 0; z < smp.var_all; z++) {
		top = smp.top[z] - 1;
		if (smp.var_type[top] != 2) { // it is not an artificial variable
			apz = -smp.z[z];
			for (y = 0; y < smp.rest_fin; y++) {
				apz += smp.Cb[y*smp.mat_adv_row] * smp.matriz[y*smp.mat_adv_row + z]; // Cj
			}
			if (apz < 0 && apz < min_apz) {
				mejorz = z;
				min_apz = apz;
				printf("MIn Zj-Cj: %f\n",  min_apz);
			}
		}
	}
	
	return mejorz;
}


int locate_min_ratio(TSimplexGPUs &smp, int zpos) {
	int mejory, y;
	double min_apy, qy, denom;

	mejory = -1;
	min_apy = MaxNReal;
	printf("qy:\t");
	for (y = 0; y < smp.rest_fin; y++) {
		denom = smp.matriz[y*smp.mat_adv_row + zpos];
		printf("%.1f / %.1f ",  smp.Xb[y*smp.mat_adv_row], denom);
		// printf("Denominador: %f\n",  denom);
		if (denom > CasiCero_Simplex) {
			qy = smp.Xb[y*smp.mat_adv_row] / denom;
			printf(" (%.1f)\t",  qy);
			if (qy > 0 && qy < min_apy) {
				mejory = y;
				min_apy = qy;
			}
		} else {
			printf(" (NA)\t");
		}
	}
	printf("Min Q: %f\n",  min_apy);
	return mejory;
	
}

bool intercambiarvars(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, invPiv;
	int i, j, ipos, k;
	
	invPiv = 1 / smp.matriz[kfil * smp.mat_adv_row + jcol];
	
	ipos = kfil * smp.mat_adv_row;
	smp.Xb[kfil*smp.mat_adv_row] *= invPiv; // Modifico Xb
	for (j = 0; j < smp.var_all; j++) { // Modifico la k fila
		smp.matriz[ipos + j] *= invPiv;
	}
	smp.matriz[kfil * smp.mat_adv_row + jcol] = 1;
	
	for (i = 0; i < smp.rest_fin; i++) {
		if (i != kfil) {
			m = smp.matriz[i *smp.mat_adv_row + jcol];
			
			smp.Xb[i*smp.mat_adv_row] -= m*smp.Xb[kfil*smp.mat_adv_row]; // Modifico Xb
			for (j = 0; j < smp.var_all; j++) { // Modifico la Matriz
				if (j != jcol) {
					smp.matriz[i *smp.mat_adv_row + j] -= m * smp.matriz[kfil*smp.mat_adv_row + j]; 
				} else {
					smp.matriz[i*smp.mat_adv_row + j] = 0;
				}
			}
		}
	}
	
	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil*smp.mat_adv_row];
	smp.left[kfil*smp.mat_adv_row] = k;
	
	smp.Cb[kfil*smp.mat_adv_row] = smp.z[jcol] ;
	
	return true;
 }


void printStatus(TSimplexGPUs &smp) {
	printf("%s, (%i, %i)\n", "Tabloide", smp.rest_fin + 6, smp.mat_adv_row);
	for(int i = 0; i < smp.rest_fin + 6; i++) {
		for(int j = 0; j < smp.mat_adv_row; j++) {
			printf("%.1f\t", smp.tabloide[i*smp.mat_adv_row + j] );
			//printf("%E \t", smp.tabloide[i*smp.NColumnas + j] );
			//printf("(%i,%i,%i)%f  \t", i, j, (i*smp.NColumnas) + j, smp.tabloide[(i*smp.NColumnas) + j]);
		}
		printf("\n");
	}
	
}

void printResult(TSimplexGPUs &smp) {
	printf("%s\n", "Resultado");
	double bi, val;
	double min = 0;
	int varType;
	
	for(int i = 0; i < smp.var_x; i++) {
		bi = findVarXbValue(smp, i);
		val = bi + smp.inf[i];
		if (val != 0) {
			printf("x%i = %.2f  (Xbi = %.2f)\n", findVarIndex(smp, i),  val, bi);
			min -= val * smp.z[i];
		}
	}
	
	for(int i = smp.var_x; i < smp.var_all; i++) {
		bi = findVarXbValue(smp, i);
		if (bi != 0) {
			varType = smp.var_type[i];
			if (varType == 1) {
				val = bi;
				printf("s%i = %.2f \n", findVarIndex(smp, i), val);
			} else {
				val = bi;
				printf("a - error%i = %.2f\n", findVarIndex(smp, i),  val);
			}
		}
	}
	
	printf("Z min = %.2f \n", min);
	
}

double findVarXbValue(TSimplexGPUs &smp, int indx) {
	int lefti;
	for(int i = 0; i < smp.rest_fin; i++) {
		lefti = ((int) smp.left[i*smp.mat_adv_row]);
		if (indx == (lefti - 1)) {
			return smp.Xb[i*smp.mat_adv_row];
		}
	}
	return 0;
}

int findVarIndex(TSimplexGPUs &smp, int indx) {
	int vind = 1;
	int varType = smp.var_type[indx];
	if ( varType == 0) {
		return indx  + 1;
	} else {
		for (int i = 0; i < indx; i++) {
			if (smp.var_type[i] == varType) vind++;
		}
	}
	return vind;
}





