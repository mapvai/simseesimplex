#include <stdio.h>
#include <math.h>

#include "toursimplexmgpus_final.h"


extern "C" void free_mem(
	TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, 
	int NTrayectorias, int NEnteras, int NVariables, int NRestricciones, int cnt_varfijas, int cnt_RestriccionesRedundantes
						
) {
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		hipFree(h_simplex_array[kTrayectoria]);
	}
	
	hipFree(d_simplex_array);
 
}
