#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>		/* abs */
#include <assert.h>	/* assert */
#include <iostream>
#include <string>
using namespace std;

#include "tsimplexgpus.h"


// const double CasiCero_Simplex_Escalado = 1.0E-30; // NOT USED
const double CasiCero_Simplex = 1.0E-7;
const double CasiCero_Simplex_CotaSup = CasiCero_Simplex * 1.0E+3;
// const double CasiCero_VarEntera = 1.0E-5; // NOT USED
const double CasiCero_CajaLaminar = 1.0E-30;
const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO


void resolver_cpu(TSimplexGPUs &simplex) ;
int fijarCajasLaminares(TSimplexGPUs &smp, int &cnt_varfijas);
void posicionarPrimeraLibre(TSimplexGPUs &smp, int cnt_varfijas, int &cnt_fijadas, int &cnt_columnasFijadas, int &kPrimeraLibre) ; // Esta funcion es interna, no se necesita declarar aca?
bool fijarVariables(TSimplexGPUs &smp, int cnt_varfijas, int &cnt_columnasFijadas, int cnt_RestriccionesRedundantes);
bool intercambiar(TSimplexGPUs &smp, int kfil, int jcol);
void actualizo_iitop(TSimplexGPUs &smp, int k);
void actualizo_iileft(TSimplexGPUs &smp, int k) ;
void intercambioColumnas(TSimplexGPUs &smp, int j1, int j2);
void intercambioFilas(TSimplexGPUs &smp, int k1, int k2);
int buscarMejorPivoteEnCol(TSimplexGPUs &smp, int jCol, int iFilaFrom, int iFilaHasta);
bool enfilarVariablesLibres(TSimplexGPUs &smp, int cnt_columnasFijadas, int &cnt_RestriccionesRedundantes, int &cnt_variablesLiberadas);
int resolverIgualdades(TSimplexGPUs &smp, int &cnt_columnasFijadas, int cnt_varfijas, int &cnt_RestriccionesRedundantes, int cnt_igualdades);
bool filaEsFactible(TSimplexGPUs &smp, int kfila, bool &fantasma);
int pasoBuscarFactibleIgualdad4(TSimplexGPUs &smp, int IgualdadesNoResueltas, int * nCerosFilas, int * nCerosCols, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes);
int reordenarPorFactibilidad(TSimplexGPUs &smp, int cnt_RestriccionesRedundantes, int &cnt_RestrInfactibles);
void cambiar_borde_de_caja(TSimplexGPUs &smp, int k_fila);
int pasoBuscarFactible(TSimplexGPUs &smp, int &cnt_RestrInfactibles, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes);
int locate_zpos(TSimplexGPUs &smp, int kfila_z, int cnt_columnasFijadas);
void capturarElMejor(double &a_iq_DelMejor, double &a_it_DelMejor, int &p, bool &filaFantasma, bool &colFantasma, double a_iq, double a_it, int i, bool xFantasma_fila); // Esta funcion es interna, no se necesita declarar aca?
int mejorpivote(TSimplexGPUs &smp, int q, int kmax, bool &filaFantasma, bool &colFantasma, bool checkearFilaOpt, int cnt_RestriccionesRedundantes);
bool cambio_var_cota_sup_en_columna(TSimplexGPUs &smp, int q);
int locate_qOK(TSimplexGPUs &smp, int p, int jhasta, int jti, int cnt_RestriccionesRedundantes);
bool test_qOK(TSimplexGPUs &smp, int p, int q, int jti, double &apq, int cnt_RestriccionesRedundantes);
int darpaso(TSimplexGPUs &smp, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes);


extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	//int NEnteras;
	//int NVariables;
	//int NRestricciones;
	//int cnt_varfijas;
	//int cnt_RestriccionesRedundantes;
	
	//TSimplexVars smp_var_arr = (SimplexVars*)malloc(NTrayectorias*sizeof(SimplexVars));
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(simplex_array[kTrayectoria]); //,  &smp_var_arr[kTrayectoria]); 
	}
	
}


void resolver_cpu(TSimplexGPUs &simplex) { //,  TSimplexVars &vars) {
	int res;
	int cnt_columnasFijadas = 0; // Cantidad de columnas FIJADAS x o y fija encolumnada
	int result = 0;
	// int cnt_varfijas = 0; // Cantidad de variables fijadas, esta variable esta en la estructura Simplex
	// simplex.cnt_varfijas = 0; // Esto viene al cargarse la estructura
	int cnt_RestrInfactibles = 0;
	int cnt_variablesLiberadas = 0;
	string mensajeDeError;
	
	// MAP: Agrego este codigo para calcular las restricciones de igualdad
	int cnt_igualdades = 0;
	for (int i = 0; i <= simplex.NRestricciones; i++) {
		if (simplex.flg_y[i] == 2) {
			cnt_igualdades++;
		}
	}
	
	fijarCajasLaminares(simplex, simplex.cnt_varfijas);

	// Fijamos las variables que se hayan declarado como constantes.
	if (!fijarVariables(simplex, simplex.cnt_varfijas, cnt_columnasFijadas, simplex.cnt_RestriccionesRedundantes)) {
		mensajeDeError = "PROBLEMA INFACTIBLE - No fijar las variable Fijas.";
		printf("%s\n", mensajeDeError.c_str());
		result = -32;
		return;
	}

	if (!enfilarVariablesLibres(simplex, cnt_columnasFijadas, simplex.cnt_RestriccionesRedundantes, cnt_variablesLiberadas)) {
		mensajeDeError = "No fue posible conmutar a filas todas las variables libres";
		printf("%s\n", mensajeDeError.c_str());
		result = -33;
		return;
	}

	if (resolverIgualdades(simplex, cnt_columnasFijadas, simplex.cnt_varfijas, simplex.cnt_RestriccionesRedundantes, cnt_igualdades) != 1) {
		mensajeDeError = "PROBLEMA INFACTIBLE - No logré resolver las restricciones de igualdad.";
		printf("%s\n", mensajeDeError.c_str());
		result = -31;
		return;
	}

	lbl_inicio:

	reordenarPorFactibilidad(simplex, simplex.cnt_RestriccionesRedundantes, cnt_RestrInfactibles); // MAP: cnt_RestrInfactibles es modificada dentro del proc
	
	res = 1;
	while (cnt_RestrInfactibles > 0) {
		res = pasoBuscarFactible(simplex, cnt_RestrInfactibles, cnt_columnasFijadas, simplex.cnt_RestriccionesRedundantes);

		switch (res) {
			case 0: 
				if (cnt_RestrInfactibles > 0) {
					mensajeDeError = "PROBLEMA INFACTIBLE - Buscando factibilidad";
					printf("%s\n", mensajeDeError.c_str());
					result = -10;
					return;
				}
				break;
			case  -1:
				mensajeDeError = "NO encontramos pivote bueno - Buscando Factibilidad";
				printf("%s\n", mensajeDeError.c_str());
				result = -11;
				return;
			case -2:
				mensajeDeError = "???cnt_infactibles= 0 - Buscando Factibilidad";
				printf("%s\n", mensajeDeError.c_str());
				result = -12;
				return;
		}
	}

	while (res == 1) {
		res = darpaso(simplex, cnt_columnasFijadas, simplex.cnt_RestriccionesRedundantes);
		if (res == -1) {
			mensajeDeError = "Error -- NO encontramos pivote bueno dando paso";
			printf("%s\n", mensajeDeError.c_str());
			result = -21;
			return;
		}
	}
	  
	if (res == 2) {
		goto lbl_inicio;
	}
		
	result = res;
	printf("%s: %d\n", "Finish, result = ", result);
}


// Si abs( cotasup - cotainf ) < AsumaCeroCaja then flg_x := 2
// retorna la cantidad de cajas fijadas.
int fijarCajasLaminares(TSimplexGPUs &smp, int &cnt_varfijas) {
	int res = 0;
	
	for (int i =  0; i < smp.NVariables; i++) { // MAP: = for 1 to nc-1
		// printf("in fijarCajasLaminares: i, flg_x[i], x_inf[i], x_sup[i] = %d, %d, %g, %g \n", i, smp.flg_x[i], smp.x_inf[i], smp.x_sup[i]);
		if (abs(smp.flg_x[i] ) < 2) { // No se aplica ni para 2 ni para 3
			if (abs(smp.x_sup[i] - smp.x_inf[i] ) < CasiCero_CajaLaminar) {
				if (smp.flg_x[i] < 0) {
					smp.flg_x[i] = -2;
				} else {
					smp.flg_x[i] = 2;
				}
				cnt_varfijas++;
				res++;
			}
		}
	}
    return res;
}


void posicionarPrimeraLibre(TSimplexGPUs &smp, int cnt_varfijas, int &cnt_fijadas, int &cnt_columnasFijadas, int &kPrimeraLibre) {
    while ((cnt_fijadas < cnt_varfijas) && 
		(((smp.top[kPrimeraLibre] < 0) && (abs(smp.flg_x[-smp.top[kPrimeraLibre]]) == 2)) || ((smp.top[kPrimeraLibre] > 0) && (abs(smp.flg_y[smp.top[kPrimeraLibre]]) == 2)))) {
		if (smp.top[kPrimeraLibre] < 0) {
			cnt_fijadas++;
		}	
		cnt_columnasFijadas++;
		kPrimeraLibre--;
	}
}


bool fijarVariables(TSimplexGPUs &smp, int cnt_varfijas, int &cnt_columnasFijadas, int cnt_RestriccionesRedundantes) {

	int kColumnas, mejorColumnaParaCambiarFila, kFor, kFilaAFijar, cnt_fijadas, kPrimeraLibre;
	double mejorAkFilai;
	bool buscando, pivoteoConUnaFijada;

	if (cnt_varfijas > 0) {
		cnt_fijadas = 0;
		kPrimeraLibre = smp.NVariables - 1; // MAP: nc - 1, - 1 debido al cambio de indices
		kColumnas = 0; // MAP: Antes 1, pero cambimos que los vectores esten indexados desde 1 a 0
		
		while ((cnt_fijadas < cnt_varfijas) && (kColumnas <= kPrimeraLibre)) {
			posicionarPrimeraLibre(smp, cnt_varfijas, cnt_fijadas, cnt_columnasFijadas, kPrimeraLibre);
			//Busco en columnas
			if ((cnt_fijadas < cnt_varfijas) && (kColumnas <= kPrimeraLibre)) {
				buscando = true;
				while (buscando && (kColumnas <= kPrimeraLibre)) {
					
					// MAP: Aca grego -1 a flg_x[-top[kColumnas] -1] para obtener el indice que empieza en 0 y top left son 2 vectores indexados en 0 pero que contienen numeros comenzados en 1, por lo tanto el indice real es top[indice] - 1,
					// esto lo voy a tener que hacer a lo largo y ancho del algoritmo
					if ((smp.top[kColumnas] < 0) && (abs(smp.flg_x[-smp.top[kColumnas] - 1]) == 2)) { 
						// Es una x fija
						buscando = false;
					} else {
						kColumnas++;
					}
				}
				if (!buscando) {
					intercambioColumnas(smp, kColumnas, kPrimeraLibre);
					kPrimeraLibre--;
					cnt_fijadas++;
					cnt_columnasFijadas++;
					kColumnas++;
				}
			}
		}

		// Se inicializa en la fila anterior a la ultima fijada
		kFilaAFijar = cnt_RestriccionesRedundantes - 1; // MAP: Agreo -1
		while (cnt_fijadas < cnt_varfijas) {
		
			posicionarPrimeraLibre(smp, cnt_varfijas, cnt_fijadas, cnt_columnasFijadas, kPrimeraLibre);
			if (cnt_fijadas < cnt_varfijas) {
		 
				//Busco en filas
				for (kFor = kFilaAFijar + 1; kFor < smp.NVariables - 1; kFor++) { // MAP: change index in loop
					if ((smp.left[kFor] < 0) && (abs(smp.flg_x[-smp.left[kFor] - 1]) == 2)) {
						kFilaAFijar = kFor;
						break;
					}
				}

				mejorColumnaParaCambiarFila = 0; // MAP: antes 1
				mejorAkFilai = abs(smp.mat[kFilaAFijar*(smp.NVariables + 1)]); // MAP: before [1], primera columna ahora es 0, [kFilaAFijar][0]
				for (kColumnas = 1; kColumnas < kPrimeraLibre; kColumnas++) { // MAP: Antes 2 to kPrimeraLibre
					if (abs(smp.mat[kFilaAFijar * (smp.NVariables + 1) + kColumnas]) > mejorAkFilai) {
						mejorColumnaParaCambiarFila = kColumnas;
						mejorAkFilai = abs(smp.mat[kFilaAFijar * (smp.NVariables + 1) + kColumnas]);
					}
				}

				pivoteoConUnaFijada = false;
				// dv@20191226 Si el término independiente es nulo, la "restricción" es reduntante
				// entonces la variable ya había quedado fijada
				if (mejorAkFilai < AsumaCero) {
					printf("%s: %g, %g\n", "fijarVariables false  mejorAkFilai < AsumaCero: ", mejorAkFilai, AsumaCero);
					return false;
				}

				intercambiar(smp, kFilaAFijar, mejorColumnaParaCambiarFila);
				cnt_fijadas++;
				
				if (!pivoteoConUnaFijada) {
					// dv@20200115 agrego esto porque no debería cambiar si pivoteó con una fijada
					if (mejorColumnaParaCambiarFila != kPrimeraLibre) {
						intercambioColumnas(smp, mejorColumnaParaCambiarFila, kPrimeraLibre);
					}
					cnt_columnasFijadas++;
					kPrimeraLibre--;
				} else {
					if (mejorColumnaParaCambiarFila != kPrimeraLibre + 1) { // En el caso de que se pivotee con una columna fija no cambia kPrimeraLibre
						intercambioColumnas(smp, mejorColumnaParaCambiarFila, kPrimeraLibre + 1);
					}
				}
			}
		}
	}
	
	return true;

}


bool intercambiar(TSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int k, j;

	piv = smp.mat[kfil * (smp.NVariables + 1) + jcol];
	invPiv = 1 / piv;

	// MAP: POSIBLE MEJORA, en GPU se encargaran hilos diferentes, recorrida por filas
	// MAP: En el codigo original se separa en 4 casos el mismo codigo, se recorre desde 1 to cnt_RestriccionesRedundantes to kfil - 1  <skipping kfil (fila pivot)> to nf - 1 to nf
	// MAP: Se concatenan estos casos en un mismo for
	for (k = 0; k < kfil; k++) {
		m = -smp.mat[k * (smp.NVariables + 1) + jcol] * invPiv;
		if (abs(m) > 0) {
			for (j = 0; j < jcol; j++) {
				smp.mat[k * (smp.NVariables + 1) + j] = smp.mat[k * (smp.NVariables + 1) + j] + m * smp.mat[kfil * (smp.NVariables + 1) + j]; 
			}
			
			smp.mat[k * (smp.NVariables + 1) + jcol] = -m;
			
			for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes "to nc"
				smp.mat[k * (smp.NVariables + 1) + j] = smp.mat[k * (smp.NVariables + 1) + j] + m * smp.mat[kfil * (smp.NVariables + 1) + j];
			}
		} else {
			smp.mat[k * (smp.NVariables + 1) + jcol] = 0; // IMPONGO_CEROS
		}
	}
	
	// Salteo la fila kfil
	
	for (k = kfil + 1; k <= smp.NRestricciones; k++) { // MAP: <= pq considera la fila de la funcion z
		m = -smp.mat[k * (smp.NVariables + 1) + jcol] * invPiv;
		if (abs(m) > 0) {
			for (j = 0; j < jcol; j++) {
				smp.mat[k * (smp.NVariables + 1) + j] = smp.mat[k * (smp.NVariables + 1) + j] + m * smp.mat[kfil * (smp.NVariables + 1) + j]; 
			}
			
			smp.mat[k * (smp.NVariables + 1) + jcol] = -m;
			
			for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes "to nc"
				smp.mat[k * (smp.NVariables + 1) + j] = smp.mat[k * (smp.NVariables + 1) + j] + m * smp.mat[kfil * (smp.NVariables + 1) + j];
			}
		} else {
			smp.mat[k * (smp.NVariables + 1) + jcol] = 0; // IMPONGO_CEROS
		}
	}

	// Completo la fila kfil
	m = -invPiv;
	for (j = 0; j < jcol; j++) {
		smp.mat[kfil * (smp.NVariables + 1) + j] = smp.mat[kfil * (smp.NVariables + 1) + j] * m;
	}

	smp.mat[kfil * (smp.NVariables + 1) + jcol] = -m;
	
	for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes jcol + 1 to nc
		smp.mat[kfil * (smp.NVariables + 1) + j] = smp.mat[kfil * (smp.NVariables + 1) + j] * m;
	}

	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil];
	smp.left[kfil] = k;

	// actualizo_iitop(jcol);
	// actualizo_iileft(kfil);

	return true;
 }

/*
// MAP: Agrego los -1 +1 por el cambio de indices CHEQUEAR QUE ES CORRECTO
void actualizo_iitop(TSimplexGPUs &smp, int k) {
	// actualizo los indices iix e iiy
	if (smp.top[k] < 0) {
		smp.iix[-smp.top[k] - 1] = k + 1;
	} else {
		smp.iiy[smp.top[k] - 1] = -k - 1;
	}
}

// MAP: Agrego los -1 +1 por el cambio de indices CHEQUEAR QUE ES CORRECTO
void actualizo_iileft(TSimplexGPUs &smp, int k) {
	// actualizo los indices iix e iiy
	if (smp.left[k] > 0) {
		smp.iiy[smp.left[k] - 1]  = k + 1; 
	} else {
		smp.iix[-smp.left[k] - 1]  = -k - 1;
	}
}
*/

void intercambioColumnas(TSimplexGPUs &smp, int j1, int j2) {

	int k;
	double m;

	for (k = 0; k <= smp.NRestricciones; k++) { // MAP: Intercambia toda la columna por lo tanto va hasta smp.NRestricciones inclusive
		m = smp.mat[k * (smp.NVariables + 1) + j1];
		smp.mat[k * (smp.NVariables + 1) + j1]  = smp.mat[k * (smp.NVariables + 1) + j2];
		smp.mat[k * (smp.NVariables + 1) + j2]  = m;
	}

	k = smp.top[j1];
	smp.top[j1] = smp.top[j2];
	smp.top[j2] = k;

	// actualizo_iitop(smp, j1);
	// actualizo_iitop(smp, j2);
}

void intercambioFilas(TSimplexGPUs &smp, int k1, int k2) {
  
	int j;
	double m;

	for (j = 0; j <= smp.NVariables; j++) { // MAP: Intercambia toda la fila por lo tanto va hasta smp.NVariables inclusive
		m = smp.mat[k1 * (smp.NVariables + 1) + j];
		smp.mat[k1 * (smp.NVariables + 1) + j] = smp.mat[k2 * (smp.NVariables + 1) + j];
		smp.mat[k2 * (smp.NVariables + 1) + j] = m;
	}

	j = smp.left[k1];
	smp.left[k1] = smp.left[k2];
	smp.left[k2] = j;

	// actualizo_iileft(smp, k1);
	// actualizo_iileft(smp, k2);

}

 // Atención esto funciona porque suponemos que el Simplex está en su estado Natural.
 // a lo sumo se conmutaron algunas columnas para fijar variables.
 // MAP: se usa solo dentro de enfilarVariablesLibres
int buscarMejorPivoteEnCol(TSimplexGPUs &smp, int jCol, int iFilaFrom, int iFilaHasta) {
	
	double a;
	int iFil, res;
	
    res = -1;
    a = 0.0;
	for (iFil = iFilaFrom; iFil <= iFilaHasta; iFil++) { // MAP: Originalmente iFilaFrom to iFilaHasta, eso no cambia ya que se le pasan los indices ya adaptados a la indexacion desde 0
		if (abs(smp.mat[iFil * (smp.NVariables + 1) + jCol]) > a) {
			a = abs(smp.mat[iFil * (smp.NVariables + 1) + jCol]);
			res = iFil;
		}
	}
	return res;
 }


bool enfilarVariablesLibres(TSimplexGPUs &smp, int cnt_columnasFijadas, int &cnt_RestriccionesRedundantes, int &cnt_variablesLiberadas) {

	int jVar, iFil, jCol;

	for (jCol = 0; jCol < smp.NVariables -  cnt_columnasFijadas; jCol++) { // MAP: Antes 1 to nc - 1 - cnt_columnasFijadas, nc - 1 = NVariables
		jVar = -smp.top[jCol];

		if ((jVar > 0) && (smp.flg_x[jVar - 1] == 3)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0
			iFil = buscarMejorPivoteEnCol(smp, jCol, cnt_RestriccionesRedundantes, smp.NRestricciones - 1); // MAP: Modifico indices paso el indice de filaFrom y filaHasta (inclusive)
			if (iFil < 1) {
				return false;
			}
			intercambiar(smp, iFil, jCol);
			cnt_RestriccionesRedundantes++;
			if (iFil > cnt_RestriccionesRedundantes) {
				intercambioFilas(smp, cnt_RestriccionesRedundantes, iFil);
			}
			cnt_variablesLiberadas++;
		}
	}
  
	return true;
}


int resolverIgualdades(TSimplexGPUs &smp, int &cnt_columnasFijadas, int cnt_varfijas, int &cnt_RestriccionesRedundantes, int cnt_igualdades) {
	int res, cnt_acomodadas, iFila, iColumna, 
		nIgualdadesResueltas, nIgualdadesAResolver, 
		iFilaLibre, iFilaAcomodando;
	int * nCerosFilas;
	int * nCerosCols;
	bool fantasma;
	string mensajeDeError;

	cnt_acomodadas = cnt_columnasFijadas - cnt_varfijas;

	// Muevo las igualdades que esten en columnas al lado derecho junto con las FIJADAS
	iColumna = smp.NVariables - cnt_columnasFijadas - 1; // MAP: Indice modificado, cambio nc por smp.NVariables 
	while ((iColumna >= 0) && (cnt_acomodadas < cnt_igualdades)) {
		if ((smp.top[iColumna] > 0) && (abs(smp.flg_y[smp.top[iColumna] - 1]) == 2)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0
			if (iColumna != (smp.NVariables - cnt_columnasFijadas - 1)) { // MAP: Indice modificado, cambio nc por smp.NVariables
				intercambioColumnas(smp, iColumna, smp.NVariables - cnt_columnasFijadas - 1); // MAP: Indice modificado, cambio nc por smp.NVariables
			}
			cnt_acomodadas++;
			cnt_columnasFijadas++;
		}
		iColumna--;
	}
	
	// rch@20130307.bugfix - begin ----------------------------
	// ahora reviso las que ya estén declaradas como redundantes a ver si hay
	// igualdades e incremento el contador de acomodadas.
	for (iFilaAcomodando = 0; iFilaAcomodando < cnt_RestriccionesRedundantes; iFilaAcomodando++) { // MAP: originalmente 1 to cnt_RestriccionesRedundantes
		if ((smp.left[iFilaAcomodando] > 0) && (abs(smp.flg_y[iFilaAcomodando]) == 2)) {
			//  Es una restricción  y es de igualdad
			cnt_acomodadas++;
		}
	}

	// En el caso de estar resolviendo un MIPSimplex, en contador de redundantes puede venir
	// incrementado del problema PADRE y pueden haber restricciones de igualdad dentro de las
	// redundantes. Esto hacía que el  "while  cnt_acomodadas < cnt_igualdades"
	// que está unas lineas abajo NO saliera por no alcanzar la condición.
	// rch@20130307.bugfix - end ----------------------------


	// ahora reordeno las igualdades y las que queden en filas las pongo al inicio
	iFilaLibre = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0
	iFilaAcomodando = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0

	while (cnt_acomodadas < cnt_igualdades) {
		if ((smp.left[iFilaAcomodando] > 0) && (abs(smp.flg_y[smp.left[iFilaAcomodando] - 1]) == 2)) { // MAP: Agrego - 1 para mover el indice a la indexacion desde 0
			//vc, dv @20200116 decia flg_y[iFilaAcomodando], se fijaba en cualquier lado
			//  Es una restricción  y es de igualdad
			if (iFilaLibre != iFilaAcomodando) {
				intercambioFilas(smp, iFilaAcomodando, iFilaLibre);
			}
			cnt_acomodadas++;
			iFilaLibre++;
		}
		iFilaAcomodando++;
	} //Al salir de aca iFilaLibre queda en la primer fila que no es de igualdad
	
	res = 1;

	nIgualdadesResueltas = 0;
	nIgualdadesAResolver = iFilaLibre - cnt_RestriccionesRedundantes; // MAP: Antes iFilaLibre - (cnt_RestriccionesRedundantes + 1),  + 1 debido a que iFilaLibre es uno menos que el original
	nCerosFilas = (int*)malloc((smp.NRestricciones + 1)*sizeof(int)); // MAP: antes setLength(nCerosFilas, nf);
	nCerosCols = (int*)malloc((smp.NVariables + 1)*sizeof(int));// MAP: antes setLength(nCerosCols, nc);
	while (nIgualdadesResueltas < nIgualdadesAResolver) {
		//    res:= pasoBuscarFactibleIgualdad( cnt_RestriccionesRedundantes + 1 + nIgualdadesResueltas );
		//    res:= pasoBuscarFactibleIgualdad2( cnt_RestriccionesRedundantes + 1 + nIgualdadesResueltas );
		//    res:= pasoBuscarFactibleIgualdad3( nIgualdadesAResolver - nIgualdadesResueltas);

		// printf("Launch  pasoBuscarFactibleIgualdad4 with:  %d, %d, %d, %d\n", nCerosFilas[0], nCerosCols[0], cnt_columnasFijadas, cnt_RestriccionesRedundantes);
		res = pasoBuscarFactibleIgualdad4(smp, nIgualdadesAResolver - nIgualdadesResueltas, nCerosFilas, nCerosCols, cnt_columnasFijadas, cnt_RestriccionesRedundantes);
		
		if (res ==1) {
			nIgualdadesResueltas = nIgualdadesResueltas + 1;
			cnt_columnasFijadas++;
		} else {
			iFila = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0
			while ((nIgualdadesResueltas < nIgualdadesAResolver) && filaEsFactible(smp, iFila, fantasma)) {
				if (iFila != cnt_RestriccionesRedundantes + 1) {
					intercambioFilas(smp, iFila, cnt_RestriccionesRedundantes + 1);
				}
				cnt_RestriccionesRedundantes++;
				nIgualdadesResueltas++;
				iFila++;
			}
			if (nIgualdadesResueltas < nIgualdadesAResolver) {
				mensajeDeError = "PROBLEMA INFACTIBLE - Resolviendo igualdades.";
				printf("%s\n", mensajeDeError.c_str());
				res = -13;
				break;
			} else {
				res = 1;
			}
		}
	}

	free(nCerosFilas); // MAP: Antes setLength(nCerosFilas, 0);
	free(nCerosCols); // MAP: Antes setLength(nCerosCols, 0);
	return res;
}


// Indica si la restricción en kfila esta siendo cumplida
bool filaEsFactible(TSimplexGPUs &smp, int kfila, bool &fantasma) {
	
	int ix;
	// if e(kfila, nc) < -CasiCero_Simplex then
	if (smp.mat[kfila * (smp.NVariables + 1) + smp.NVariables] < -CasiCero_Simplex) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		// Si la fila es < 0
		fantasma = false;
		return false;
	} else if (smp.left[kfila] < 0) {
		// Si rval es >= 0 reviso si es una variable con cota superior
		ix = -smp.left[kfila];
		// if (flg_x[ix] <> 0) and (e(kfila, nc) > (x_sup.pv[ix] + CasiCero_Simplex_CotaSup)) then
		if ((smp.flg_x[ix - 1] != 0) && ((smp.mat[kfila * (smp.NVariables + 1) + smp.NVariables]) > (smp.x_sup[ix - 1] + CasiCero_Simplex_CotaSup))) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice y agrego -1 para correr el indice
			//Si violo la cota superior
			fantasma = true;
			return false;
		} else {
			fantasma = false;
			return true;
		}
	} else {
		// Si es una y >= 0
		fantasma = false;
		return true;
	}
}


int pasoBuscarFactibleIgualdad4(TSimplexGPUs &smp, int nIgualdadesNoResueltas, int * nCerosFilas, int * nCerosCols, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes) {

	int iFila, iColumna, columnasLibres,
		filaPiv, colPiv;
	double  maxVal, m;
	
	// Tengo todas las igualdades en columnas al final y las igualdades en filas al principio
	columnasLibres = smp.NVariables - cnt_columnasFijadas; // MAP: remuevo -1 ya que smp.NVariables = nc - 1 
	for (iColumna = 0; iColumna < columnasLibres; iColumna++) { // MAP: Originalmente 1 to columnasLibres
		nCerosCols[iColumna] = 0;
	}
	
	
	// Busco el máximo valor absoluto y cuento la cantidad de ceros en filas y columnas en la caja desde
	// cnt_RestriccionesRedundantes + 1 hasta cnt_RestriccionesRedundantes + nIgualdadesNoResueltas
	// la caja de las igualdades sin resolver
	maxVal = -MaxNReal;
	filaPiv = -1;
	colPiv = -1;
	// MAP: remuevo el +1 debido a la nueva indexacion desde 0, originalmente cnt_RestriccionesRedundantes + 1 to cnt_RestriccionesRedundantes + nIgualdadesNoResueltas
	for (iFila = cnt_RestriccionesRedundantes; iFila < cnt_RestriccionesRedundantes + nIgualdadesNoResueltas; iFila++) { 
		for (iColumna = 0; iColumna <columnasLibres; iColumna++) { // MAP: muevo el indice una lugar hacia atras para considerar el cambio de indexacion desde 0
			m = abs(smp.mat[iFila * (smp.NVariables + 1) + iColumna]);
			if (m < AsumaCero) {
				nCerosFilas[iFila]++;
				nCerosCols[iColumna]++;
			} else if (m > maxVal) {
				maxVal = m;
				filaPiv = iFila;
				colPiv = iColumna;
			}
		}
	}
	
	// Termino de contar la cantidad de ceros en columnas con el resto de las filas
	for (iFila = cnt_RestriccionesRedundantes + nIgualdadesNoResueltas; iFila < smp.NRestricciones; iFila++) { // MAP: cnt_RestriccionesRedundantes + nIgualdadesNoResueltas + 1 to nf - 1
		for (iColumna = 0; iColumna < columnasLibres - 1; iColumna++) { // MAP: Originalmente 1 to columnasLibres - 1
			if (abs(smp.mat[iFila * (smp.NVariables + 1) + iColumna]) < AsumaCero) {
				nCerosCols[iColumna]++;
			}
		}
	}
	
	if (maxVal > CasiCero_Simplex) {
		for (iFila = cnt_RestriccionesRedundantes; iFila < cnt_RestriccionesRedundantes + nIgualdadesNoResueltas; iFila++) { // MAP: Originalmente cnt_RestriccionesRedundantes + 1 to cnt_RestriccionesRedundantes +	nIgualdadesNoResueltas
			for (iColumna = 0; iColumna < columnasLibres; iColumna++) { // MAP: Originalmente 1 to columnasLibres
				if (abs(smp.mat[iFila * (smp.NVariables + 1) + iColumna]) * 10 >= maxVal) {
					// Lo considero como posible pivote
					if ((nCerosFilas[filaPiv] + nCerosCols[colPiv]) < (nCerosFilas[iFila] + nCerosCols[iColumna])) {
						filaPiv = iFila;
						colPiv = iColumna;
					}
				}
			}
		}
		
		// printf("In pasoBuscarFactibleIgualdad4 %d, %d, %d \n", filaPiv, cnt_RestriccionesRedundantes, nIgualdadesNoResueltas);
		// Muevo la fila a intercambiar al final asi me siguen quedando las que voy a acomodar en bloque desde cnt_RestriccionesRedundantes
		if (filaPiv != cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1) { // MAP: Agrego -1 para correr el indice, filaPiv ya esta en el indice correcto
			intercambioFilas(smp, filaPiv, cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1); // MAP: Agrego -1 para correr el indice, filaPiv ya esta en el indice correcto
		}

		intercambiar(smp, cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1, colPiv); // MAP: Agrego -1 para correr el indice, colPiv ya esta en el indice correcto

		if (colPiv != columnasLibres) {
			intercambioColumnas(smp, colPiv, columnasLibres - 1); // MAP: Agrego -1 para correr el indice, colPiv ya esta en el indice correcto
		}
		return 1;
	} else {
		return -1;
	}	
}

int reordenarPorFactibilidad(TSimplexGPUs &smp, int cnt_RestriccionesRedundantes, int &cnt_RestrInfactibles) {

	int kfil, ix;
	double rval;

	/*
		Primero recorremos las restricciones y
		si la restricción no está violada me fijo si corresponde a una variable
		con restricción de cota superior y si es así verificamos que tampoco esté
		violada la restricción fantasma, si la fantasma se viola hacemos el cambio
		de variable para volverla explícita
	*/
	for (kfil = cnt_RestriccionesRedundantes; kfil < smp.NRestricciones; kfil++) { // MAP: originalmente cnt_RestriccionesRedundantes + 1 to nf - 1
		// rval := e(kfil, nc);
		rval = smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		if (rval > 0) {
		  // Si es = 0 no chequeo pues la fantasma no puede estar violada
			if (smp.left[kfil] < 0) {
				ix = -smp.left[kfil] - 1; // MAP: Agrego -1 para ajustar el indice a la indexacion desde 0
				if ((smp.flg_x[ix] != 0) && (smp.x_sup[ix] < rval)) {
					// Parece que violo la cota superior
					if ((smp.x_sup[ix] + CasiCero_Simplex_CotaSup) < rval) {
						// La viola realmente
						cambiar_borde_de_caja(smp, kfil);
					} else {
						// La viola por errores númericos
						// pon_e(kfil, nc, x_sup.pv[ix])
						smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables] = smp.x_sup[ix]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
					}
				}
			}
		} else {
			if (rval > -CasiCero_Simplex_CotaSup) {
			  // pon_e(kfil, nc, 0);
			  smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables] = 0; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
			}
		}
	}

	// Ahora sabemos que las violadas están explícitas, movemos todas las
	//restricciones violadas al final
	kfil = cnt_RestriccionesRedundantes; // MAP: Remuevo + 1 para ajustar el indice a la indexacion desde 0
	cnt_RestrInfactibles = 0;
	while (kfil < (smp.NRestricciones - cnt_RestrInfactibles)) { // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
		// rval:= e(kfil, nc);
		rval = smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		if (rval < 0) {
			cnt_RestrInfactibles++;
			// while (e(nf-cnt_RestrInfactibles, nc ) < 0)
			// MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice, idem para nc	
			while ((smp.mat[(smp.NRestricciones - cnt_RestrInfactibles) * (smp.NVariables + 1) + smp.NVariables] < 0) && (kfil < (smp.NRestricciones - cnt_RestrInfactibles))) {
				cnt_RestrInfactibles++;
			}
			if (kfil < (smp.NRestricciones - cnt_RestrInfactibles)) { // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
				intercambioFilas(smp, kfil, smp.NRestricciones - cnt_RestrInfactibles); // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
			}
		}
		kfil++;
	}
	return cnt_RestrInfactibles;
}

void cambiar_borde_de_caja(TSimplexGPUs &smp, int k_fila) {
	int ix, k;
	/*
		Realizamos el cambio de variable x'= x_sup - x para que la restricción
		violada sea representada por x' >= 0
		Observar que para la nueva variable la restricción x >= 0 se transforma
		en x' <= x_sup. Es decir que la cota superior de x' es también x_sup.
	*/
	
	// MAP: Confio en el comentario debajo que eso da positivo por lo que resto 1 para ajustar el indice a la indexacion desde 0, de hecho si diera negativo como se trata de un indice fallaria
	ix = -smp.left[k_fila] - 1; // Se supone que esto da positivo, sino no es una x
	for (k = 0; k < smp.NVariables; k++) { // MAP: Muevo los indices 1 to nc-1 un lado a la izquierda para para ajustar el indice a la indexacion desde 0
		smp.mat[k_fila * (smp.NVariables + 1) + k] = -smp.mat[k_fila * (smp.NVariables + 1) + k];
	}
	
	smp.mat[k_fila * (smp.NVariables + 1) + smp.NVariables] = smp.x_sup[ix] - smp.mat[k_fila * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice

	if (abs(smp.flg_x[ix]) != 1) {
		printf("%s\n", "mmmm ... porqué?");
	}
	smp.flg_x[ix] = -smp.flg_x[ix];
  
}


int pasoBuscarFactible(TSimplexGPUs &smp, int &cnt_RestrInfactibles, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes) {
	
	int pFilaOpt, ppiv, qpiv, ix, res;
	double rval;
	bool filaFantasma, colFantasma;

	pFilaOpt = smp.NRestricciones - cnt_RestrInfactibles; // MAP: Antes nf - cnt_RestrInfactibles, indice ajustado
	// rval:= e(pFilaOpt, nc);
	rval = smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice

	/* OJO LE AGREGO ESTE CHEQUEO PARA PROBAR */
	// Si parece satisfecha verifico que no se esté violándo la  fantasma
	if (rval > 0) {
		if (smp.left[pFilaOpt] < 0) {
			ix = -smp.left[pFilaOpt] - 1; // MAP: Ajusto indice con -1 
			if ((smp.flg_x[ix] != 0) && (rval > smp.x_sup[ix])) {
				if (rval > smp.x_sup[ix] + CasiCero_Simplex) {
					cambiar_borde_de_caja(smp, pFilaOpt);
					// rval:= e(pFilaOpt, nc );
					rval = smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
				} else {
					// pon_e(pFilaOpt, nc, x_sup.pv[ix]);
					smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables] = smp.x_sup[ix]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
					rval = smp.x_sup[ix];
				}
			}
		}
	} else if (rval > -CasiCero_Simplex) {
		// pon_e(pFilaOpt, nc, 0);
		smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables] = 0; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		rval = 0;
	}

	if (rval >= 0) {
		// ya es factible, probablemente se arregló con algún cambio anterior.
		cnt_RestrInfactibles--;
		res = 1;
	} else {
		// Nos planteamos el problema de optimización con objetivo el valor de la restricción violada
		if (cnt_RestrInfactibles > 0) {
			qpiv = locate_zpos(smp, pFilaOpt, cnt_columnasFijadas);
			if (qpiv >= 0) { // MAP: Antes > 0, pero con el corrimiento de indice queda >=
				ppiv = mejorpivote(smp, qpiv, pFilaOpt, filaFantasma, colFantasma, true, cnt_RestriccionesRedundantes);
				if (ppiv < 0) { // MAP: Antes < 1, pero con el corrimiento de indice queda < 0
					res = -1; // ShowMessage('No encontre pivote bueno ');
				} else {
					if (!colFantasma) {
						intercambiar(smp, ppiv, qpiv);
						if (filaFantasma) {
							cambio_var_cota_sup_en_columna(smp, qpiv);
						}
						// if ( e( pFilaOpt, nc) >= 0 ) then
						if (smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables] >= 0) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
							cnt_RestrInfactibles--;
						}
						res = 1;
					} else {
						cambio_var_cota_sup_en_columna(smp, ppiv);
						// if ( e( pFilaOpt, nc) >= 0 ) then
						if (smp.mat[pFilaOpt * (smp.NVariables + 1) + smp.NVariables] >= 0) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
							cnt_RestrInfactibles--;
						}
						res = 1;
					}
				}
			} else {
				res = 0; // ShowMessage('No encontre z - positivo ' );
			}
		} else {
			res = -2;
		}

		if (res == -1) {
			// Pruebo si soluciono la infactibildad con un intercambio de la infactible con una de las Activas
			qpiv = locate_qOK(smp, pFilaOpt, smp.NVariables - cnt_columnasFijadas - 1, smp.NVariables, cnt_RestriccionesRedundantes); // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
			if (qpiv > 0) {
				intercambiar(smp, pFilaOpt, qpiv);
				cnt_RestrInfactibles--;
				res = 1;
			}
		}
	}

	return res;
}


// Buscamos la columna que en la ultima fila (fila z) tenga el valor positivo mas grande retorna el número de columna si lo encontro, -1 si son todos < 0
// Este paso se da en el Simplex para minimizar, en el de maximizar busca el menos negativo
int locate_zpos(TSimplexGPUs &smp, int kfila_z, int cnt_columnasFijadas) {
	int j, ires;
	double maxval;
	ires = -1;
	maxval = CasiCero_Simplex;
	for (j = 0; j <= smp.NVariables - cnt_columnasFijadas; j++) { // MAP: Antes 1 to nc - 1 - cnt_columnasFijadas
		if (smp.mat[kfila_z * (smp.NVariables + 1) + j] > maxval) {
			maxval = smp.mat[kfila_z * (smp.NVariables + 1) + j];
			ires = j;
		}
	}
	return ires;
}


// MAP: Este procedimeinto es interno al procedimiento mejorpivote. ESTE PROCEDIMIENTO HABRIA QUE ELIMINARLO Y PEGAR EL CODIGO DIRECTO EN mejorpivote
void capturarElMejor(double &a_iq_DelMejor, double &a_it_DelMejor, int &p, bool &filaFantasma, bool &colFantasma, double a_iq, double a_it, int i, bool xFantasma_fila) {
  a_iq_DelMejor = a_iq;
  a_it_DelMejor = a_it;
  p = i;
  filaFantasma = xFantasma_fila;
  colFantasma = false; // Solo por si el primero era la Fantasma de la Columna
}


int mejorpivote(TSimplexGPUs &smp, int q, int kmax, bool &filaFantasma, bool &colFantasma, bool checkearFilaOpt, int cnt_RestriccionesRedundantes) {
  
	int i, p, ix;
	double a_iq, a_it, abs_a_pq,
		a_iq_DelMejor, a_it_DelMejor, abs_a_pq_DelMejor;
	bool xFantasma_fila, esCandidato;

	// inicializaciones no necesarias, solo para evitar el warning
	a_it = 0;
	xFantasma_fila = false;
	a_it_DelMejor = 0;
	a_iq_DelMejor = 1;

	/*11/9/2006 le voy a agregar para que si la q corresponde a una x con manejo
	de cota superior considere la existencia de una fila adicional correspondiente
	a la cota superior.
	Dicha fila tiene un -1 en la coluna q y el valor x_sup como término independiente

	rch.30/3/2007 Agrego el manejo del CasiCero_Simplex

	PA.21/06/2007 Le agrego que al buscar el mejorpivote para optimizar la fila
	kmax chequee si esta es una variable con restriccion de cota superior y que el
	pivote elegido no la viole*/

	ix = -smp.top[q];
	if ((ix > 0) && ( abs(smp.flg_x[ix - 1] ) == 2)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0
		printf("%s\n", "OPATROPA!");
	}

	if ((ix > 0) and ( abs(smp.flg_x[ix - 1] ) == 1 )) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
		// en la columna q hay una x con manejo de cota superior
		colFantasma = true;
		p = q;
		//l o fijamos en -1 porque todas las restricciones fantasma tienen un -1 en
		// en el coeficiente de la variable y x_sup como termino independiente
		a_iq_DelMejor = -1;
		a_it_DelMejor = smp.x_sup[ix - 1]; // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
		abs_a_pq_DelMejor = 1;
	} else {
		p = -1;
		colFantasma = false;
	}

	filaFantasma = false;

	for (i = cnt_RestriccionesRedundantes; i <= kmax - 1; i++) { // MAP: Antes cnt_RestriccionesRedundantes + 1 to kmax - 1, muevo indice para ajustar a la nueva indexacion desde 0, kmax ya viene indexado con el ejuste
		// b(i) >= 0 para todo i / cnt_RestriccionesRedundantes < i < kmax-1
		// Buscamos la fila i que tenga el maximo b(i)/a(i,q) con a(i,q) < 0
		// aiq:= e( i, q );
		a_iq = smp.mat[i * (smp.NVariables + 1) + q];
		if (a_iq >  CasiCero_Simplex) { // Si es positivo, verificamos si se trata de una x y entonces agregamos la fantasma.
			ix = -smp.left[i];
			if ((ix > 0) && (abs(smp.flg_x[ix - 1]) ==1)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
				// La variable en la fila i tiene cota superior, hay que probar con el cambio de variable
				a_iq = -a_iq;
				a_it = smp.x_sup[ix] - smp.mat[i * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
				xFantasma_fila = true;
				esCandidato = true;
			} else {
				esCandidato = false;
			}
		} else if (a_iq < -CasiCero_Simplex) { // Si es negativo es candidato
			a_it = smp.mat[i * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
			esCandidato = true;
			xFantasma_fila = false;
		} else {
			esCandidato = false;
		}


		if (esCandidato) { // Considero el coeficiente para elegir el pivote
			abs_a_pq = abs(smp.mat[i * (smp.NVariables + 1) + q]); // MAP: Cambio abs( e( i, q ) ); por abs(smp.mat[i][q]));
			/*
			se supone que a_iq < 0 y a_iq_DelMejor < 0 pues sino no son candidatos.
			El término independiente de cualquier fila k, se transformará al usar a_iq como pivote
			como:
			a_k,nc = a_k,nc - a_kq / a_iq * a_i,nc
			y tiene que mantenerce >= 0 para cualquier k <> i.
			a_k,nc - a_kq / a_iq * a_i,nc >= 0  ec.(1)
			dividiendo por a_kq < 0 se tiene
			a_k,nc / a_kq <= a_i,nc / a_iq  ec.(2)
			observar que cada lado de la desigualdad depende solo de los coeficientes
			de la fila k (izquieda) o de la fila i (derecha), Esto nos permite
			ir recorriendo las filas con a_iq < 0 y quedarnos con el de mayor cociente
			a_i,nc/a_iq.

			Para no hacer las divisiones, en lugar de chequear la ec.(2), chequeamos
			la ec.(3) obtenida de la ec.(1) multiplicando por a_iq < 0
			a_k,nc * a_iq <= a_kq  * a_i,nc >= 0  ec.(3)

			Si se cumple la ec.3 a_iq es mejor pivote que a_kp.
			*/
			//aiq < 0 por como lo tomamos para esCandidato
			//bi >= 0 para todo i / cnt_RestriccionesRedundantes < i < kmax-1
			//El pivote es aquel que tenga mayor bi/aiq siempre que bi/aiq < 0 y aiq < 0
			//Ademas bi/aiq y b_max/a_max tienen el mismo signo =>
			//bi/aiq > b_max/a_max <=> bi * a_max > b_max * aiq
			if (p < 0) { // Es el primer candidato
				capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
				
			} else if (( a_it_DelMejor * a_iq) < ( a_it * a_iq_DelMejor ))  {
				capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
				
			} else if (( a_it * a_iq_DelMejor ) == ( a_it_DelMejor * a_iq)) {
				if (abs_a_pq > abs_a_pq_DelMejor) {
					capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
				}
			}
		}
	}
	
	
	//MEJORA de codigo apturarElMejor( i ) es la misma instruccion en todos los casos del if elsif por lo que puede ser englobado en una sola condicion,  i:= kmax; no le veo objeto a esta definicion se puede usar kmax directamente y es mas claro el codigo
	if (checkearFilaOpt) {
		i = kmax;
		ix = -smp.left[kmax];
		if (ix > 0) {  // Es una fila "x"
			if ( abs(smp.flg_x[ix] ) == 1) {   // tiene manejo de cota superior
				// agregamos su fila fantasma como una más candidata a pivotear y a controlar
				// su factibilidad en caso de pivotear con otra.
				// En la fila kmax el aiq es positivo, pues fue elegido con locate_zpos
				//      aiq:= -e(kmax, q);
				a_iq = -smp.mat[kmax * (smp.NVariables + 1) + q];
				a_it = smp.x_sup[ix - 1] - smp.mat[kmax * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
				abs_a_pq = abs(a_iq);
				printf("%s\n", "aiq >= 0 en tsimplex.mejorpivote");
				assert(a_iq < 0);
				// static_assert(a_iq < 0, "aiq >= 0 en tsimplex.mejorpivote");
				//      b_:= x_sup.pv[ix] - e(kmax, nc);
				xFantasma_fila = true;
				if (p < 0) { // es el primer candidato
					capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
				} else if (( a_it * a_iq_DelMejor ) > ( a_it_DelMejor * a_iq)) {
					capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
				} else if (( a_it * a_iq_DelMejor ) == ( a_it_DelMejor * a_iq)) {
					if (abs_a_pq > abs_a_pq_DelMejor) {
						capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
					}
				}
			}
		}


		// ahora bien, independientemente de que se trate de una x o una y
		// a( kmax, q ) > 0 por selección del q y a( kmax, ti ) < 0 porque
		// por eso estamos tratando de optimizar chequeando kmax para volverla
		// factible.
		// Agregamos entonces la posibilidad de pivotear con kmax como forma de volverla factible.
		a_iq = smp.mat[kmax * (smp.NVariables + 1) + q];
		a_it = smp.mat[kmax * (smp.NVariables + 1) + smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		xFantasma_fila = false;
		abs_a_pq = abs(a_iq);
		if (p < 0) { // Es el primer candidato
			capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
		} else if (( a_it * a_iq_DelMejor ) < ( a_it_DelMejor * a_iq)) { // OJO, observar que es un "<"
			capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
		} else if (( a_it * a_iq_DelMejor ) == ( a_it_DelMejor * a_iq)) {
			if (abs_a_pq > abs_a_pq_DelMejor) {
				capturarElMejor(a_iq_DelMejor, a_it_DelMejor, p, filaFantasma, colFantasma, a_iq, a_it, i, xFantasma_fila);
			}
		}
	}

	return p;
}


bool cambio_var_cota_sup_en_columna(TSimplexGPUs &smp, int q) {

	int ix, kfil;
	double xsup;
	bool res;
  
	res = false;
	ix = -smp.top[q];
	// MAP: Agrego -1 para correr el indice a la indexacion desde 0 
	if ((ix > 0) && (abs(smp.flg_x[ix - 1]) == 1)) { // Corresponde a una x con cota sup
		// if abs(flg_x[ix] ) <> 1  then  writeln( 'mmmmm ' );

		// cambio de variable en la misma columna
		smp.flg_x[ix - 1] = -smp.flg_x[ix - 1]; // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
		xsup = smp.x_sup[ix - 1]; // MAP: Agrego -1 para correr el indice a la indexacion desde 0
		
		// MAP: Aca el codigo original esta dividido en 3 casos segun las restricciones redundantes y la variable de compilacion MODIFICAR_REDUNDANTES, como nosotros consideramos la variable siempre activa concateno todo en un caso con un unico for
		for (kfil = 0; kfil <= smp.NRestricciones; kfil++) { // MAP: Antes 1 to cnt_RestriccionesRedundantes
			smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables] = smp.mat[kfil * (smp.NVariables + 1) + smp.NVariables] + smp.mat[kfil * (smp.NVariables + 1) + q] * xsup; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
			smp.mat[kfil * (smp.NVariables + 1) + q] = -smp.mat[kfil * (smp.NVariables + 1) + q];
		}

		res = true;
	} else {
		// MAP: IMPLEMENTAR LOS SIGUIENTE QUE ESTA COMENTADO PARA EL MANEJO DE ERROR
		/*
		self.DumpSistemaToXLT_('simplex_quehagoaca.xlt', '' );
		writeln('??? qué hago acá ???');
		raise Exception.Create( 'QUé hago acá' );
		*/
	}
	
	return res;
}


int locate_qOK(TSimplexGPUs &smp, int p, int jhasta, int jti, int cnt_RestriccionesRedundantes) {
	int mejorq, q;
	double max_apq, apq;

	mejorq = -1;
	max_apq = -1;
	for (q = 0; q <= jhasta; q++) { // MAP: Antes 1 to jhasta, jhasta ya viene con el indice ajustado a la indexacion por 0
		if (test_qOK(smp, p, q, jti, apq, cnt_RestriccionesRedundantes) && ((mejorq < 0) or (apq > max_apq))) {
			mejorq = q;
			max_apq = apq;
		}
	}
	
	return mejorq;
}


/*
Esta función retorna true si la columna q soluciona la infactibilidad
de la fila p. Se supone que (jti) es la columna de los términos constantes
(generalmente la nc ) la dejamos como parámetro por si es necesario

El valor retornado apq, es e(p,q) y puede usarse para
elegir el q que devuelva el valor más grande para disminuir los
errores numéricos.
*/
bool test_qOK(TSimplexGPUs &smp, int p, int q, int jti, double &apq, int cnt_RestriccionesRedundantes) {
	int k, ix;
	double alfa_p, akq,
		nuevo_ti;

	// apq:= e(p, q);
	apq = smp.mat[p * (smp.NVariables + 1) + q];
	// if ( apq  <= AsumaCero) then
	if (abs(apq) <= AsumaCero) { // rch@202012081043 agrego el abs()
		return false;
	} else {
		// alfa_p:= -e( p, jti ) / apq;
		alfa_p = -smp.mat[p * (smp.NVariables + 1) + jti] / apq;
		ix = -smp.top[q];
		if (ix > 0) { // la col q es una x
			if (smp.flg_x[ix - 1] != 0) {  // tiene manejo de cotasup, MAP: Agrego -1 para correr el indice a la indexacion desde 0 
				if (alfa_p > smp.x_sup[ix - 1]) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
					// De intercambiar esta columna se violaría la cotas superior
					if (alfa_p > (smp.x_sup[ix - 1] + CasiCero_Simplex)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
						return false;
					}
					// else begin //writeln( 'OJO, TSimplex.test_qOk, de intercambiar esta  columna se viola por poquito la cota sup y la dejo pasar' );
				}
			}
		}
		
		for (k = cnt_RestriccionesRedundantes; k < p; k++) { // MAP: cnt_RestriccionesRedundantes + 1 to p - 1, p ya viene con el indice ajustado a la indexacion desde 0
			// akq = e(k, q);
			akq = smp.mat[k * (smp.NVariables + 1) + q];
			// nuevo_ti = e(k,jti) + akq * alfa_p;
			nuevo_ti = smp.mat[k * (smp.NVariables + 1) + jti] + akq * alfa_p;
			if (nuevo_ti < -CasiCero_Simplex) {
				return false;
			} else {
				ix = -smp.left[k];
				if ((ix > 0) and (smp.flg_x[ix - 1] != 0)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
					//  if nuevo_ti > x_sup.pv[ix]  then
					if (nuevo_ti > (smp.x_sup[ix - 1] + CasiCero_Simplex)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0 
						return false;
					}
				}
			}
		}

		return true;
	}
}


int darpaso(TSimplexGPUs &smp, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes) {

	int ppiv, qpiv,
		res;
	bool filaFantasma, colFantasma;

	// cnt_paso++;
	qpiv = locate_zpos(smp, smp.NRestricciones, cnt_columnasFijadas); // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
	if (qpiv >= 0) { // MAP: Antes > 0, pero con el corrimiento de indice queda >=
		ppiv = mejorpivote(smp, qpiv, smp.NRestricciones, filaFantasma, colFantasma, false, cnt_RestriccionesRedundantes); // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
		if (ppiv < 0) { // MAP: Antes < 1, pero con el corrimiento de indice queda < 0
			return -1;
		}
		
		if (!colFantasma) {
			if  (!intercambiar(smp, ppiv, qpiv)) {
				return -1;
			}
			
			if (filaFantasma) {
				if (!cambio_var_cota_sup_en_columna(smp, qpiv)) {
					return -1;
				}
			}
			res = 1;
		} else {
			printf("%s\n", "Si Es FantasmaDeCol tenía que ser ppiv = qpiv");
			assert(ppiv == qpiv);
			//static_assert(ppiv = qpiv , "Si Es FantasmaDeCol tenía que ser ppiv = qpiv");
			cambio_var_cota_sup_en_columna(smp, ppiv);
			res = 1;
		} 
	} else {
		res = 0; // ShowMessage('No encontre z - positivo ' );
	}
		
	return res;
}

