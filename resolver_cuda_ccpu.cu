#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>     /* abs */


#include "tsimplexgpus.h"


const double CasiCero_Simplex_Escalado = 1.0E-30;
const double CasiCero_Simplex = 1.0E-7;
const double CasiCero_Simplex_CotaSup = CasiCero_Simplex * 1.0E+3;
const double CasiCero_VarEntera = 1.0E-5;
const double CasiCero_CajaLaminar = 1.0E-30;
const double AsumaCero =  1.0E-16; // EPSILON de la maquina en cuentas con Double, CONFIRMAR SI ESTO ES CORRECTO (double 64 bits, 11 exponente y 53 mantisa, 53 log10(2) ≈ 15.955 => 2E−53 ≈ 1.11 × 10E−16 => EPSILON  ≈ 1.0E-16)
const double MaxNReal = 1.7E+308; // Aprox, CONFIRMAR SI ESTO ES CORRECTO


extern "C" void resolver_cuda(TDAOfSimplexGPUs &simplex_array, TDAOfSimplexGPUs &d_simplex_array, TDAOfSimplexGPUs &h_simplex_array, int NTrayectorias) {
	int NEnteras;
	int NVariables;
	int NRestricciones;
	int cnt_varfijas;
	int cnt_RestriccionesRedundantes;
	
	//TSimplexVars smp_var_arr = (SimplexVars*)malloc(NTrayectorias*sizeof(SimplexVars));
	
	for (int kTrayectoria = 0; kTrayectoria < NTrayectorias; kTrayectoria++) {
		resolver_cpu(&simplex_array[kTrayectoria]); //,  &smp_var_arr[kTrayectoria]); 
	}
	
}


void resolver_cpu(TDAOfSimplexGPUs &simplex) { //,  TSimplexVars &vars) {
	int res;
	int cnt_columnasFijadas = 0; // Cantidad de columnas FIJADAS x o y fija encolumnada
	int result = 0;
	int cnt_varfijas = 0; // Cantidad de variables fijadas
	string mensajeDeError;

	fijarCajasLaminares(simplex, cnt_varfijas);

	// Fijamos las variables que se hayan declarado como constantes.
	if (!fijarVariables(simplex, cnt_varfijas, cnt_columnasFijadas)) {
		mensajeDeError = 'PROBLEMA INFACTIBLE - No fijar las variable Fijas.';
		result = -32;
		return;
	}

	if (!enfilarVariablesLibres(simplex, cnt_columnasFijadas, cnt_RestriccionesRedundantes, cnt_VariablesLiberadas)) {
		mensajeDeError = 'No fue posible conmutar a filas todas las variables libres';
		result = -33;
		return;
	}

	if (resolverIgualdades(simplex, cnt_columnasFijadas, cnt_varfijas, cnt_RestriccionesRedundantes) != 1) {
		mensajeDeError = 'PROBLEMA INFACTIBLE - No logré resolver las restricciones de igualdad.';
		result = -31;
		return;
	}

	lbl_inicio:

	reordenarPorFactibilidad(simplex, cnt_RestriccionesRedundantes, cnt_RestrInfactibles); // MAP: cnt_RestrInfactibles es modificada dentro del proc

	res = 1;
	while (cnt_RestrInfactibles > 0) {
    res = pasoBuscarFactible(simplex);

    switch (res) {
		case 0: 
			if (cnt_RestrInfactibles > 0) {
				mensajeDeError := 'PROBLEMA INFACTIBLE - Buscando factibilidad';
				result = -10;
				return;
			}
			break;
		case  -1:
			mensajeDeError = 'NO encontramos pivote bueno - Buscando Factibilidad';
			result = -11;
			return;
		case -2:
			mensajeDeError = '???cnt_infactibles= 0 - Buscando Factibilidad';
			result = -12;
			return;
	}

	while (res == 1) {
		res = darpaso(simplex);
		if (res == -1) {
			mensajeDeError = 'Error -- NO encontramos pivote bueno dando paso';
			result = -21;
			return;
		}
	}
  
	if (res == 2) {
		goto lbl_inicio;
	}
	
	result = res;
}


// Si abs( cotasup - cotainf ) < AsumaCeroCaja then flg_x := 2
// retorna la cantidad de cajas fijadas.
int fijarCajasLaminares(TDAOfSimplexGPUs &smp, int &cnt_varfijas) {
	int res = 0;
	
	for (int i =  0; i < smp.NVariables; i++) { // MAP: = for 1 to nc-1 
		if (abs(smp.flg_x[i] ) < 2) { // No se aplica ni para 2 ni para 3
			if (abs(smp.x_sup[i] - smp.x_inf[i] ) < CasiCero_CajaLaminar) {
				if (smp.flg_x[i] < 0) {
					smp.flg_x[i] = -2;
				} else {
					smp.flg_x[i] = 2;
				}
				cnt_varfijas++;
				res++;
			}
		}
	}
    return res;
}


void posicionarPrimeraLibre(TDAOfSimplexGPUs &smp, int cnt_varfijas, int &cnt_fijadas, int &cnt_columnasFijadas, int &kPrimeraLibre) {
    while (cnt_fijadas < cnt_varfijas) &&
      (((smp.top[kPrimeraLibre] < 0) && (abs(smp.flg_x[-smp.top[kPrimeraLibre]]) == 2)) or
        ((smp.top[kPrimeraLibre] > 0) && (abs(smp.flg_y[smp.top[kPrimeraLibre]]) == 2))) {
		if (smp.top[kPrimeraLibre] < 0) {
			cnt_fijadas++;
		}	
		cnt_columnasFijadas++;
		kPrimeraLibre--;
	}
}


bool fijarVariables(TDAOfSimplexGPUs &smp, int cnt_varfijas, int &cnt_columnasFijadas) {

	int kColumnas, mejorColumnaParaCambiarFila, kFor, kFilaAFijar, cnt_fijadas, kPrimeraLibre;
	double mejorAkFilai;
	bool buscando, pivoteoConUnaFijada;

	if (cnt_varfijas > 0) {
		cnt_fijadas = 0;
		kPrimeraLibre = smp.NVariables - 1; // MAP: nc - 1, - 1 debido al cambio de indices
		kColumnas = 0; // MAP: Antes 1, pero cambimos que los vectores esten indexados desde 1 a 0
		
		while ((cnt_fijadas < cnt_varfijas) && (kColumnas <= kPrimeraLibre)) {
			posicionarPrimeraLibre(smp, cnt_varfijas, cnt_columnasFijadas);
			//Busco en columnas
			if ((cnt_fijadas < cnt_varfijas) && (kColumnas <= kPrimeraLibre)) {
				buscando = true;
				while (buscando && (kColumnas <= kPrimeraLibre)) {
					
					// MAP: Aca grego -1 a flg_x[-top[kColumnas] -1] para obtener el indice que empieza en 0 y top left son 2 vectores indexados en 0 pero que contienen numeros comenzados en 1, por lo tanto el indice real es top[indice] - 1,
					// esto lo voy a tener que hacer a lo largo y ancho del algoritmo
					if ((smp.top[kColumnas]) < 0) && (abs(smp.flg_x[-smp.top[kColumnas] -1]) == 2) ) { 
						//es una x fija
						buscando = false
					} else {
						kColumnas++;
					}
				}
				if (!buscando) {
					intercambioColumnas(smp, kColumnas, kPrimeraLibre);
					kPrimeraLibre--;
					cnt_fijadas++;
					cnt_columnasFijadas++;
					kColumnas++;
				}
			}
		}

		// Se inicializa en la fila anterior a la ultima fijada
		kFilaAFijar = cnt_RestriccionesRedundantes - 1; // MAP: Agreo -1
		while (cnt_fijadas < cnt_varfijas) {
		
			posicionarPrimeraLibre(smp, cnt_varfijas, cnt_columnasFijadas);
			if (cnt_fijadas < cnt_varfijas) {
		 
				//Busco en filas
				for (kFor = kFilaAFijar + 1; kFor < smp.NVariables - 1; kFor++) { // MAP: change index in loop
					if ((smp.left[kFor] < 0) && (abs(smp.flg_x[-smp.left[kFor]-1]) == 2)) {
						kFilaAFijar = kFor;
						break;
					}
				}

				mejorColumnaParaCambiarFila = 0; // MAP: antes 1
				mejorAkFilai = abs(smp.mat[kFilaAFijar][0]); // MAP: before [1], primera columna ahora es 0
				for (kColumnas = 1; kColumnas < kPrimeraLibre; kColumnas++) { // MAP: Antes 2 to kPrimeraLibre
					if (abs(smp.mat[kFilaAFijar][kColumnas]) > mejorAkFilai) {
						mejorColumnaParaCambiarFila = kColumnas;
						mejorAkFilai = abs(smp.mat[kFilaAFijar][kColumnas]);
					}
				}

				pivoteoConUnaFijada = false;
				// dv@20191226 Si el término independiente es nulo, la "restricción" es reduntante
				// entonces la variable ya había quedado fijada
				if (mejorAkFilai < AsumaCero) {
					return false;
				}

				intercambiar(smp, kFilaAFijar, mejorColumnaParaCambiarFila);
				cnt_fijadas++;
				
				if (!pivoteoConUnaFijada) {
					// dv@20200115 agrego esto porque no debería cambiar si pivoteó con una fijada
					if (mejorColumnaParaCambiarFila != kPrimeraLibre) {
						intercambioColumnas(smp, mejorColumnaParaCambiarFila, kPrimeraLibre);
					}
					cnt_columnasFijadas++;
					kPrimeraLibre--;
				} else {
					if (mejorColumnaParaCambiarFila != kPrimeraLibre + 1) { // En el caso de que se pivotee con una columna fija no cambia kPrimeraLibre
						intercambioColumnas(smp, mejorColumnaParaCambiarFila, kPrimeraLibre + 1);
					}
				}
			}
		}
	}
	
	return true;

}


bool intercambiar(TDAOfSimplexGPUs &smp, int kfil, int jcol) {

	double m, piv, invPiv;
	int k, j;

	piv := smp.mat[kfil][jcol];
	invPiv := 1 / piv;

	// MAP: POSIBLE MEJORA, en GPU se encargaran hilos diferentes, recorrida por filas
	// MAP: En el codigo original se separa en 4 casos el mismo codigo, se recorre desde 1 to cnt_RestriccionesRedundantes to kfil - 1  <skipping kfil (fila pivot)> to nf - 1 to nf
	// MAP: Se concatenan estos casos en un mismo for
	for (k = 0; k < kfil; k++) {
		m := -smp.mat[k][jcol] * invPiv;
		if (abs(m) > 0) {
			for (j = 0; j < jcol; j++) {
				smp.mat[k][j] = smp.mat[k][j] + m * smp.mat[kfil][j]; 
			}
			
			smp.mat[k][jcol] = -m;
			
			for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes "to nc"
				smp.mat[k][j] = smp.mat[k][j] + m * smp.mat[kfil][j];
			}
		} else {
			smp.mat[k][jcol] = 0; // IMPONGO_CEROS
		}
	}
	
	// Salteo la fila kfil
	
	for (k = kfil + 1; k <= smp.NRestricciones; k++) { // MAP: <= pq considera la fila de la funcion z
		m := -smp.mat[k][jcol] * invPiv;
		if (abs(m) > 0) {
			for (j = 0; j < jcol; j++) {
				smp.mat[k][j] = smp.mat[k][j] + m * smp.mat[kfil][j]; 
			}
			
			smp.mat[k][jcol] = -m;
			
			for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes "to nc"
				smp.mat[k][j] = smp.mat[k][j] + m * smp.mat[kfil][j];
			}
		} else {
			smp.mat[k][jcol] = 0; // IMPONGO_CEROS
		}
	}

	// Completo la fila kfil
	m := -invPiv;
	for (j = 0; j < jcol; j++) {
		smp.mat[kfil][j] = smp.mat[kfil][j] * m;
	}

	smp.mat[kfil][jcol] = -m;
	
	for (j = jcol + 1; j <= smp.NVariables; j++) { // MAP: Antes jcol + 1 to nc
		smp.mat[kfil][j] = smp.mat[kfil][j] * m;
	}

	k = smp.top[jcol];
	smp.top[jcol] = smp.left[kfil];
	smp.left[kfil] = k;

	actualizo_iitop(jcol);
	actualizo_iileft(kfil);

  return true;
 }

// MAP: Agrego los -1 +1 por el cambio de indices CHEQUEAR QUE ES CORRECTO
void actualizo_iitop(TDAOfSimplexGPUs &smp, int k) {
	// actualizo los indices iix e iiy
	if (smp.top[k] < 0) {
		smp.iix[-smp.top[k] - 1] = k + 1;
	} else {
		smp.iiy[smp.top[k] - 1] = -k - 1;
	}
}

// MAP: Agrego los -1 +1 por el cambio de indices CHEQUEAR QUE ES CORRECTO
void actualizo_iileft(TDAOfSimplexGPUs &smp, int k) {
	// actualizo los indices iix e iiy
	if (smp.left[k] > 0) {
		smp.iiy[smp.left[k] - 1]  = k + 1; 
	} else {
		smp.iix[-smp.left[k] - 1]  = -k - 1;
	}
}

void intercambioColumnas(TDAOfSimplexGPUs &smp, int j1, int j2) {

	int k;
	double m;

	for (k = 0; k <= smp.NRestricciones; k++) { // MAP: Intercambia toda la columna por lo tanto va hasta smp.NRestricciones inclusive
		m = smp.mat[k][j1];
		smp.mat[k].pv[j1]  = smp.mat[k][j2];
		smp.mat[k].pv[j2]  = m;
	}

	k = smp.top[j1];
	smp.top[j1] = smp.top[j2];
	smp.top[j2] = k;

	actualizo_iitop(smp, j1);
	actualizo_iitop(smp, j2);
}

void intercambioFilas(TDAOfSimplexGPUs &smp, int k1, int k2) {
  
	int j;
	double m;

	for (j = 0; j <= smp.NVariables; j++) { // MAP: Intercambia toda la fila por lo tanto va hasta smp.NVariables inclusive
		m = smp.mat[k1][j];
		smp.mat[k1].pv[j] = smp.mat[k2][j];
		smp.mat[k2].pv[j] = m;
	}

	j = smp.left[k1];
	smp.left[k1] = smp.left[k2];
	smp.left[k2] = j;

	actualizo_iileft(smp, k1);
	actualizo_iileft(smp, k2);

}

 // Atención esto funciona porque suponemos que el Simplex está en su estado Natural.
 // a lo sumo se conmutaron algunas columnas para fijar variables.
 // MAP: se usa solo dentro de enfilarVariablesLibres
 int buscarMejorPivoteEnCol(TDAOfSimplexGPUs &smp, int jCol, int iFilaFrom, int iFilaHasta) {
	
	double a;
	int iFil, res;
	
    res = -1;
    a = 0.0;
	for (iFil = iFilaFrom; iFil <= iFilaHasta; iFil++) { // MAP: Originalmente iFilaFrom to iFilaHasta, eso no cambia ya que se le pasan los indices ya adaptados a la indexacion desde 0
		if (abs(smp.mat[iFil][jCol]) > a) {
			a = abs(smp.mat[iFil][jCol]);
			res = iFil;
		}
	}
	return res;
 }


bool enfilarVariablesLibres(TDAOfSimplexGPUs &smp, int cnt_columnasFijadas, int &cnt_RestriccionesRedundantes, int &cnt_VariablesLiberadas) {

	int jVar, iFil, jCol;

	for (jCol = 0; jCol < smp.NVariables -  cnt_columnasFijadas; jCol++) { // MAP: Antes 1 to nc - 1 - cnt_columnasFijadas, nc - 1 = NVariables
		jVar = -smp.top[jCol];

		if ((jVar > 0) && (smp.flg_x[jVar - 1] == 3)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0
			iFil = buscarMejorPivoteEnCol(smp, jCol, cnt_RestriccionesRedundantes, smp.NRestricciones - 1); // MAP: Modifico indices paso el indice de filaFrom y filaHasta (inclusive)
			if (iFil < 1) {
				return false;
			}
			intercambiar(smp, iFil, jCol);
			cnt_RestriccionesRedundantes++;
			if (iFil > cnt_restriccionesRedundantes) {
				IntercambioFilas(smp, cnt_RestriccionesRedundantes, iFil);
			}
			cnt_VariablesLiberadas++;
		}
	}
  
	return true;
}


int resolverIgualdades(TDAOfSimplexGPUs &smp, int &cnt_columnasFijadas, int cnt_varfijas, int &cnt_RestriccionesRedundantes) {
	int res, cnt_acomodadas, ifila, icolumna, 
		nIgualdadesResueltas, nIgualdadesAResolver, 
		iFilaLibre, iFilaAcomodando;
	int * nCerosFilas;
	int * nCerosCols;
	bool fantasma;

	cnt_acomodadas = cnt_columnasFijadas - cnt_varfijas;

	// Muevo las igualdades que esten en columnas al lado derecho junto con las FIJADAS
	icolumna = smp.NVariables - cnt_columnasFijadas - 1; // MAP: Indice modificado, cambio nc por smp.NVariables 
	while ((icolumna >= 0) && (cnt_acomodadas < cnt_Igualdades)) {
		if ((smp.top[icolumna] > 0) && (abs(smp.flg_y[top[icolumna] - 1]) == 2)) { // MAP: Agrego -1 para correr el indice a la indexacion desde 0
			if (icolumna <> (nc - cnt_columnasFijadas - 1)) {
				IntercambioColumnas(icolumna, nc - cnt_columnasFijadas - 1);
			}
			cnt_acomodadas++;
			cnt_columnasFijadas++;
		}
		icolumna--;
	}

	// rch@20130307.bugfix - begin ----------------------------
	// ahora reviso las que ya estén declaradas como redundantes a ver si hay
	// igualdades e incremento el contador de acomodadas.
	for (iFilaAcomodando = 0; iFilaAcomodando < cnt_RestriccionesRedundantes; iFilaAcomodando++) { // MAP: originalmente 1 to cnt_RestriccionesRedundantes
		if ((smp.left[iFilaAcomodando] > 0) && (abs(smp.flg_y[iFilaAcomodando]) == 2)) {
			//  Es una restricción  y es de igualdad
			cnt_acomodadas++;
		}
	}

	// En el caso de estar resolviendo un MIPSimplex, en contador de redundantes puede venir
	// incrementado del problema PADRE y pueden haber restricciones de igualdad dentro de las
	// redundantes. Esto hacía que el  "while  cnt_acomodadas < cnt_Igualdades"
	// que está unas lineas abajo NO saliera por no alcanzar la condición.
	// rch@20130307.bugfix - end ----------------------------


	// ahora reordeno las igualdades y las que queden en filas las pongo al inicio
	iFilaLibre = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0
	iFilaAcomodando = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0

	while (cnt_acomodadas < cnt_Igualdades) {
		if ((smp.left[iFilaAcomodando] > 0) and (abs(smp.flg_y[smp.left[iFilaAcomodando] - 1]) == 2)) { // MAP: Agrego - 1 para mover el indice a la indexacion desde 0
			//vc, dv @20200116 decia flg_y[iFilaAcomodando], se fijaba en cualquier lado
			//  Es una restricción  y es de igualdad
			if (iFilaLibre <> iFilaAcomodando) {
				IntercambioFilas(iFilaAcomodando, iFilaLibre);
			}
			cnt_acomodadas++;
			iFilaLibre++;
		}
		iFilaAcomodando++;
	} //Al salir de aca iFilaLibre queda en la primer fila que no es de igualdad

	res = 1;

	nIgualdadesResueltas = 0;
	nIgualdadesAResolver = iFilaLibre - (cnt_RestriccionesRedundantes + 1);
	nCerosFilas = (int*)malloc((smp.NRestricciones + 1)*sizeof(int)); // MAP: antes setLength(nCerosFilas, nf);
	nCerosCols = (int*)malloc((smp.NVariables + 1)*sizeof(int));// MAP: antes setLength(nCerosCols, nc);
	while (nIgualdadesResueltas < nIgualdadesAResolver) {
		//    res:= pasoBuscarFactibleIgualdad( cnt_RestriccionesRedundantes + 1 + nIgualdadesResueltas );
		//    res:= pasoBuscarFactibleIgualdad2( cnt_RestriccionesRedundantes + 1 + nIgualdadesResueltas );
		//    res:= pasoBuscarFactibleIgualdad3( nIgualdadesAResolver - nIgualdadesResueltas);
		res = pasoBuscarFactibleIgualdad4(nIgualdadesAResolver - nIgualdadesResueltas, nCerosFilas, nCerosCols, cnt_columnasFijadas, cnt_RestriccionesRedundantes);

		if (res = 1) {
			nIgualdadesResueltas = nIgualdadesResueltas + 1;
			cnt_columnasFijadas++;
		} else {
			ifila = cnt_RestriccionesRedundantes; // MAP: remuevo el +1 debido a la nueva indexacion desde 0
			while ((nIgualdadesResueltas < nIgualdadesAResolver) && filaEsFactible(ifila, fantasma)) {
				if (iFila <> cnt_RestriccionesRedundantes + 1) {
					IntercambioFilas(ifila, cnt_RestriccionesRedundantes + 1);
				}
				cnt_RestriccionesRedundantes++;
				nIgualdadesResueltas++;
				ifila++;
			}
			if (nIgualdadesResueltas < nIgualdadesAResolver) {
				mensajeDeError = 'PROBLEMA INFACTIBLE - Resolviendo igualdades.';
				res = -13;
				break;
			} else {
				res = 1;
			}
		}
	}

	free(nCerosFilas); // MAP: Antes setLength(nCerosFilas, 0);
	free(nCerosCols); // MAP: Antes setLength(nCerosCols, 0);
	return res;
}


int pasoBuscarFactibleIgualdad4(TDAOfSimplexGPUs &smp, int IgualdadesNoResueltas, int * nCerosFilas, int * nCerosCols, int cnt_columnasFijadas, int cnt_RestriccionesRedundantes) {

	int iFila, iColumna, columnasLibres,
		filaPiv, colPiv;
	double  maxVal, m;

	// Tengo todas las igualdades en columnas al final y las igualdades en filas al principio
	columnasLibres = smp.NVariables - cnt_columnasFijadas; // MAP: remuevo -1 ya que smp.NVariables = nc - 1 
	for (iColumna = 0; iColumna < columnasLibres; iColumna++) { // MAP: Originalmente 1 to columnasLibres
		nCerosCols[iColumna] = 0;
	}

	// Busco el máximo valor absoluto y cuento la cantidad de ceros en filas y columnas en la caja desde
	// cnt_RestriccionesRedundantes + 1 hasta cnt_RestriccionesRedundantes + nIgualdadesNoResueltas
	// la caja de las igualdades sin resolver
	maxVal = -MaxNReal;
	filaPiv = -1;
	colPiv = -1;
	// MAP: remuevo el +1 debido a la nueva indexacion desde 0, originalmente cnt_RestriccionesRedundantes + 1 to cnt_RestriccionesRedundantes + nIgualdadesNoResueltas
	for (iFila = cnt_RestriccionesRedundantes; iFila < cnt_RestriccionesRedundantes + nIgualdadesNoResueltas; iFila++) { 
		for (iColumna = 0; iColumna <columnasLibres; iColumna++) { // MAP: muevo el indice una lugar hacia atras para considerar el cambio de indexacion desde 0
			m = abs(smp.mat[iFila][iColumna]);
			if (m < AsumaCero) {
				Inc(nCerosFilas[iFila]);
				Inc(nCerosCols[iColumna]);
			} else if (m > maxVal) {
				maxVal = m;
				filaPiv = iFila;
				colPiv = iColumna;
			}
		}
	}

	// Termino de contar la cantidad de ceros en columnas con el resto de las filas
	for (iFila := cnt_RestriccionesRedundantes + nIgualdadesNoResueltas; iFila < smp.NRestricciones; iFila++) { // MAP: cnt_RestriccionesRedundantes + nIgualdadesNoResueltas + 1 to nf - 1
		for (iColumna = 0; iColumna < columnasLibres - 1; iColumna++) { // MAP: Originalmente 1 to columnasLibres - 1
			if (abs(smp.mat[iFila][iColumna]) < AsumaCero) {
				nCerosCols[iColumna]++;
			}
		}
	}

	if (maxVal > CasiCero_Simplex) {
		for iFila := cnt_RestriccionesRedundantes + 1 to cnt_RestriccionesRedundantes +	nIgualdadesNoResueltas) { // MAP: Originalmente cnt_RestriccionesRedundantes + 1 to cnt_RestriccionesRedundantes +	nIgualdadesNoResueltas
			for (iColumna = 0; iColumna < columnasLibres; iColumna++) { // MAP: Originalmente 1 to columnasLibres
				if (abs(smp.mat[iFila][iColumna]) * 10 >= maxVal) {
					// Lo considero como posible pivote
					if (nCerosFilas[filaPiv] + nCerosCols[colPiv]) < (nCerosFilas[iFila] + nCerosCols[iColumna]) {
						filaPiv = iFila;
						colPiv = iColumna;
					}
				}
			}
		}

		// Muevo la fila a intercambiar al final asi me siguen quedando las que voy a acomodar en bloque desde cnt_RestriccionesRedundantes
		if (filaPiv <> cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1) { // MAP: Agrego -1 para correr el indice, filaPiv ya esta en el indice correcto
			IntercambioFilas(filaPiv, cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1); // MAP: Agrego -1 para correr el indice, filaPiv ya esta en el indice correcto
		}
		intercambiar(cnt_RestriccionesRedundantes + nIgualdadesNoResueltas - 1, colPiv); // MAP: Agrego -1 para correr el indice, colPiv ya esta en el indice correcto
		if (colPiv <> columnasLibres) {
			IntercambioColumnas(colPiv, columnasLibres - 1); // MAP: Agrego -1 para correr el indice, colPiv ya esta en el indice correcto
		}
		return 1;
	} else {
		return -1;
	}	
}

int reordenarPorFactibilidad(TDAOfSimplexGPUs &smp, int cnt_RestriccionesRedundantes, int &cnt_RestrInfactibles) {

	int kfil, ix;
	double rval;

	/*
		Primero recorremos las restricciones y
		si la restricción no está violada me fijo si corresponde a una variable
		con restricción de cota superior y si es así verificamos que tampoco esté
		violada la restricción fantasma, si la fantasma se viola hacemos el cambio
		de variable para volverla explícita
	*/
  for (kfil = cnt_RestriccionesRedundantes; kfil < smp.NRestricciones; kfil++) { // MAP: originalmente cnt_RestriccionesRedundantes + 1 to nf - 1
    // rval := e(kfil, nc);
    rval = smp.mat[kfil][nc];
    if (rval > 0) {
      // Si es = 0 no chequeo pues la fantasma no puede estar violada
		if (smp.left[kfil] < 0) {
			ix := -smp.left[kfil] - 1; // MAP: Agrego -1 para ajustar el indice a la indexacion desde 0
			if (smp.flg_x[ix] <> 0) and (x_sup[ix] < rval) then
				// Parece que violo la cota superior
				if ((smp.x_sup[ix] + CasiCero_Simplex_CotaSup) < rval) {
					// La viola realmente
					cambiar_borde_de_caja(smp, kfil);
				} else {
					// La viola por errores númericos
					// pon_e(kfil, nc, x_sup.pv[ix])
					smp.mat[kfil][smp.NVariables] = smp.x_sup[ix]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
				}
			}
		}
    } else {
		if (rval > -CasiCero_Simplex_CotaSup) {
		  // pon_e(kfil, nc, 0);
		  smp.mat[kfil][smp.NVariables] = 0; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		}
	}

	// Ahora sabemos que las violadas están explícitas, movemos todas las
	//restricciones violadas al final
	kfil = cnt_RestriccionesRedundantes; // MAP: Remuevo + 1 para ajustar el indice a la indexacion desde 0
	cnt_RestrInfactibles = 0;
	while (kfil < (smp.NRestricciones - cnt_RestrInfactibles)) { // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
		// rval:= e(kfil, nc);
		rval = smp.mat[kfil][smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		if (rval < 0) {
			Icnt_RestrInfactibles++;
			// while (e(nf-cnt_RestrInfactibles, nc ) < 0)
			// MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice, idem para nc	
			while ((smp.mat[smp.NRestricciones - cnt_RestrInfactibles][smp.NVariables] < 0) && (kfil < (smp.NRestricciones - cnt_RestrInfactibles))) {
				cnt_RestrInfactibles++;
			}
			if (kfil < (smp.NRestricciones - cnt_RestrInfactibles)) { // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
				IntercambioFilas(kfil, smp.NRestricciones - cnt_RestrInfactibles); // MAP: Cambio nf por smp.NRestricciones, dado que nf = smp.NRestricciones - 1 entonce ajusta el indice
			}
		}
		kfil++;
	}
	return cnt_RestrInfactibles;
}

void cambiar_borde_de_caja(TDAOfSimplexGPUs &smp, int k_fila) {
	int ix, k;
	/*
		Realizamos el cambio de variable x'= x_sup - x para que la restricción
		violada sea representada por x' >= 0
		Observar que para la nueva variable la restricción x >= 0 se transforma
		en x' <= x_sup. Es decir que la cota superior de x' es también x_sup.
	*/
	
	// MAP: Confio en el comentario debajo que eso da positivo por lo que resto 1 para ajustar el indice a la indexacion desde 0, de hecho si diera negativo como se trata de un indice fallaria
	ix = -smp.left[k_fila] - 1; // Se supone que esto da positivo, sino no es una x
	for (k = 0; k < smp.NVariables; k++) { // MAP: Muevo los indices 1 to nc-1 un lado a la izquierda para para ajustar el indice a la indexacion desde 0
		smp.mat[k_fila][k] = -smp.mat[k_fila][k];
	}
	
	smp.mat[k_fila][nc] = smp.x_sup[ix] - smp.mat[k_fila][nc];

	if (abs(smp.flg_x[ix]) <> 1) {
		writeln('mmmm ... porqué?');
	}
	smp.flg_x[ix] := -smp.flg_x[ix];
  
}


int pasoBuscarFactible(TDAOfSimplexGPUs &smp, int &cnt_RestrInfactibles) {
	
	int pFilaOpt, ppiv, qpiv, ix, res;
	double rval;
	bool filaFantasma, colFantasma;

	pFilaOpt = smp.NRestricciones - cnt_RestrInfactibles; // MAP: Antes nf - cnt_RestrInfactibles, indice ajustado
	// rval:= e(pFilaOpt, nc);
	rval = smp.mat[pFilaOpt][smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice

	/* OJO LE AGREGO ESTE CHEQUEO PARA PROBAR */
	// Si parece satisfecha verifico que no se esté violándo la  fantasma
	if (rval > 0) {
		if (smp.left[pFilaOpt] < 0) {
			ix = -smp.left[pFilaOpt] - 1; // MAP: Ajusto indice con -1 
			if (smp.flg_x[ix] != 0) and (rval > smp.x_sup[ix]) {
				if (rval > smp.x_sup[ix] + CasiCero_Simplex) {
					cambiar_borde_de_caja(pFilaOpt);
					// rval:= e(pFilaOpt, nc );
					rval = smp.mat[pFilaOpt][smp.NVariables]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
				} else {
					// pon_e(pFilaOpt, nc, x_sup.pv[ix]);
					smp.mat[pFilaOpt][smp.NVariables] = smp.x_sup[ix]; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
					rval = smp.x_sup[ix];
				}
			}
		}
	} else if (rval > -CasiCero_Simplex) {
		// pon_e(pFilaOpt, nc, 0);
		smp.mat[pFilaOpt][smp.NVariables] = 0; // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
		rval = 0;
	}

	if (rval >= 0) {
		// ya es factible, probablemente se arregló con algún cambio anterior.
		cnt_RestrInfactibles--;
		res = 1;
	} else {
		// Nos planteamos el problema de optimización con objetivo el valor de la restricción violada
		if (cnt_RestrInfactibles > 0) {
			qpiv = locate_zpos(smp, pFilaOpt);
			if (qpiv > 0) {
				ppiv = mejorpivote(smp, qpiv, pFilaOpt, filaFantasma, colFantasma, True);
				if (ppiv < 1) {
					res = -1; // ShowMessage('No encontre pivote bueno ');
				} else {
					if (!colFantasma) {
						intercambiar(ppiv, qpiv);
						if (filaFantasma) {
							cambio_var_cota_sup_en_columna(smp, qpiv);
						}
						// if ( e( pFilaOpt, nc) >= 0 ) then
						if (smp.mat[pFilaOpt][smp.NVariables] >= 0) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
							cnt_RestrInfactibles--;
						}
						res = 1;
					} else {
						cambio_var_cota_sup_en_columna(smp, ppiv);
						// if ( e( pFilaOpt, nc) >= 0 ) then
						if (smp.mat[pFilaOpt][smp.NVariables] >= 0) { // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
							cnt_RestrInfactibles--;
						}
						res = 1;
					}
				}
			} else {
				res = 0; // ShowMessage('No encontre z - positivo ' );
			}
		} else {
			res = -2;
		}

		if (res = -1) {
			// Pruebo si soluciono la infactibildad con un intercambio de la infactible con una de las Activas
			qpiv = locate_qOK(pFilaOpt, smp.NVariables - cnt_columnasFijadas - 1, smp.NVariables); // MAP: Cambio nc por smp.NVariables, dado que nc = smp.NVariables - 1 entonce ajusta el indice
			if (qpiv > 0) {
				intercambiar(smp, pFilaOpt, qpiv);
				cnt_RestrInfactibles--;
				res = 1;
			}
		}
	}

	return res;
}

// Revisa si puede decrementar la cantidad de restricciones infactibles y lo hace en caso de poder hacerlo
// MAP: Eso de revisa se ve que me lo debe, declaro en el nombre del padre de hijo y del espiritu santo este procedimeinto como procedimiento al p... y sustituyo las llamadas por cnt_RestrInfactibles--
void decCnt_RestrInfactibles(int &cnt_RestrInfactibles) {
	cnt_RestrInfactibles--;
}

// Buscamos la columna que en la ultima fila (fila z) tenga el valor positivo mas grande retorna el número de columna si lo encontro, -1 si son todos < 0
// Este paso se da en el Simplex para minimizar, en el de maximizar busca el menos negativo
int locate_zpos(int kfila_z) {
	int j, ires;
	double maxval;
	ires = -1;
	maxval = CasiCero_Simplex;
	for (j = 0; j <= smp.NVariables - cnt_columnasFijadas; j++) { // MAP: Antes 1 to nc - 1 - cnt_columnasFijadas
		if (smp.mat[kfila_z][j] > maxval) {
			maxval = smp.ma[tkfila_z][j];
			ires = j;
		}
	}
	return ires;
}


// NEXT mejorpivote
















procedure TSimplex.limpiar;
var
  k: integer;
begin
  cnt_paso := 0;
  mensajeDeError := '???';

  for k := 1 to nc - 1 do
  begin
    top[k] := -k;
    iix[k] := k;
    flg_x[k] := 0;
  end;

  for k := 1 to nf - 1 do
  begin
    left[k] := k;
    iiy[k] := k;
    flg_y[k] := 0;
  end;

  x_inf.Ceros;
  x_sup.Ceros; // creo que no es necesario

  cnt_RestrInfactibles := 0;
  cnt_igualdades := 0;
  cnt_varfijas := 0;
  cnt_columnasFijadas := 0;
  cnt_VariablesLiberadas := 0;
  ;
  cnt_RestriccionesRedundantes := 0;

  self.Ceros;

end;

function TSimplex.xval(ix: integer): NReal;
var
  k: integer;
  res: NReal;
begin
  k := iix[ix];
  if k > 0 then
  begin
    if flg_x[ix] >= 0 then
      res := 0
    else
      res := x_sup.pv[ix];
  end
  else
  begin
    if flg_x[ix] >= 0 then
      //      res:= e( -k, nc )
      res := pm[-k].pv[nc]
    else
      //      res:= x_sup.pv[ix] - e( -k, nc );
      res := x_sup.pv[ix] - pm[-k].pv[nc];
  end;

  if x_inf.pv[ix] <> 0 then
    res := res + x_inf.pv[ix];
  Result := res;
end;

function TSimplex.yval(iy: integer): NReal;
var
  k: integer;
  res: NReal;
begin
  k := iiy[iy];
  if k < cnt_RestriccionesRedundantes then
    res := 0
  else
   res := pm[k].pv[nc];

  Result := res;
end;

function TSimplex.xmult_caja(ix: integer; var aflg_x: shortint): NReal;
var
  k: integer;
  res: NReal;
begin
  k := iix[ix];
  if k > 0 then
  begin
    aflg_x := flg_x[ix];
    res := pm[nf].pv[k];
  end
  else
  begin
    aflg_x := -33;
    res := 0;
  end;

  Result:= res;

end;

function TSimplex.xmult(ix: integer): NReal;
var
  k: integer;
  res: NReal;
begin
  k := iix[ix];
  if k > 0 then
    if flg_x[ix] >= 0 then
      res := pm[nf].pv[k]
    else
      res := -pm[nf].pv[k]
  //   rch@201408250739 le agrego el signo de menos. Me parece que faltaba.
  else
    res := 0;

    Result:= res;

end;


function TSimplex.ymult(iy: integer): NReal;
var
  k: integer;
  res: NReal;
begin
  k := -iiy[iy];
  if k > 0 then
  begin
    if flg_y[iy] >= 0 then
      res := pm[nf].pv[k]
    else
      res := -pm[nf].pv[k];
  end
  else
    res := 0;
  Result := res;
end;

function TSimplex.fval: NReal;
begin
  Result := pm[nf].pv[nc];
end;




procedure TSimplex.FijarRestriccionIgualdad(kfila: integer);
begin
  Inc(cnt_igualdades);
  flg_y[kfila] := 2;
end;




constructor TSimplex.Create_init(m, n: integer;
  xfGetNombreVar, xfGetNombreRes: TFuncNombre);
begin
  inherited Create_init(m, n);
  cnt_resolver := 0;
  cnt_RestriccionesRedundantes := 0;
  cnt_VariablesLiberadas := 0;

  x_inf := TVectR.Create_Init(n - 1);
  x_sup := TVectR.Create_Init(n - 1);

  setlength(flg_x, n);
  setlength(flg_y, m);
  setlength(top, n + 1);
  setlength(left, m + 1);
  setlength(iix, n + 1);
  setlength(iiy, m + 1);

  if Assigned(xfGetNombreVar) then
    self.fGetNombreVar := xfGetNombreVar
  else
    self.fGetNombreVar := defaultGetNombreVar;
  if Assigned(xfGetNombreRes) then
    self.fGetNombreRes := xfGetNombreRes
  else
    self.fGetNombreRes := defaultGetNombreRes;

  limpiar;
end;

constructor TSimplex.Create_clone(spx: TSimplex);
begin
  inherited Create_Clone(spx);
  mensajeDeError := '';
  cnt_resolver := 0;
  cnt_paso := 0;

  top := copy(spx.top);
  left := copy(spx.left);
  iix := copy(spx.iix);
  iiy := copy(spx.iiy);

  cnt_RestriccionesRedundantes := spx.cnt_RestriccionesRedundantes;
  cnt_VariablesLiberadas := spx.cnt_VariablesLiberadas;
  cnt_RestrInfactibles := spx.cnt_RestrInfactibles;

  cnt_igualdades := spx.cnt_igualdades;
  cnt_varfijas := spx.cnt_varfijas;

  x_inf := TVectR.Create_clone(spx.x_inf);
  x_sup := TVectR.Create_clone(spx.x_sup);

  flg_x := copy(spx.flg_x);
  flg_y := copy(spx.flg_y);

  self.fGetNombreVar := spx.fGetNombreVar;
  self.fGetNombreRes := spx.fGetNombreRes;

end;


procedure TSimplex.cota_inf_set(ivar: integer; vxinf: NReal);
var
  kfila, k: integer;
  old_cotainf: NReal;
begin
  old_cotainf := x_inf.pv[ivar];
  x_inf.pv[ivar] := vxinf;

  if (old_cotainf <> 0) then
    vxinf := vxinf - old_cotainf;

  // me fijo si ya fue fijada una cota superior para esta variable
  // la cambio para reflejar la nueva cota para la nueva variable
  if (flg_x[ivar] <> 0) then
    x_sup.pv[ivar] := x_sup.pv[ivar] - vxinf;

  if flg_x[ivar] >= 0 then //Es la variable directa
  begin
    k := iix[ivar];
    if k > 0 then //Estoy arriba y es la variable directa
      // hacemos el cambio de variables
      for kfila := 1 to nf do
          //        acum_e(kfila, nc, e(kfila, k) * vxinf)
          pm[kfila].pv[nc] := pm[kfila].pv[nc] + pm[kfila].pv[k] * vxinf
    else//Estoy abajo y es la variable directa
      //      acum_e(-k, nc, -vxinf);
      pm[-k].pv[nc] := pm[-k].pv[nc] - vxinf;
  end;
end;


procedure TSimplex.cota_sup_set(ivar: integer; vxsup: NReal);
var
  k, kfila: integer;
  deltaCotaSup: NReal;
  a: NReal;
begin
  vxsup := vxsup - x_inf.pv[ivar];
  if flg_x[ivar] = 0 then
  begin
    flg_x[ivar] := 1;
    x_sup.pv[ivar] := vxsup;
  end
  else
  begin // ya tiene fijada cota sup la cambio
    deltaCotaSup := vxsup - x_sup.pv[ivar];
    x_sup.pv[ivar] := vxsup;
    if flg_x[ivar] < 0 then //Es la variable complementaria
    begin
      k := iix[ivar];
      if k > 0 then //Estoy arriba y es la variable complementaria
        // hacemos el cambio de variables
        for kfila := 1 to nf do
            //          acum_e( kfila, nc, -e(kfila, k) * deltaCotaSup)
            pm[kfila].pv[nc] := pm[kfila].pv[nc] - pm[kfila].pv[k] * deltaCotaSup
      else//Estoy abajo y es la variable complementaria
        //        acum_e( -k, nc, deltaCotaSup);
        pm[-k].pv[nc] := pm[-k].pv[nc] + deltaCotaSup;
    end;
  end;
end;

procedure TSimplex.FijarVariable(ivar: integer; valor: NReal);
begin
  if abs(flg_x[ivar]) <> 2 then
  begin
    cota_inf_set(ivar, valor);
    cota_sup_set(ivar, valor);
    if flg_x[ivar] >= 0 then
      flg_x[ivar] := 2
    else
      flg_x[ivar] := -2;
    Inc(cnt_varfijas);
  end
  else
    cota_inf_set(ivar, valor);
end;

destructor TSimplex.Destroy;
begin
  setlength(top, 0);
  setlength(left, 0);
  setlength(iix, 0);
  setlength(iiy, 0);

  x_inf.Free;
  x_sup.Free;
  setlength(flg_x, 0);
  setlength(flg_y, 0);
  
  inherited Destroy;
end;


function TSimplex.mejorpivote(q, kmax: integer; out filaFantasma, colFantasma: boolean;  checkearFilaOpt: boolean): integer;
var
  i, p: integer;
  a_iq, a_it, abs_a_pq: NReal;
  a_iq_DelMejor, a_it_DelMejor, abs_a_pq_DelMejor: NReal;
  ix: integer;
  xfantasma_Fila: boolean;
  esCandidato: boolean;


procedure CapturarElMejor( i: integer );
begin
  a_iq_DelMejor := a_iq;
  a_it_DelMejor := a_it;
  p := i;
  filaFantasma := xfantasma_Fila;
  colFantasma := False; // Solo por si el primero era la Fantasma de la Columna
end;

begin
  // inicializaciones no necesarias, solo para evitar el warning
  a_it := 0;
  xfantasma_Fila := False;
  a_it_DelMejor := 0;
  a_iq_DelMejor := 1;

/*11/9/2006 le voy a agregar para que si la q corresponde a una x con manejo
de cota superior considere la existencia de una fila adicional correspondiente
a la cota superior.
Dicha fila tiene un -1 en la coluna q y el valor x_sup como término independiente

rch.30/3/2007 Agrego el manejo del CasiCero_Simplex

PA.21/06/2007 Le agrego que al buscar el mejorpivote para optimizar la fila
kmax chequee si esta es una variable con restriccion de cota superior y que el
pivote elegido no la viole*/

  ix := -top[q];
  if (ix > 0) and ( abs(flg_x[ix] ) = 2 ) then
    writeln( 'OPATROPA!');

  if (ix > 0) and ( abs(flg_x[ix] ) = 1 ) then
  begin  // en la columna q hay una x con manejo de cota superior
    colFantasma := True;
    p := q;
    //l o fijamos en -1 porque todas las restricciones fantasma tienen un -1 en
    // en el coeficiente de la variable y x_sup como termino independiente
    a_iq_DelMejor := -1;
    a_it_DelMejor := x_sup.pv[ix];
    abs_a_pq_DelMejor := 1;
  end
  else
  begin
    p := -1;
    colFantasma := False;
  end;

  filaFantasma := False;

  for i := cnt_RestriccionesRedundantes + 1 to kmax - 1 do
  begin
    // b(i) >= 0 para todo i / cnt_RestriccionesRedundantes < i < kmax-1
    // Buscamos la fila i que tenga el maximo b(i)/a(i,q) con a(i,q) < 0
    // aiq:= e( i, q );
    a_iq := pm[i].pv[q];
    if a_iq >  CasiCero_Simplex then // si es positivo, verificamos si se trata de una x y entonces agregamos la fantasma.
    begin
      ix := -left[i];
      if (ix > 0) and (abs(flg_x[ix]) = 1) then
        // la variable en la fila i tiene cota superior, hay que probar con el cambio de variable
      begin
        a_iq := -a_iq;
        a_it := x_sup.pv[ix] - pm[i].pv[nc];
        xfantasma_Fila := True;
        esCandidato := True;
      end
      else
        esCandidato := False;
    end
    else if a_iq < -CasiCero_Simplex then // Si es negativo es candidato
    begin
      a_it := pm[i].pv[nc];
      esCandidato := True;
      xfantasma_Fila := False;
    end
    else
      esCandidato := False;


    if esCandidato then //considero el coeficiente para elegir el pivote
    begin
      abs_a_pq := abs( e( i, q ) );
	/*
	se supone que a_iq < 0 y a_iq_DelMejor < 0 pues sino no son candidatos.
	El término independiente de cualquier fila k, se transformará al usar a_iq como pivote
	como:
		a_k,nc = a_k,nc - a_kq / a_iq * a_i,nc
		y tiene que mantenerce >= 0 para cualquier k <> i.
		a_k,nc - a_kq / a_iq * a_i,nc >= 0  ec.(1)
		dividiendo por a_kq < 0 se tiene
		a_k,nc / a_kq <= a_i,nc / a_iq  ec.(2)
		observar que cada lado de la desigualdad depende solo de los coeficientes
		de la fila k (izquieda) o de la fila i (derecha), Esto nos permite
		ir recorriendo las filas con a_iq < 0 y quedarnos con el de mayor cociente
		a_i,nc/a_iq.

		Para no hacer las divisiones, en lugar de chequear la ec.(2), chequeamos
		la ec.(3) obtenida de la ec.(1) multiplicando por a_iq < 0
		a_k,nc * a_iq <= a_kq  * a_i,nc >= 0  ec.(3)

		Si se cumple la ec.3 a_iq es mejor pivote que a_kp.
	*/
      //aiq < 0 por como lo tomamos para esCandidato
      //bi >= 0 para todo i / cnt_RestriccionesRedundantes < i < kmax-1
      //El pivote es aquel que tenga mayor bi/aiq siempre que bi/aiq < 0 y aiq < 0
      //Ademas bi/aiq y b_max/a_max tienen el mismo signo =>
      //bi/aiq > b_max/a_max <=> bi * a_max > b_max * aiq
      if (p < 0) then // es el primer candidato
        CapturarElMejor( i )
      else if ( a_it_DelMejor * a_iq) < ( a_it * a_iq_DelMejor )  then
        CapturarElMEjor( i )
      else if ( a_it * a_iq_DelMejor ) = ( a_it_DelMejor * a_iq) then
        if abs_a_pq > abs_a_pq_DelMejor then
          CapturarElMejor( i );
    end;
  end;
	
	
	//MEJORA de codigo apturarElMejor( i ) es la misma instruccion en todos los casos del if elsif por lo que puede ser englobado en una sola condicion,  i:= kmax; no le veo objeto a esta definicion se puede usar kmax directamente y es mas claro el codigo
  if checkearFilaOpt then
  begin
    i:= kmax;
    ix := -left[kmax];
    if (ix > 0) then  // es una fila "x"
    begin
      if ( abs(flg_x[ix] ) = 1) then   // tiene manejo de cota superior
      begin
   // agregamos su fila fantasma como una más candidata a pivotear y a controlar
  // su factibilidad en caso de pivotear con otra.
     // En la fila kmax el aiq es positivo, pues fue elegido con locate_zpos
        //      aiq:= -e(kmax, q);
        a_iq := -pm[kmax].pv[q];
        a_it := x_sup.pv[ix] - pm[kmax].pv[nc];
        abs_a_pq:= abs( a_iq );
        assert(a_iq < 0, 'aiq >= 0 en tsimplex.mejorpivote');
        //      b_:= x_sup.pv[ix] - e(kmax, nc);
        xfantasma_Fila := True;
        if (p < 0) then // es el primer candidato
          CapturarElMejor( i )
        else if ( a_it * a_iq_DelMejor ) > ( a_it_DelMejor * a_iq) then
          CapturarElMEjor( i )
        else if ( a_it * a_iq_DelMejor ) = ( a_it_DelMejor * a_iq) then
          if abs_a_pq > abs_a_pq_DelMejor then
            CapturarElMejor( i );
      end;
    end;


    // ahora bien, independientemente de que se trate de una x o una y
    // a( kmax, q ) > 0 por selección del q y a( kmax, ti ) < 0 porque
    // por eso estamos tratando de optimizar chequeando kmax para volverla
    // factible.
    // Agregamos entonces la posibilidad de pivotear con kmax como forma de volverla factible.
    a_iq:= pm[kmax].pv[q];
    a_it:= pm[kmax].pv[nc];
    xFantasma_fila:= False;
    abs_a_pq:= abs( a_iq );
    if (p < 0) then // es el primer candidato
      CapturarElMejor( i )
    else if ( a_it * a_iq_DelMejor ) < ( a_it_DelMejor * a_iq) then // OJO, observar que es un "<"
      CapturarElMEjor( i )
    else if ( a_it * a_iq_DelMejor ) = ( a_it_DelMejor * a_iq) then
      if abs_a_pq > abs_a_pq_DelMejor then
        CapturarElMejor( i );

  end;

  Result := p;
end;



function TSimplex.cambio_var_cota_sup_en_columna(q: integer): boolean;
var
  ix: integer;
  res: boolean;
  kfil: integer;
  xsup: NReal;
begin
  res := False;
  ix := -top[q];
  if (ix > 0) and (abs(flg_x[ix]) = 1) then // corresponde a una x con cota sup
  begin
    // if abs(flg_x[ix] ) <> 1  then  writeln( 'mmmmm ' );

    // cambio de variable en la misma columna
    flg_x[ix] := -flg_x[ix];
    xsup := x_sup.pv[ix];
    for kfil := 1 to cnt_RestriccionesRedundantes do
    begin
      pm[kfil].pv[nc] := pm[kfil].pv[nc] + pm[kfil].pv[q] * xsup;
      pm[kfil].pv[q] := -pm[kfil].pv[q];
    end;

    res := True;
    for kfil := cnt_RestriccionesRedundantes + 1 to nf-1 do
    begin
      pm[kfil].pv[nc] := pm[kfil].pv[nc] + pm[kfil].pv[q] * xsup;
      pm[kfil].pv[q] := -pm[kfil].pv[q];
    end;

    for kfil := nf to nf do
    begin
      pm[kfil].pv[nc] := pm[kfil].pv[nc] + pm[kfil].pv[q] * xsup;
      pm[kfil].pv[q] := -pm[kfil].pv[q];
    end;
  end
  else
  begin
    self.DumpSistemaToXLT_('simplex_quehagoaca.xlt', '' );
    writeln('??? qué hago acá ???');
    raise Exception.Create( 'QUé hago acá' );
  end;
  Result := res;
end;



function TSimplex.locate_qOK(p, jhasta, jti: integer): integer;
var
  mejorq, q: integer;
  max_apq, apq: NReal;
begin
  mejorq := -1;
  max_apq := -1;
  for q := 1 to jhasta do
    if test_qOK(p, q, jti, apq) and ((mejorq < 0) or (apq > max_apq)) then
    begin
      mejorq := q;
      max_apq := apq;
    end;
  Result := mejorq;
end;



function TSimplex.darpaso: integer;
var
  ppiv, qpiv: integer;
  res: integer;
  filaFantasma, colFantasma: boolean;
begin
  Inc(cnt_paso);
  qpiv := locate_zpos(nf);
  if qpiv > 0 then
  begin
    ppiv := mejorpivote(qpiv, nf, filaFantasma, colFantasma, False);
    if ppiv < 1 then
    begin
      Result := -1; //ShowMessage('No encontre pivote bueno ');
      exit;
    end;

    if not colFantasma then
    begin
      if not intercambiar(ppiv, qpiv) then
      begin
        result:= -1;
        exit;
      end;
      if filaFantasma then
        if not cambio_var_cota_sup_en_columna( qpiv ) then
        begin
          result:= -1;
          exit;
        end;
      res := 1;
    end
    else
    begin
      assert( ppiv = qpiv , 'Si Es FantasmaDeCol tenía que ser ppiv = qpiv ' );
      cambio_var_cota_sup_en_columna( ppiv );
      res := 1;
    end;
  end
  else
    res := 0; //  ShowMessage('No encontre z - positivo ' );

  Result := res;
end;


procedure TSimplex.declararRestriccionRedundante(kres: integer);
var
  i, ires: integer;
  maxAbs: NReal;
  filaPiv: integer;
begin
  ires := iiy[kres];

  if ires < 0 then
  begin
    // La restricción está en una columna. Antes de declararla redundante
    // debemos conmutarla con una fila.
    // busca una fila con quien conmutar, las intercambia y sigue
    ires := -ires;
    maxAbs := 0;
    filaPiv := -1;
    for i := cnt_RestriccionesRedundantes + 1 to nf - 1 do
      if abs(pm[i].pv[ires]) > maxAbs then
      begin
        maxAbs := abs(pm[i].pv[ires]);
        filaPiv := i;
      end;

    Assert(filaPiv <> -1, 'TSimplex.declararRestriccionRedundante: la restricción esta en una columna y no hay una fila con que pivotear');
    intercambiar(filaPiv, ires);
    ires := filaPiv;
  end;

  if (ires > cnt_RestriccionesRedundantes + 1) then IntercambioFilas(ires, cnt_RestriccionesRedundantes + 1);

  Inc(cnt_RestriccionesRedundantes);
end;

procedure TSimplex.rearmarIndicesiiXiiY;
var
  k: integer;
begin
  for k := 1 to nc - 1 do
  begin
    if top[k] < 0 then //es una x
      iix[-top[k]] := k
    else //es una y
      iiy[top[k]] := -k;
  end;
  for k := 1 to nf - 1 do
  begin
    if left[k] < 0 then //es una x
      iix[-left[k]] := -k
    else //es una y
      iiy[left[k]] := k;
  end;
end;

//Indica si la restricción en kfila esta siendo cumplida
function TSimplex.filaEsFactible(kfila: integer; out fantasma: boolean): boolean;
var
  ix: integer;
begin
  //  if e(kfila, nc) < -CasiCero_Simplex then
  if pm[kfila].pv[nc] < -CasiCero_Simplex then
  begin
    //Si la fila es < 0
    fantasma := False;
    Result := False;
  end
  else if (left[kfila] < 0) then
  begin
    //Si rval es >= 0 reviso si es una variable con cota superior
    ix := -left[kfila];
    //    if (flg_x[ix] <> 0) and (e(kfila, nc) > (x_sup.pv[ix] + CasiCero_Simplex_CotaSup)) then
    if (flg_x[ix] <> 0) and ((pm[kfila].pv[nc]) >
      (x_sup.pv[ix] + CasiCero_Simplex_CotaSup)) then
    begin
      //Si violo la cota superior
      fantasma := True;
      Result := False;
    end
    //Sino
    else
    begin
      fantasma := False;
      Result := True;
    end;
  end
  else
  begin
    //Si es una y >= 0
    fantasma := False;
    Result := True;
  end;
end;

function TSimplex.defaultGetNombreVar(i: integer): string;
begin
  Result := 'Var' + IntToStr(i);
end;

function TSimplex.defaultGetNombreRes(i: integer): string;
begin
  Result := 'Res' + IntToStr(i);
end;

function TSimplex.getNombreFila(kfila: integer): string;
begin
  if left[kfila] < 0 then
  begin
    if Assigned(fGetNombreVar) then
      Result := fGetNombreVar(-left[kfila])
    else
    Result := 'x' + IntToStr(-left[kfila]);
  end
  else
  begin
    if Assigned(fGetNombreRes) then
      Result := fGetNombreRes(left[kfila])
    else
    Result := 'y' + IntToStr(left[kfila]);
  end;
end;

function TSimplex.getNombreCol(kcol: integer): string;
begin
  if top[kcol] < 0 then
  begin
    if Assigned(fGetNombreVar) then
      Result := fGetNombreVar(-top[kcol])
    else
    Result := 'x' + IntToStr(-top[kcol]);
  end
  else
  if Assigned(fGetNombreRes) then
    Result := fGetNombreRes(top[kcol])
  else
  Result := 'y' + IntToStr(top[kcol]);
end;

function TSimplex.esColumnaDeIgualdad(kColumna: integer): boolean;
begin
  Result := ((top[kColumna] > 0) and (abs(flg_y[top[kColumna]]) = 2));
end;


procedure ejemplo;
var
  i: integer;
  spx: TSimplex;
begin
  {
  min z = x1 + 3x2 + 2x3
  s.a.
    x1 + x2 + x3 >= 10.5
    x1 + x2 = 5.3
    x1 - x3 <= 2.9
    0 <= x1 <= 12, -6 <= x2 <= 6, -5 <= x3 <= 5

  =>

  max -z = -x1 -3x2 -2x3
  s.a.
    x1 + x2 + x3 -10.5 >= 0
    x1 + x2 - 5.3 = 0
    -x1 + x3 + 2.9 >= 0
    0 <= x1 <= 12, -6 <= x2 <= 6, -5 <= x3 <= 5
  }

  //Creamos un simplex vacío cuya matriz M tendrá:
  //3 restricciones + la función objetivo
  //3 variables + los términos independientes
  spx := TSimplex.Create_init(4, 4, nil, nil);

  //Cargamos la fila 1, pon_e(k, j, x) hace Mkj:= x
  spx.pon_e(1, 1, 1);
  spx.pon_e(1, 2, 1);
  spx.pon_e(1, 3, 1);
  spx.pon_e(1, spx.nc, -10.5);

  //Cargamos la fila 2 y la declaramos como de igualdad
  spx.pon_e(2, 1, 1);
  spx.pon_e(2, 2, 1);
  spx.pon_e(2, 3, 0);
  spx.pon_e(2, spx.nc, -5.3);
  spx.FijarRestriccionIgualdad(2);

  //Cargamos la fila 3
  spx.pon_e(3, 1, -1);
  spx.pon_e(3, 2, 0);
  spx.pon_e(3, 3, 1);
  spx.pon_e(3, spx.nc, 2.9);

  //Cargamos la fila objetivo z
  spx.pon_e(spx.nf, 1, -1);
  spx.pon_e(spx.nf, 2, -3);
  spx.pon_e(spx.nf, 3, -2);

  //cota_inf_set(i, x) fija la cota inferior de la variable en la
  //posición i a x, sota_sup_set hace lo propio con la cota superior
  //Cotas inferior y superior de x1
  spx.cota_inf_set(1, 0);
  spx.cota_sup_set(1, 12);

  //Cotas inferior y superior de x2
  spx.cota_inf_set(2, -6);
  spx.cota_sup_set(2, 6);

  //Cotas inferior y superior de x3
  spx.cota_inf_set(3, -5);
  spx.cota_sup_set(3, 5);

  //Vuelco el simplex al archivo 'ProblemaEjemplo.xlt' para verificar
  //que el problema armado sea el que quería
  //MAP COMENTED no needed now spx.DumpSistemaToXLT_('ProblemaEjemplo.xlt', '');

  //intento resolver
  if spx.resolver = 0 then
  begin
    //ok, encontró solución
    Writeln('Solución óptima encontrada:');
    //spx.fval obtiene el valor de z
    Writeln('z= ', FloatToStrF(-spx.fval, ffGeneral, 8, 4));
    Writeln;
    for i := 1 to 3 do
      //spx.xval(i) obtiene el valor de la variable i
      Writeln(#9, spx.fGetNombreVar(i), '= ', FloatToStrF(spx.xval(i), ffGeneral, 8, 3));
    Writeln;
    for i := 1 to 3 do
      //spx.yval(i) obtiene el valor de la restriccion i
      Writeln(#9, spx.fGetNombreRes(i), '= ', FloatToStrF(spx.yval(i), ffGeneral, 8, 3));
    Writeln('Presione <Enter> para continuar');
    Readln;
  end
  else
    //Error, lanzamos la excepción
    raise Exception.Create('Error resolviendo simplex: ' + spx.mensajeDeError);

  //Liberamos la memoria usada por el objeto
  spx.Free;
end;

end.
